//%%cu
//#include<bits/stdc++.h>
#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>

#define maxSize 10000000
#define block_size 1024

using namespace std;

struct cmap{
    int key;
    int value;
    int fill;
};


struct element_pair{
    int key;
    int value;
    int find;
};

struct element{
    int key;
    int find;
};


__global__ void kernel_index_to_fill(cmap *d_map, int *d_index_to_fill_in_hash_table, int *counter, int map_size){
    
     int tid = blockDim.x * blockIdx.x + threadIdx.x;                                              
     if(tid < map_size){
      
          if(d_map[tid].fill == 0){               
              
                int index = atomicInc((unsigned *)counter, maxSize);
                d_index_to_fill_in_hash_table[index] = tid;                
          }    
        
     }                     
    
}

__global__ void kernel_to_insert(cmap *d_map, element_pair *d_input, int *d_index_to_fill_in_hash_table, int insert_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;                                              
    if(tid < insert_batch_size && d_input[tid].find==0){
          d_map[d_index_to_fill_in_hash_table[tid]].key = d_input[tid].key;
          d_map[d_index_to_fill_in_hash_table[tid]].value = d_input[tid].value;
          d_map[d_index_to_fill_in_hash_table[tid]].fill = 1; 
    }
    
}


__global__ void search_kernel_pair(element_pair *d_search_input_pair, cmap *d_map, int a_size, int search_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < search_batch_size * a_size){

        int index_of_element = tid % search_batch_size;
        int index_of_location = tid / search_batch_size;

        if(d_search_input_pair[index_of_element].key == d_map[index_of_location].key
           && d_search_input_pair[index_of_element].value == d_map[index_of_location].value && d_map[index_of_location].fill == 1){

            d_search_input_pair[index_of_element].find = 1;

        }

    }
}


__global__ void search_kernel_key(element *d_search_input_key, cmap *d_map, int a_size, int search_batch_size, int *d_count){

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < search_batch_size * a_size){

        int index_of_element = tid % search_batch_size;
        int index_of_location = tid / search_batch_size;

        if(d_search_input_key[index_of_element].key == d_map[index_of_location].key){

            int temp = atomicInc((unsigned int *)d_count, -1);
            //d_search_input_pair[index_of_element].find = 1;

        }
    }

}

__global__ void fill_search_kernel_key(element *d_search_input_key, cmap *d_map, int a_size, int search_batch_size, element_pair *d_search_input_key_ans, int *index){

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < search_batch_size * a_size){

        int index_of_element = tid % search_batch_size;
        int index_of_location = tid / search_batch_size;

        if(d_search_input_key[index_of_element].key == d_map[index_of_location].key){
            int ind = atomicInc((unsigned int *)index, -1);
            d_search_input_key_ans[ind].key = d_map[index_of_location].key;
            d_search_input_key_ans[ind].value = d_map[index_of_location].value;
        }
    }
}



__global__ void delete_kernel_pair(element_pair *d_delete_input_pair, cmap *d_map, int a_size, int delete_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < delete_batch_size * a_size){

        int index_of_element = tid % delete_batch_size;
        int index_of_location = tid / delete_batch_size;

        if(d_delete_input_pair[index_of_element].key == d_map[index_of_location].key
           && d_delete_input_pair[index_of_element].value == d_map[index_of_location].value){

            d_map[index_of_location].key = 0;
            d_map[index_of_location].value = 0;
            d_map[index_of_location].fill = 0;
        }

    }

}

__global__ void delete_kernel(element *d_delete_input, cmap *d_map, int a_size, int delete_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < delete_batch_size * a_size){

        int index_of_element = tid % delete_batch_size;
        int index_of_location = tid / delete_batch_size;

        if(d_delete_input[index_of_element].key == d_map[index_of_location].key){

            d_map[index_of_location].key = 0;
            d_map[index_of_location].value = 0;
            d_map[index_of_location].fill = 0;
        }

    }

}



int main(){

    int map_size = 100000002;
    int map_element_counter = 0;

    cmap *d_map, *h_map;

    h_map = (cmap*)malloc(map_size*sizeof(cmap));
    memset(h_map, 0, map_size*sizeof(cmap));

    hipMalloc(&d_map, map_size*sizeof(cmap));
    hipMemset(d_map, 0, map_size*sizeof(cmap));

    struct timeval t1, t2;
    double seconds, microSeconds;

    int insert=1, search=0, delet=0,search_with_pair=0, search_with_key=1, insert_again = 0;

    
    if(insert == 1){       
        
        int insert_batch_size = 100000000;

        element_pair *d_input, *h_input;
        
        h_input = (element_pair*)malloc(insert_batch_size * sizeof(element_pair));

        
        for(int i=0 ; i<insert_batch_size ; i++){
            h_input[i].key = 1;
            h_input[i].value = i+1;
            h_input[i].find = 0;
        }
        
        gettimeofday(&t1, NULL);

        map_element_counter += insert_batch_size;

        if(map_element_counter < map_size)
        {

            hipMalloc(&d_input, insert_batch_size*sizeof(element_pair));
            hipMemcpy(d_input, h_input, insert_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);

            int block = ceil((float)(map_size*insert_batch_size)/(float)block_size);

            search_kernel_pair<<<block, block_size>>>(d_input, d_map, map_size, insert_batch_size);



            int *d_index_to_fill_in_hash_table, *counter;

            hipMalloc(&d_index_to_fill_in_hash_table, map_size*sizeof(int));
            hipMemset(d_index_to_fill_in_hash_table, 0, map_size*sizeof(int));

            hipMalloc(&counter, sizeof(int));
            hipMemset(counter, 0, sizeof(int));

            block = ceil((float)(map_size)/(float)block_size);

            kernel_index_to_fill<<<block, block_size>>>(d_map, d_index_to_fill_in_hash_table, counter, map_size);
            
            hipDeviceSynchronize();

            block = ceil((float)(insert_batch_size)/(float)block_size);

            kernel_to_insert<<<block, block_size>>>(d_map, d_input, d_index_to_fill_in_hash_table, insert_batch_size);

            hipDeviceSynchronize();

            hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

            gettimeofday(&t2, NULL);
            seconds = t2.tv_sec - t1.tv_sec;
            microSeconds = t2.tv_usec - t1.tv_usec;
            printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);


            int count = 0;
            for(int i=0 ; i<map_size ; i++){
                if(h_map[i].fill==1){
                  //cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
                  count++;
                }
            }

            cout<<"Total Value1 = "<<count<<endl;
            cout<<endl;

        }else
        {
            ;
        }
        
    }

    if(insert_again == 1){

        int insert_batch_size = 5;
        
        
        element_pair *d_input, *h_input;

        h_input = (element_pair*)malloc(insert_batch_size * sizeof(element_pair));

        for(int i=0 ; i<insert_batch_size ; i++){
            h_input[i].key = i;
            h_input[i].value = i;
            h_input[i].find = 0;
        }

        hipMalloc(&d_input, insert_batch_size*sizeof(element_pair));
        hipMemcpy(d_input, h_input, insert_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);

        
        


        int block = ceil((float)(map_size*insert_batch_size)/(float)block_size);

        search_kernel_pair<<<block, block_size>>>(d_input, d_map, map_size, insert_batch_size);

        hipDeviceSynchronize();
        int *d_index_to_fill_in_hash_table, *counter;

        hipMalloc(&d_index_to_fill_in_hash_table, map_size*sizeof(int));
        hipMemset(d_index_to_fill_in_hash_table, 0, map_size*sizeof(int));

        hipMalloc(&counter, sizeof(int));
        hipMemset(counter, 0, sizeof(int));

        block = ceil((float)(map_size)/(float)block_size);

        kernel_index_to_fill<<<block, block_size>>>(d_map, d_index_to_fill_in_hash_table, counter, map_size);
        
        hipDeviceSynchronize();

        block = ceil((float)(insert_batch_size)/(float)block_size);

        kernel_to_insert<<<block, block_size>>>(d_map, d_input, d_index_to_fill_in_hash_table, insert_batch_size);

        hipDeviceSynchronize();

        hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

        gettimeofday(&t2, NULL);
        seconds = t2.tv_sec - t1.tv_sec;
        microSeconds = t2.tv_usec - t1.tv_usec;
        printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);


        int count = 0;
        for(int i=0 ; i<map_size ; i++){
            if(h_map[i].fill==1){
             // cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
              count++;
            }
        }

        cout<<"Total Value1 = "<<count<<endl;
        cout<<endl;


        

    

    }



     // Search Code Data
    if(search==1){

        if(search_with_pair){

            element_pair *h_search_input_pair;
            element_pair *d_search_input_pair;

            //key, value to search

            int search_batch_size = 10000;
            

            h_search_input_pair = (element_pair*) malloc(search_batch_size * sizeof(element_pair));

            memset(h_search_input_pair, 0, search_batch_size*sizeof(element_pair));

            for(int i=0 ; i<search_batch_size ; i++){
                h_search_input_pair[i].key = i;
                h_search_input_pair[i].value = i+1;
            }

            gettimeofday(&t1, NULL);

            hipMalloc(&d_search_input_pair, search_batch_size*sizeof(element_pair));
            hipMemcpy(d_search_input_pair, h_search_input_pair, search_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);

                                      //existing array Size * batch_size
            int sblock = ceil((float)(map_size*search_batch_size)/(float)block_size);

            search_kernel_pair<<<sblock, block_size>>>(d_search_input_pair, d_map, map_size, search_batch_size);

            hipMemcpy(h_search_input_pair, d_search_input_pair, search_batch_size*sizeof(element_pair), hipMemcpyDeviceToHost);

            gettimeofday(&t2, NULL);
            seconds = t2.tv_sec - t1.tv_sec;
            microSeconds = t2.tv_usec - t1.tv_usec;
            printf("Time taken search pair(ms): %.3f\n", 1000*seconds + microSeconds/1000);

            int flag = 1;

            cout<<"\n\n\nFound value : \n";
            for(int i=0 ; i<search_batch_size ; i++){
                if(h_search_input_pair[i].find == 1 && flag){
                  cout<<"key = "<<h_search_input_pair[i].key<<" value = "<<h_search_input_pair[i].value<<" ";
                  flag=0;
                }
            }

            cout<<endl;
        }

      if(search_with_key){

            element *h_search_input_key;
            element *d_search_input_key;

            element_pair *h_search_input_key_ans;
            element_pair *d_search_input_key_ans;


            int *d_count, h_count, *d_index;


            hipMalloc(&d_count, sizeof(int));
            hipMemset(d_count, 0, sizeof(int));

            hipMalloc(&d_index, sizeof(int));
            hipMemset(d_index, 0, sizeof(int));

            //only key to search

            int search_batch_size = 3;
            

            h_search_input_key = (element*) malloc(search_batch_size * sizeof(element));

            memset(h_search_input_key, 0, search_batch_size*sizeof(element));

            for(int i=0 ; i<search_batch_size ; i++){
                h_search_input_key[i].key = 1;
            }

            gettimeofday(&t1, NULL);

            hipMalloc(&d_search_input_key, search_batch_size*sizeof(element));
            hipMemcpy(d_search_input_key, h_search_input_key, search_batch_size*sizeof(element), hipMemcpyHostToDevice);


                                      //existing array Size * batch_size
            int sblock = ceil((float)(map_size*search_batch_size)/(float)block_size);

            search_kernel_key<<<sblock, block_size>>>(d_search_input_key, d_map, map_size, search_batch_size, d_count);

            hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

            cout<<"\nFound Entry with Keys = "<<h_count<<endl;

            h_search_input_key_ans = (element_pair*)malloc(h_count * sizeof(element_pair));
            memset(h_search_input_key_ans, 0, h_count*sizeof(element_pair));


            hipMalloc(&d_search_input_key_ans, h_count*sizeof(element_pair));
            hipMemset(d_search_input_key_ans, 0, h_count*sizeof(element_pair));


                                      //existing array Size * batch_size
            sblock = ceil((float)(map_size*search_batch_size)/(float)block_size);

            fill_search_kernel_key<<<sblock, block_size>>>(d_search_input_key, d_map, map_size, search_batch_size, d_search_input_key_ans, d_index);



            hipMemcpy(h_search_input_key_ans, d_search_input_key_ans, h_count*sizeof(element_pair), hipMemcpyDeviceToHost);

            gettimeofday(&t2, NULL);
            seconds = t2.tv_sec - t1.tv_sec;
            microSeconds = t2.tv_usec - t1.tv_usec;
            printf("Time taken search keys(ms): %.3f\n", 1000*seconds + microSeconds/1000);



            int flag = 1;

            cout<<"\n\n\nFound value with keys : \n";
            for(int i=0 ; i<h_count ; i++){
                if(flag){
                  cout<<"key = "<<h_search_input_key_ans[i].key<<" value = "<<h_search_input_key_ans[i].value<<" ";
                  flag=0;
                }
            }

            cout<<endl;


      }

    }
    // Search Code ended

    // Delete Code
    if(delet == 1){

        //Delete Element with Key value pair
        int delete_with_pair = 0;
        if(delete_with_pair){

            element_pair *h_delete_input_pair;
            element_pair *d_delete_input_pair;

            int delete_batch_size = 3;
            

            h_delete_input_pair = (element_pair*) malloc(delete_batch_size * sizeof(element_pair));

            memset(h_delete_input_pair, 0, delete_batch_size*sizeof(element_pair));

            for(int i=0 ; i<delete_batch_size ; i++){
                h_delete_input_pair[i].key = 1;
                h_delete_input_pair[i].value = i+1;
            }

            gettimeofday(&t1, NULL);

            hipMalloc(&d_delete_input_pair, delete_batch_size*sizeof(element_pair));
            hipMemcpy(d_delete_input_pair, h_delete_input_pair, delete_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);


                                      //existing array Size * batch_size
            int sblock = ceil((float)(map_size*delete_batch_size)/(float)block_size);

            delete_kernel_pair<<<sblock, block_size>>>(d_delete_input_pair, d_map, map_size, delete_batch_size);

            hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

            gettimeofday(&t2, NULL);
            seconds = t2.tv_sec - t1.tv_sec;
            microSeconds = t2.tv_usec - t1.tv_usec;
            printf("Time taken Delate pair (ms): %.3f\n", 1000*seconds + microSeconds/1000);

            cout<<"\n\n After Delettion:\n\n";
            cout<<endl;
            for(int i=0 ; i<map_size ; i++){
                if(h_map[i].fill==1)
                  cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
                //cout<<h_map[i].value<<" ";
            }
        }else
        {

            element *h_delete_input;
            element *d_delete_input;

            int delete_batch_size = 1;
            

            h_delete_input = (element*) malloc(delete_batch_size * sizeof(element));

            memset(h_delete_input, 0, delete_batch_size*sizeof(element));

            for(int i=0 ; i<delete_batch_size ; i++){
                h_delete_input[i].key = 1;
            }

            hipMalloc(&d_delete_input, delete_batch_size*sizeof(element));
            hipMemcpy(d_delete_input, h_delete_input, delete_batch_size*sizeof(element), hipMemcpyHostToDevice);

                                      //existing array Size * batch_size
            int sblock = ceil((float)(map_size*delete_batch_size)/(float)block_size);

            delete_kernel<<<sblock, block_size>>>(d_delete_input, d_map, map_size, delete_batch_size);

            hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

            gettimeofday(&t2, NULL);
            seconds = t2.tv_sec - t1.tv_sec;
            microSeconds = t2.tv_usec - t1.tv_usec;
            printf("Time taken Delate key (ms): %.3f\n", 1000*seconds + microSeconds/1000);

            cout<<"\n\n After Delettion:\n\n";
            cout<<endl;
            for(int i=0 ; i<map_size ; i++){
                if(h_map[i].fill==1)
                  cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
                //cout<<h_map[i].value<<" ";
            }


        }
    }
    // Delete Code Ended




    return 0;
}
