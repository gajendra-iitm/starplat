// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRankDSLV3.h"

void ComputePageRank(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_src;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.srcList[i];
    h_src[i] = temp;
  }


  int* d_meta;
  int* d_src;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_pageRank;
  hipMalloc(&d_pageRank, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  float numNodes = (float)g.num_nodes( ); // asst in .cu

  float* d_pageRankNext;
  hipMalloc(&d_pageRankNext, sizeof(float)*(V));

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,(float)1 / numNodes);

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRankNext,(float)0);

  int iterCount = 0; // asst in .cu

  float diff; // asst in .cu

  do{
    diff = 0.000000;
    hipMemcpyToSymbol(HIP_SYMBOL(::diff), &diff, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::delta), &delta, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::numNodes), &numNodes, sizeof(float), 0, hipMemcpyHostToDevice);
    ComputePageRank_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_src,d_rev_meta,d_pageRank);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&diff, HIP_SYMBOL(::diff), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&delta, HIP_SYMBOL(::delta), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&numNodes, HIP_SYMBOL(::numNodes), sizeof(float), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    hipMemcpy(d_pageRank, d_pageRankNext, sizeof(float)*V, hipMemcpyDeviceToDevice);
    iterCount++;

  }while((diff > beta) && (iterCount < maxIter));

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_pageRankNext);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
