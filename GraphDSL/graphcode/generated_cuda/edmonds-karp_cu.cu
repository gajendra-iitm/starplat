// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "edmonds-karp_cu.h"

void ek(graph& g,int s,int d,int* weight
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  int* d_weight;
  hipMalloc(&d_weight, sizeof(int)*(V));


  //BEGIN DSL PARSING 
  int flow = 0; // asst in .cu

  int curint prevint new_flow = -1; // asst in .cu

  int cnew_flow = 0; // asst in .cu

  int e_cap = 0; // asst in .cu

  bool terminate = false; // asst in .cu

  int* d_cap;
  hipMalloc(&d_cap, sizeof(int)*(E));

  do{
    new_flow = 0;
    int* d_n_flow;
    hipMalloc(&d_n_flow, sizeof(int)*(V));

    int* d_par;
    hipMalloc(&d_par, sizeof(int)*(V));

    bool* d_in_bfs;
    hipMalloc(&d_in_bfs, sizeof(bool)*(V));

    merged_kernel_1<<<numBlocks,threadsPerBlock>>>(V, d_n_flow, (int)INT_MAX, d_par, (int)-1, d_in_bfs, (bool)false);
    d_in_bfs[s] = true;
    d_par[s] = -2;
    terminate = false;

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;
    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));  hipMemset(d_hops_from_source,0,sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    //EXTRA vars INITIALIZATION
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_level,-1);
    initIndex<int><<<1,1>>>(V,d_level,s, 0);

    // long k =0 ;// For DEBUG
    do {
      finished = true;
      hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished,d_weight); ///DONE from varList

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST - rupesh
      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    }while(!finished);

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished
        ,d_weight); ///DONE from varList

      hops_from_source--;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
    }
    //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);
    flow = flow + new_flow;
    cur = d;

    //hipFree up!! all propVars in this BLOCK!
    hipFree(d_in_bfs);
    hipFree(d_par);
    hipFree(d_n_flow);

  }while(new_flow != 0);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(  weight, d_weight, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN
