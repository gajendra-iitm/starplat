// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "graphColoringNew.h"

void colorGraph(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int numNodes = g.num_nodes( ); // asst in .cu

  long* d_color;
  hipMalloc(&d_color, sizeof(long)*(V));

  int* d_color1;
  hipMalloc(&d_color1, sizeof(int)*(V));

  int* d_color2;
  hipMalloc(&d_color2, sizeof(int)*(V));

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<long> <<<numBlocks,threadsPerBlock>>>(V,d_color,(long)0);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified_next,(bool)false);

  int fpoint1 = 0; // asst in .cu

  int diff = 0; // asst in .cu

  int diff_old = 0; // asst in .cu

  int cnt = 0; // asst in .cu

  int iter = 0; // asst in .cu

  do{
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(int), 0, hipMemcpyHostToDevice);
    colorGraph_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_modified,d_color,d_modified_next);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(int), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    iter = iter + 1;
    diff_old = diff;
    diff = fpoint1 - iter;
    if (diff == diff_old){ // if filter begin 
      cnt = cnt + 1;

    } // if filter end
    else
    if (diff != diff_old){ // if filter begin 
      cnt = 0;

    } // if filter end
    if (cnt == 3){ // if filter begin 
      iter = iter + numNodes - fpoint1;
      fpoint1 = numNodes;

    } // if filter end

  }while(fpoint1 < numNodes);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified_next);
  hipFree(d_modified);
  hipFree(d_color2);
  hipFree(d_color1);
  hipFree(d_color);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
