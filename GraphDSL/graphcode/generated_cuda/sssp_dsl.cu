#include "hip/hip_runtime.h"
#include"sssp_dsl.h"

__global__ void Compute_SSSP_kernel(graph& g,int src)

{
  int* dist=new int[g.num_nodes()];
  bool* modified=new bool[g.num_nodes()];
  bool* modified_nxt=new bool[g.num_nodes()];
  #pragma omp parallel for
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    dist[t] = INT_MAX;
    modified[t] = false;
    modified_nxt[t] = false;
  }
  modified[src] = true;
  dist[src] = 0;
  bool finished = false;
  while ( !finished )
  {
    finished = true;
    unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int v =id
    {
      if (modified[v] == true ){
        for (int edge = gpu_OA[id]; edge < gpu_OA[id+1]; edge ++) 
        {int nbr = g.edgeList[edge] ;
          int e = edge;
           int dist_new = dist[v] + weight[e];
          bool modified_new = true;
          if(dist[nbr] > dist_new)
          {
            int oldValue = dist[nbr];
            atomicMin(&dist[nbr],dist_new);
            if(oldValue > dist[nbr])
            {
              modified_nxt[nbr] = modified_new;
              finished = false ;
            }
          }
        }
      }
    }
  }

}