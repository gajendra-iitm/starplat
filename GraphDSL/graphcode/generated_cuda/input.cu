// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "input.h"

void test(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();







  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_prop;
  hipMalloc(&d_prop, sizeof(int)*(V));

  int x = 0; // asst in .cu

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_prop,(int)10);

  hipMemcpyToSymbol(HIP_SYMBOL(::x), &x, sizeof(int), 0, hipMemcpyHostToDevice);
  test_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_prop);
  hipDeviceSynchronize();




  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_prop);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
