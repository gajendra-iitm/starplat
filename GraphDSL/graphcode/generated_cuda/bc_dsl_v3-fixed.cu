#include "bc_dsl_v3.h"
///usr/local/cuda-9.2/bin/nvcc -o "bc_dsl_v3-fixed".out "bc_dsl_v3-fixed.cu"  -std=c++14 -rdc=true -arch=sm_70
void Compute_BC(graph& g,double* BC,std::set<int>& sourceSet)
{
  // CSR BEGIN
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  double * d_BC;hipMalloc(&d_BC, sizeof(int)*(V));
  //LAUNCH CONFIG

  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  //END CSR

  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING
  initKernel<double> <<<numBlocks,numThreads>>>(V,d_BC,0);

  double* d_sigma;
  hipMalloc(&d_sigma, sizeof(double)*(V));

  double* d_delta;
  hipMalloc(&d_delta, sizeof(double)*(V));

  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++)
  {
    unsigned src = *itr;
    initKernel<double> <<<numBlocks,numThreads>>>(V,d_delta,0);

    initKernel<double> <<<numBlocks,numThreads>>>(V,d_sigma,0);

    initIndex<double><<<1,1>>>(V,d_sigma,src, 1.0);


    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;

    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    initKernel<int> <<<numBlocks,numThreads>>>(V,d_level,-1);
    initIndex<int><<<1,1>>>(V,d_level,src, 0);

    long k =0 ;// For DEBUG
    do {
      finished = true;
      hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,numThreads>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished, d_BC);

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST - rupesh
      ++hops_from_source; // updating the level to process in the next iteration
      k++;

      hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    }while(!finished);

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,numThreads>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished, d_BC);

      hops_from_source--;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
    }
  }

  //ADD TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(BC,d_BC , sizeof(double) * (V), hipMemcpyDeviceToHost);

}

// driver program to test above function
int main(int argc , char ** argv)
{
  graph G(argv[1]);
  G.parseGraph();
  bool printAns = false;

  std::set<int> src;

  if(argc>3) { // ./a.out inputfile srcFile -p
      printAns = true;
  }

  // Check and READ Src file =================

  std::string line;
  std::ifstream srcfile(argv[2]);
  if (!srcfile.is_open()) {
    std::cout << "Unable to open src file :" << argv[1] << std::endl;
    exit(1);
  }


  int nodeVal;
  while ( std::getline (srcfile,line) ) {
   std::stringstream ss(line);
   ss>> nodeVal;
   //~ std::cout << "src " << nodeVal << '\n';
   src.insert(nodeVal);
  }

  srcfile.close();
  printf("#srces:%d\n",src.size());
  //==========================================




    //~ hipEvent_t start, stop; // should not be here!
    //~ hipEventCreate(&start);
    //~ hipEventCreate(&stop);
    //~ float milliseconds = 0;
    //~ hipEventRecord(start,0);
    unsigned V = G.num_nodes();
    unsigned E = G.num_nodes();
    double* BC = (double *)malloc(sizeof(double)*V);
    Compute_BC(G,BC,src);

    int LIMIT = 9;
    if(printAns)
     LIMIT=V;

    for (int i = 0; i < LIMIT; i++){
      printf("%lf\n", BC[i]);
    }

    //~ hipDeviceSynchronize();

    //~ hipEventRecord(stop,0);
    //~ hipEventSynchronize(stop);
    //~ hipEventElapsedTime(&milliseconds, start, stop);
    //~ printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    cudaCheckError();

  return 0;

}
