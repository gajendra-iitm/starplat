// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "sssp_dslV2.h"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  int* h_dist;
  h_dist= (int*)malloc(sizeof(int)*(V+1));
  int** d_dist;
  d_dist = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_dist[i], sizeof(int)*(V+1));
  }


  //BEGIN DSL PARSING 
  bool* h_modified1;
  h_modified1=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_modified1;
  d_modified1 = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified1[i], sizeof(bool)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist[i],(int)INT_MAX);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_dist+h_vertex_partition[i],d_dist[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified1[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified1+h_vertex_partition[i],d_modified1[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  //hi2
  h_modified1[src]=true;
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initIndex<bool><<<1,1>>>(V,d_modified1[i],src,(bool)true); //InitIndexDevice
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  //hi2
  h_dist[src]=0;
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initIndex<int><<<1,1>>>(V,d_dist[i],src,(int)0); //InitIndexDevice
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  bool finished = false; // asst in .cu 
  //fixed_pt_var
  bool** h_finished;
  h_finished = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_finished[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_finished;
  d_finished = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_finished[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_finished[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  bool** d_modified1_next;
  d_modified1_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified1_next[i], sizeof(bool)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified1_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!finished) {

    finished = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_finished[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_SSSP_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_dist[i],d_modified1[i],d_modified1_next[i],d_finished[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //global loop var v iden v
    if(devicecount>1){
      //pull based

      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_dist+h_vertex_partition[i],d_dist[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_dist[i],h_dist,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    bool* h_modified11;
    if(devicecount==1){
      hipMemcpy(d_modified1[0],d_modified1_next[0],sizeof(bool)*(V+1),hipMemcpyDeviceToDevice);
    }
    if(devicecount>1){
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified1+h_vertex_partition[i],d_modified1_next[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified1[i],h_modified1,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified1_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_finished[i], d_finished[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      finished&=h_finished[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_finished[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    int s = h_vertex_partition[i], e = h_vertex_partition[i+1] ; 
    hipMemcpyAsync(    dist + s,   d_dist[i] + s, sizeof(int)*(e-s), hipMemcpyDeviceToHost);
  }
  for (int i = 0 ; i < devicecount; i++){
    hipSetDevice(i) ; 
    hipDeviceSynchronize();
  }
} //end FUN
