#include "multicolor.h"

void colorGraph(graph& g){
    int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  printf(" device count %d\n",devicecount);
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  unsigned long ** d_color ;
  d_color = (unsigned long**)malloc(sizeof(unsigned long*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_color[i],sizeof(unsigned long)*(V+1));
  }
  unsigned int ** d_color1 ;
  d_color1 = (unsigned int**)malloc(sizeof(unsigned int*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_color1[i],sizeof(unsigned int)*(V+1));
  }
  unsigned int ** d_color2 ;
  d_color2 = (unsigned int**)malloc(sizeof(unsigned int*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_color2[i],sizeof(unsigned int)*(V+1));
  }
  


  // unsigned long** h_color;
  // h_color = (unsigned long**)malloc(sizeof(unsigned long*)*devicecount);
  // for(int i=0;i<devicecount;i+=1){
  //   h_color[i]=(unsigned long*)malloc(sizeof(unsigned long)*(V+1));
  // }

  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_modified[i],sizeof(bool)*(V+1));
  }
  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i],sizeof(bool)*(V+1));
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initKernel<unsigned long><<<numBlocks,threadsPerBlock>>>(V,d_color[i],(unsigned long)0.000);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V,d_modified_next[i],(bool)false);
    hipDeviceSynchronize();
  }

  int fpoint1 = 0; // asst in .cu 
  int** h_fpoint1;
  h_fpoint1 = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint1[i] = (int*)malloc(sizeof(int));
  }

  int** d_fpoint1;
  d_fpoint1 = (int**)malloc(sizeof(int*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_fpoint1[i],(int)0);
  }

  int iter = 0;
  do{
    printf("iter val %d\n",iter);
    // printf("iter value %d\n",iter);
    iter+=1;
    int x = rand();
    int y = rand();
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        hiprandSetPseudoRandomGeneratorSeed(gen,x);       
        hiprandGenerate(gen,d_color1[i],(V+1));
        hiprandSetPseudoRandomGeneratorSeed(gen,y);
        hiprandGenerate(gen,d_color2[i],(V+1));
        initialize<<<numBlocks,numThreads>>>(V,d_color[i],d_color1[i],d_color2[i]);
        // hipMemcpyAsync(h_color[i],d_color[i],(V+1)*sizeof(unsigned long),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    // printf("=========================================\n");
    // for(int i=0;i<devicecount;i+=1){
    //   for(int j=0;j<=V;j+=1){
    //     printf("%0.15lf ",h_color[i][j]);
    //   }
    //   printf("\n");
    // }

    //   printf("=========================================\n");
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        compute_colors<<<numBlocks,numThreads>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_src[i],d_rev_meta[i],d_color[i],d_modified[i],d_modified_next[i],d_fpoint1[i]);
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i+=1){
        hipSetDevice(i);
        hipMemcpyAsync(h_fpoint1[i],d_fpoint1[i],sizeof(int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize(); 
    }
    fpoint1 = 0;
    for(int i=0;i<devicecount;i+=1){
      fpoint1+=h_fpoint1[i][0];
    }
    // printf("%d\n",fpoint1);
    // for(int i=0;i<devicecount;i+=1){
    //   hipSetDevice(i);
    //   initKernel<int> <<<1,1>>>(1,d_fpoint1[i],(int)0);
    //   hipDeviceSynchronize();
    // }
    bool* h_modified_next;
    h_modified_next =(bool*)malloc((V+1)*sizeof(bool));
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified_next+h_vertex_partition[i],d_modified_next[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified[i],h_modified_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified_next[i],h_modified_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }
    // for(int i=0;i<=V;i+=1){
      // printf("%d ",h_modified_next[i]);
    // }
    // printf("\n");
    // printf("%d ended\n",iter);

  }while(fpoint1<V);
  printf("num colors %d\n",iter);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
}

int main(int argc,char* argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  colorGraph(g);
  return 0;
}
