// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_dsl1.h"

void Boruvka(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;

  if(devicecount>1){
    numBlocksKernel = numBlocksKernel/devicecount+1;
  }


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* h_nodeId;
  h_nodeId=(int*)malloc(sizeof(int)*V+1);
  int** d_nodeId;
  d_nodeId = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_nodeId[i], sizeof(int)*(V+1));
  }

  int* h_color;
  h_color=(int*)malloc(sizeof(int)*V+1);
  int** d_color;
  d_color = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_color[i], sizeof(int)*(V+1));
  }

  bool* h_isMSTEdge;
  h_isMSTEdge=(bool*)malloc(sizeof(bool)*E);
  bool** d_isMSTEdge;
  d_isMSTEdge = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_isMSTEdge[i], sizeof(bool)*E);
  }

  bool* h_isMSTEdge_temp1 = (bool*)malloc(E*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_isMSTEdge_temp1;
  hipMalloc(&d_isMSTEdge_temp1,E*(devicecount)*sizeof(bool));
  bool* d_isMSTEdge_temp2;
  hipMalloc(&d_isMSTEdge_temp2,E*(devicecount)*sizeof(bool));


  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_nodeId[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_nodeId+h_vertex_partition[i],d_nodeId[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_color+h_vertex_partition[i],d_color[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks_Edge,threadsPerBlock>>>(E,d_isMSTEdge[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  hipMemcpyAsync(h_isMSTEdge,d_isMSTEdge[0],E*sizeof(bool),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    Boruvka_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_color[i],d_nodeId[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  if(devicecount>1){
    //u u
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_nodeId+h_vertex_partition[i],d_nodeId[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_nodeId[i],h_nodeId,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //u u
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_color+h_vertex_partition[i],d_color[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_color[i],h_color,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  bool* h_modified;
  h_modified=(bool*)malloc(sizeof(bool)*V+1);
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified+h_vertex_partition[i],d_modified[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  int* h_minEdgeOfComp;
  h_minEdgeOfComp=(int*)malloc(sizeof(int)*V+1);
  int** d_minEdgeOfComp;
  d_minEdgeOfComp = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_minEdgeOfComp[i], sizeof(int)*(V+1));
  }

  int* h_minEdgeOfComp_temp1 = (int*)malloc(V+1*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_minEdgeOfComp_temp1;
  hipMalloc(&d_minEdgeOfComp_temp1,V+1*(devicecount)*sizeof(int));
  int* d_minEdgeOfComp_temp2;
  hipMalloc(&d_minEdgeOfComp_temp2,V+1*(devicecount)*sizeof(int));


  int* h_minEdge;
  h_minEdge=(int*)malloc(sizeof(int)*V+1);
  int** d_minEdge;
  d_minEdge = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_minEdge[i], sizeof(int)*(V+1));
  }

  bool noNewComp = false; // asst in .cu 
  bool** h_noNewComp;
  h_noNewComp = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_noNewComp[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_noNewComp;
  d_noNewComp = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_noNewComp[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_noNewComp[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!noNewComp) {

    noNewComp = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_noNewComp[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdge[i],(int)-1);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_minEdge+h_vertex_partition[i],d_minEdge[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Boruvka_kernel2<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_minEdge[i],d_color[i],d_noNewComp[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_minEdge+h_vertex_partition[i],d_minEdge[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_minEdge[i],h_minEdge,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_minEdge+h_vertex_partition[i],d_minEdge[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_minEdge[i],h_minEdge,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdgeOfComp[i],(int)-1);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_minEdgeOfComp+h_vertex_partition[i],d_minEdgeOfComp[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    bool finishedMinEdge = false; // asst in .cu 
    bool** h_finishedMinEdge;
    h_finishedMinEdge = (bool**)malloc(sizeof(bool*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_finishedMinEdge[i] = (bool*)malloc(sizeof(bool));
    }

    bool** d_finishedMinEdge;
    d_finishedMinEdge = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_finishedMinEdge[i],sizeof(bool));
      initKernel<bool> <<<1,1>>>(1,d_finishedMinEdge[i],false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    while(!finishedMinEdge) {

      finishedMinEdge = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_finishedMinEdge[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        Boruvka_kernel3<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_minEdge[i],d_minEdgeOfComp[i],d_color[i],d_finishedMinEdge[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }




      if(devicecount>1){
        //u comp
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_minEdgeOfComp_temp1+i*V+1,d_minEdgeOfComp[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_minEdgeOfComp_temp1,h_minEdgeOfComp_temp1,V+1*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_minEdgeOfComp_temp2+i*V+1,h_minEdgeOfComp,sizeof(int)*V+1,hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_minEdgeOfComp_temp1,d_minEdgeOfComp_temp2,V+1,devicecount);
        hipMemcpy(h_minEdgeOfComp,d_minEdgeOfComp_temp1,V+1*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_minEdgeOfComp[i],h_minEdgeOfComp,V+1*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //u comp
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_minEdgeOfComp_temp1+i*V+1,d_minEdgeOfComp[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_minEdgeOfComp_temp1,h_minEdgeOfComp_temp1,V+1*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_minEdgeOfComp_temp2+i*V+1,h_minEdgeOfComp,sizeof(int)*V+1,hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_minEdgeOfComp_temp1,d_minEdgeOfComp_temp2,V+1,devicecount);
        hipMemcpy(h_minEdgeOfComp,d_minEdgeOfComp_temp1,V+1*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_minEdgeOfComp[i],h_minEdgeOfComp,V+1*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_finishedMinEdge[i], d_finishedMinEdge[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        finishedMinEdge&=h_finishedMinEdge[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_finishedMinEdge[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Boruvka_kernel4<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_color[i],d_nodeId[i],d_minEdgeOfComp[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //src dstLead
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_minEdgeOfComp_temp1+i*V+1,d_minEdgeOfComp[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_minEdgeOfComp_temp1,h_minEdgeOfComp_temp1,V+1*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_minEdgeOfComp_temp2+i*V+1,h_minEdgeOfComp,sizeof(int)*V+1,hipMemcpyHostToDevice);
      }
      Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_minEdgeOfComp_temp1,d_minEdgeOfComp_temp2,V+1,devicecount);
      hipMemcpy(h_minEdgeOfComp,d_minEdgeOfComp_temp1,V+1*sizeof(int),hipMemcpyDeviceToHost);
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_minEdgeOfComp[i],h_minEdgeOfComp,V+1*sizeof(int),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Boruvka_kernel5<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_color[i],d_nodeId[i],d_minEdgeOfComp[i],d_isMSTEdge[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //src srcMinEdge
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_isMSTEdge_temp1+i*E,d_isMSTEdge[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_isMSTEdge_temp1,h_isMSTEdge_temp1,E*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_isMSTEdge_temp2+i*E,h_isMSTEdge,sizeof(bool)*E,hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_isMSTEdge_temp1,d_isMSTEdge_temp2,E,devicecount);
      hipMemcpy(h_isMSTEdge,d_isMSTEdge_temp1,E*sizeof(bool),hipMemcpyDeviceToHost);
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_isMSTEdge[i],h_isMSTEdge,E*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Boruvka_kernel6<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_color[i],d_nodeId[i],d_minEdgeOfComp[i],d_noNewComp[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_color+h_vertex_partition[i],d_color[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_color[i],h_color,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    bool finished = false; // asst in .cu 
    bool** h_finished;
    h_finished = (bool**)malloc(sizeof(bool*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_finished[i] = (bool*)malloc(sizeof(bool));
    }

    bool** d_finished;
    d_finished = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_finished[i],sizeof(bool));
      initKernel<bool> <<<1,1>>>(1,d_finished[i],false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    while(!finished) {

      finished = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_finished[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        Boruvka_kernel7<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_color[i],d_finished[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }


      if(devicecount>1){
        //u u
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_color+h_vertex_partition[i],d_color[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_color[i],h_color,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_finished[i], d_finished[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        finished&=h_finished[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_finished[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_noNewComp[i], d_noNewComp[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      noNewComp&=h_noNewComp[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_noNewComp[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
