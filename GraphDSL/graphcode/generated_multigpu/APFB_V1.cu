// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "APFB_V1.h"

void APFB(graph& g,int nc)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;

  if(devicecount>1){
    numBlocksKernel = numBlocksKernel/devicecount+1;
  }


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  int** d_nc;
  d_nc = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_nc[i], sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_nc[i],nc);
  }

  int** h_nc = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_nc[i] = (int*)malloc(sizeof(int));
  }

  //BEGIN DSL PARSING 
  bool* h_modified;
  h_modified=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified+h_vertex_partition[i],d_modified[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  int* h_rmatch;
  h_rmatch=(int*)malloc(sizeof(int)*(V+1));
  int** d_rmatch;
  d_rmatch = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_rmatch[i], sizeof(int)*(V+1));
  }

  int* h_rmatch_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_rmatch_temp1;
  hipMalloc(&d_rmatch_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_rmatch_temp2;
  hipMalloc(&d_rmatch_temp2,(V+1)*(devicecount)*sizeof(int));


  int* h_cmatch;
  h_cmatch=(int*)malloc(sizeof(int)*(V+1));
  int** d_cmatch;
  d_cmatch = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_cmatch[i], sizeof(int)*(V+1));
  }

  int* h_cmatch_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_cmatch_temp1;
  hipMalloc(&d_cmatch_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_cmatch_temp2;
  hipMalloc(&d_cmatch_temp2,(V+1)*(devicecount)*sizeof(int));


  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_rmatch[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_rmatch+h_vertex_partition[i],d_rmatch[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_cmatch[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_cmatch+h_vertex_partition[i],d_cmatch[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  bool noNewPaths = false; // asst in .cu 
  bool** h_noNewPaths;
  h_noNewPaths = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_noNewPaths[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_noNewPaths;
  d_noNewPaths = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_noNewPaths[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_noNewPaths[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  int* h_bfsArray;
  h_bfsArray=(int*)malloc(sizeof(int)*(V+1));
  int** d_bfsArray;
  d_bfsArray = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_bfsArray[i], sizeof(int)*(V+1));
  }

  int* h_bfsArray_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_bfsArray_temp1;
  hipMalloc(&d_bfsArray_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_bfsArray_temp2;
  hipMalloc(&d_bfsArray_temp2,(V+1)*(devicecount)*sizeof(int));


  int* h_predeccesor;
  h_predeccesor=(int*)malloc(sizeof(int)*(V+1));
  int** d_predeccesor;
  d_predeccesor = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_predeccesor[i], sizeof(int)*(V+1));
  }

  int* h_predeccesor_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_predeccesor_temp1;
  hipMalloc(&d_predeccesor_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_predeccesor_temp2;
  hipMalloc(&d_predeccesor_temp2,(V+1)*(devicecount)*sizeof(int));


  bool* h_compress;
  h_compress=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_compress;
  d_compress = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_compress[i], sizeof(bool)*(V+1));
  }

  bool* h_compress_next;
  h_compress_next=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_compress_next;
  d_compress_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_compress_next[i], sizeof(bool)*(V+1));
  }

  bool* h_compress_next_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_compress_next_temp1;
  hipMalloc(&d_compress_next_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_compress_next_temp2;
  hipMalloc(&d_compress_next_temp2,(V+1)*(devicecount)*sizeof(bool));


  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!noNewPaths) {

    noNewPaths = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_noNewPaths[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    int L0 = 0; // asst in .cu 
    int** h_L0;
    h_L0 = (int**)malloc(sizeof(int*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_L0[i] = (int*)malloc(sizeof(int));
    }

    int** d_L0;
    d_L0 = (int**)malloc(sizeof(int*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_L0[i],sizeof(int));
      initKernel<int> <<<1,1>>>(1,d_L0[i],0);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    int NOT_VISITED = L0 - 1; // asst in .cu 
    int** h_NOT_VISITED;
    h_NOT_VISITED = (int**)malloc(sizeof(int*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_NOT_VISITED[i] = (int*)malloc(sizeof(int));
    }

    int** d_NOT_VISITED;
    d_NOT_VISITED = (int**)malloc(sizeof(int*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_NOT_VISITED[i],sizeof(int));
      initKernel<int> <<<1,1>>>(1,d_NOT_VISITED[i],L0 - 1);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_bfsArray[i],(int)NOT_VISITED);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_bfsArray+h_vertex_partition[i],d_bfsArray[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      APFB_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_nc[i],d_cmatch[i],d_L0[i],d_bfsArray[i],d_noNewPaths[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //c c
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_bfsArray+h_vertex_partition[i],d_bfsArray[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_bfsArray[i],h_bfsArray,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_predeccesor[i],(int)-1);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_predeccesor+h_vertex_partition[i],d_predeccesor[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    int bfsLevel = L0; // asst in .cu 
    int** h_bfsLevel;
    h_bfsLevel = (int**)malloc(sizeof(int*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_bfsLevel[i] = (int*)malloc(sizeof(int));
    }

    int** d_bfsLevel;
    d_bfsLevel = (int**)malloc(sizeof(int*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_bfsLevel[i],sizeof(int));
      initKernel<int> <<<1,1>>>(1,d_bfsLevel[i],L0);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    bool noNewVertices = false; // asst in .cu 
    bool** h_noNewVertices;
    h_noNewVertices = (bool**)malloc(sizeof(bool*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_noNewVertices[i] = (bool*)malloc(sizeof(bool));
    }

    bool** d_noNewVertices;
    d_noNewVertices = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_noNewVertices[i],sizeof(bool));
      initKernel<bool> <<<1,1>>>(1,d_noNewVertices[i],false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    while(!noNewVertices) {

      noNewVertices = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_noNewVertices[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        APFB_kernel2<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_bfsArray[i],d_nc[i],d_bfsLevel[i],d_rmatch[i],d_NOT_VISITED[i],d_noNewVertices[i],d_predeccesor[i],d_noNewPaths[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }

      if(devicecount>1){
        //col_vertex col_match
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_bfsArray_temp1+i*(V+1),d_bfsArray[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_bfsArray_temp1,h_bfsArray_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_bfsArray_temp2+i*(V+1),h_bfsArray,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_bfsArray_temp1,d_bfsArray_temp2,V,devicecount);
        hipMemcpy(h_bfsArray,d_bfsArray_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_bfsArray[i],h_bfsArray,(V+1)*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //col_vertex neigh_row
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_predeccesor_temp1+i*(V+1),d_predeccesor[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_predeccesor_temp1,h_predeccesor_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_predeccesor_temp2+i*(V+1),h_predeccesor,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_predeccesor_temp1,d_predeccesor_temp2,V,devicecount);
        hipMemcpy(h_predeccesor,d_predeccesor_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_predeccesor[i],h_predeccesor,(V+1)*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //col_vertex neigh_row
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_rmatch_temp1+i*(V+1),d_rmatch[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_rmatch_temp1,h_rmatch_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_rmatch_temp2+i*(V+1),h_rmatch,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_rmatch_temp1,d_rmatch_temp2,V,devicecount);
        hipMemcpy(h_rmatch,d_rmatch_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_rmatch[i],h_rmatch,(V+1)*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //col_vertex neigh_row
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_predeccesor_temp1+i*(V+1),d_predeccesor[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_predeccesor_temp1,h_predeccesor_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_predeccesor_temp2+i*(V+1),h_predeccesor,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_predeccesor_temp1,d_predeccesor_temp2,V,devicecount);
        hipMemcpy(h_predeccesor,d_predeccesor_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_predeccesor[i],h_predeccesor,(V+1)*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      bfsLevel = bfsLevel + 1;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        //printed here

        initKernel<int> <<<1,1>>>(1,d_bfsLevel[i],(int)bfsLevel);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_noNewVertices[i], d_noNewVertices[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        noNewVertices&=h_noNewVertices[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_noNewVertices[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_compress[i],(bool)false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_compress+h_vertex_partition[i],d_compress[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_compress_next[i],(bool)false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_compress_next+h_vertex_partition[i],d_compress_next[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      APFB_kernel3<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_nc[i],d_rmatch[i],d_compress[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //r r
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_compress+h_vertex_partition[i],d_compress[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_compress[i],h_compress,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    bool compressed = false; // asst in .cu 
    bool** h_compressed;
    h_compressed = (bool**)malloc(sizeof(bool*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_compressed[i] = (bool*)malloc(sizeof(bool));
    }

    bool** d_compressed;
    d_compressed = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_compressed[i],sizeof(bool));
      initKernel<bool> <<<1,1>>>(1,d_compressed[i],false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    while(!compressed) {

      compressed = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_compressed[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        APFB_kernel4<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_compress[i],d_nc[i],d_cmatch[i],d_predeccesor[i],d_compressed[i],d_compress_next[i],d_rmatch[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }



      if(devicecount>1){
        //row_vertex matched_col
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_cmatch_temp1+i*(V+1),d_cmatch[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_cmatch_temp1,h_cmatch_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_cmatch_temp2+i*(V+1),h_cmatch,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_cmatch_temp1,d_cmatch_temp2,V,devicecount);
        hipMemcpy(h_cmatch,d_cmatch_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_cmatch[i],h_cmatch,(V+1)*sizeof(int),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //row_vertex row_vertex
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_rmatch+h_vertex_partition[i],d_rmatch[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_rmatch[i],h_rmatch,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //row_vertex matched_row
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_compress_next_temp1+i*(V+1),d_compress_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_compress_next_temp1,h_compress_next_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_compress_next_temp2+i*(V+1),h_compress_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_compress_next_temp1,d_compress_next_temp2,V,devicecount);
        hipMemcpy(h_compress_next,d_compress_next_temp1,(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_compress_next[i],h_compress_next,(V+1)*sizeof(bool),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        APFB_kernel5<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_nc[i],d_compress_next[i],d_compress[i],d_compressed[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      if(devicecount>1){
        //row_vertex row_vertex
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_compress+h_vertex_partition[i],d_compress[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_compress[i],h_compress,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //row_vertex row_vertex
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_compress_next+h_vertex_partition[i],d_compress_next[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_compress_next[i],h_compress_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_compressed[i], d_compressed[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        compressed&=h_compressed[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_compressed[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      APFB_kernel6<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_nc[i],d_cmatch[i],d_rmatch[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    if(devicecount>1){
      //r r
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_rmatch+h_vertex_partition[i],d_rmatch[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_rmatch[i],h_rmatch,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //r r
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_rmatch+h_vertex_partition[i],d_rmatch[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_rmatch[i],h_rmatch,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_noNewPaths[i], d_noNewPaths[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      noNewPaths&=h_noNewPaths[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_noNewPaths[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
