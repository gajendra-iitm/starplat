#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdlib.h>
#include <chrono>
#include <ctime>
#include <ratio>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

#include <climits>
#include <algorithm>
#include <assert.h> 

using namespace std;

const int INF = 1000 * 1000 * 1000;

#define cudaCheckError()                                                                     \
    {                                                                                        \
        hipError_t e = hipGetLastError();                                                  \
        if (e != hipSuccess)                                                                \
        {                                                                                    \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(0);                                                                         \
        }                                                                                    \
    }

struct edge
{
    int from, to, wt = 0;
};


template <typename T>
__global__ void initKernel(int V, T *init_array, T init_value)
{
    unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < V)
    {
        init_array[id] = init_value;
    }
}

template <typename T>
__global__ void initKernel0(T *init_array, T id, T init_value)
{ 
    init_array[id] = init_value;
}

__global__ void SSSPKernel(int vertex_partition_start, int vertex_partition_end, int *gpu_dist, int *gpu_modified, struct edge *adj, int *offset)
{
    unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
    int num_vertices = vertex_partition_end - vertex_partition_start;
    if (id < num_vertices)
    {
        int vertex = vertex_partition_start + id;
        //printf("%d\n",vertex);
        for (int i = offset[vertex]; i < offset[vertex + 1]; i++)
        {
            int u = adj[i].to, v = adj[i].from, w = adj[i].wt;
            //assert(v == vertex);
            int new_dist = gpu_dist[u] + w;
           // printf("vertex %d %d %d %d\n", v,u,w,new_dist);
            if (gpu_dist[v] > new_dist)
            {
                gpu_dist[v] = new_dist;
                gpu_modified[0] = 1;
            }
        }
    }
}

void sssp(int n, int m, struct edge *adj, int *offset)
{
    int devices = 1;
    //cudaGetDeviceCount(&devices);
    cout << "Num devices = " << devices << endl;

    int *vertex_partition_start = (int *)malloc((devices + 1) * sizeof(int));
    int vertex_per_gpu = n / devices;
    int curr = 0;
    vertex_partition_start[0] = 0;
    vertex_partition_start[devices] = n;
    for (int i = 1; i < devices; i++)
    {
        if (i <= (n % devices))
        {
            curr += vertex_per_gpu + 1;
        }
        else
            curr += vertex_per_gpu;
        vertex_partition_start[i] = curr;
    }

    int block_size = n;
    int num_blocks = 1;
    if (n > 1024)
    {
        block_size = 1024;
        num_blocks = (n + block_size - 1) / block_size; // avoid ceil fun call
    }

    cudaCheckError(); 

    hipStream_t *streams;
    streams = (hipStream_t *)malloc(devices * sizeof(hipStream_t));
    for(int i=0 ;i< devices;i++){
        hipSetDevice(i);
        hipStreamCreate(&streams[i]);
    }

    int *host_dist, **device_dist;
    hipHostMalloc(&host_dist, n * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&device_dist, devices * sizeof(int *), hipHostMallocDefault);
    for (int i = 0; i < devices; i++)
    {
        hipSetDevice(i);
        hipMalloc(&device_dist[i], n * sizeof(int));
        initKernel<<<num_blocks, block_size,0,streams[i]>>>(n, device_dist[i], INF);
        initKernel0<<<1, 1,0,streams[i]>>>(device_dist[i], 0, 0);
    }
    

    cudaCheckError();

    int **host_modified, **device_modified;
    hipHostMalloc(&host_modified, devices * sizeof(int *), hipHostMallocDefault);
    hipHostMalloc(&device_modified, devices * sizeof(int *), hipHostMallocDefault);
    for (int i = 0; i < devices; i++)
    {
        hipHostMalloc(&host_modified[i], sizeof(int), hipHostMallocDefault);
    }
    for (int i = 0; i < devices; i++)
    {
        hipSetDevice(i);
        hipMalloc(&device_modified[i], sizeof(int));
        initKernel0<<<1, 1,0,streams[i]>>>(device_modified[i], 0, 0);
    }
    
    cudaCheckError();

    struct edge **device_edges;
    hipHostMalloc(&device_edges, devices * sizeof(struct edge *), hipHostMallocDefault);
    for (int i = 0; i < devices; i++)
    {
        hipSetDevice(i);
        hipMalloc(&device_edges[i], m * sizeof(struct edge));
        hipMemcpyAsync(device_edges[i], adj, m * sizeof(struct edge), hipMemcpyHostToDevice,streams[i]);
    }

    cudaCheckError();

    int **device_offset;
    hipHostMalloc(&device_offset, devices * sizeof(int *), hipHostMallocDefault);
    for (int i = 0; i < devices; i++)
    {
        hipSetDevice(i);
        hipMalloc(&device_offset[i], (n+1) * sizeof(int));
        hipMemcpyAsync(device_offset[i], offset, (n+1) * sizeof(int), hipMemcpyHostToDevice,streams[i]);
    }

    for(int i=0;i<devices;i++){
        hipSetDevice(i);
             hipStreamSynchronize(streams[i]);
    }


    hipSetDevice(0);
    hipEvent_t start_event, stop_event; /// TIMER START
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    float milliseconds = 0;
    hipEventRecord(start_event, 0);
    cudaCheckError();

    // for(int i=0;i<=devices;i++){
    //     cout<<i<<" "<<vertex_partition_start[i]<<endl;
    // }


    int iterations = 0;

    while (iterations < n)
    {

        for (int i = 0; i < devices; i++)
        {
            hipSetDevice(i);
            SSSPKernel<<<num_blocks, block_size,0,streams[i]>>>(vertex_partition_start[i], vertex_partition_start[i + 1], device_dist[i], device_modified[i], device_edges[i], device_offset[i]);
            hipMemcpyAsync(host_dist + vertex_partition_start[i], device_dist[i] + vertex_partition_start[i], (vertex_partition_start[i + 1] - vertex_partition_start[i])*sizeof(int), hipMemcpyDeviceToHost,streams[i]);
            hipMemcpyAsync(host_modified[i], device_modified[i], sizeof(int), hipMemcpyDeviceToHost,streams[i]);
        }

        for(int i=0;i<devices;i++){
            hipSetDevice(i);
             hipStreamSynchronize(streams[i]);
        }

        // for(int i=0;i<n;i++){
        //     cout<<iterations<<" "<<i<<" "<<host_dist[i]<<endl;
        // }
        
        int modified = 0;
        for (int i = 0; i < devices; i++)
        {
            modified |= host_modified[i][0];
            host_modified[i][0] = 0;
        }

        if (modified == 0)
        {
            cout << "Completed" << endl;
            break;
        }

        for (int i = 0; i < devices; i++)
        {
            hipSetDevice(i);
            hipMemcpyAsync(device_dist[i], host_dist, n * sizeof(int), hipMemcpyHostToDevice,streams[i]);
            hipMemcpyAsync(device_modified[i], host_modified[i], sizeof(int), hipMemcpyHostToDevice,streams[i]);
        }

        for(int i=0;i<devices;i++){
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        iterations++;
        if (iterations > n + 1 || iterations > 10000)
        {
            cout << "Aborting" << endl;
            break;
        }

        //cudaCheckError();
    }


    hipSetDevice(0);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    printf("GPU Time: %.6f ms \nIterations:%d\n", milliseconds, iterations);

    cudaCheckError();

    int debug = 0;
    if (debug)
    {
        for (int i = 0; i < n; i++)
        {
            cout << i << " " << host_dist[i] << endl;
        }
    }
    cout<<"Distance from vertex 0 to vertex 6 is ";
    cout<<host_dist[6]<<endl;
}

int main(int argc, char **argv)
{
    cout << "Reading input" << endl;
    ifstream input(argv[1]);
    int n, m;
    input >> n >> m;
    m *= 2;
    struct edge *adj;
    adj = (struct edge *)malloc(m * sizeof(struct edge));
    for (int i = 0; i < m; i += 2)
    {
        int u, v, w;
        input >> u >> v >> w;
        adj[i].from = u;
        adj[i].to = v;
        adj[i].wt = w;
        adj[i + 1].to = u;
        adj[i + 1].from = v;
        adj[i + 1].wt = w;
    }
    cout << "Reading input successfull" << endl;

    sort(adj, adj + m, [&](struct edge a, struct edge b)
         {
        if(a.from != b.from){
            return a.from<b.from;
        }
        else return a.to<b.to; });

    int *offset = (int *)malloc((n + 1) * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        offset[i] = -1;
    }
    offset[n] = m;
    int curr = -1;
    for (int i = 0; i < m; i++)
    {
        if (adj[i].from > curr)
        {
            curr = adj[i].from;
            offset[curr] = i;
        }
    }
    for (int i = n - 1; i >= 0; i--)
    {
        if (offset[i] == -1)
        {
            offset[i] = offset[i + 1];
        }
    }

    // for (int i = 0; i < m; i++)
    // {
    //     cout << adj[i].from << " " << adj[i].to << " " << adj[i].wt << endl;
    // }

    // for (int i = 0; i <= n; i++)
    // {
    //     cout << offset[i] << endl;
    // }

    sssp(n, m, adj, offset);

    // cout << "*******************************" << endl;
    // cout << "Operation succuessful" << endl;

    // cout << "*******************************" << endl;
}
