// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "scc_V2.h"

void vHong(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;

  if(devicecount>1){
    numBlocksKernel = numBlocksKernel/devicecount+1;
  }


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* h_modified;
  h_modified=(int*)malloc(sizeof(int)*(V+1));
  int** d_modified;
  d_modified = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(int)*(V+1));
  }

  int* h_outDeg;
  h_outDeg=(int*)malloc(sizeof(int)*(V+1));
  int** d_outDeg;
  d_outDeg = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_outDeg[i], sizeof(int)*(V+1));
  }

  int* h_inDeg;
  h_inDeg=(int*)malloc(sizeof(int)*(V+1));
  int** d_inDeg;
  d_inDeg = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_inDeg[i], sizeof(int)*(V+1));
  }

  bool* h_visitFw;
  h_visitFw=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_visitFw;
  d_visitFw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitFw[i], sizeof(bool)*(V+1));
  }

  bool* h_visitFw_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_visitFw_temp1;
  hipMalloc(&d_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_visitFw_temp2;
  hipMalloc(&d_visitFw_temp2,(V+1)*(devicecount)*sizeof(bool));


  bool* h_visitBw;
  h_visitBw=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_visitBw;
  d_visitBw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitBw[i], sizeof(bool)*(V+1));
  }

  bool* h_visitBw_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_visitBw_temp1;
  hipMalloc(&d_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_visitBw_temp2;
  hipMalloc(&d_visitBw_temp2,(V+1)*(devicecount)*sizeof(bool));


  bool* h_propFw;
  h_propFw=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_propFw;
  d_propFw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_propFw[i], sizeof(bool)*(V+1));
  }

  bool* h_propBw;
  h_propBw=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_propBw;
  d_propBw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_propBw[i], sizeof(bool)*(V+1));
  }

  bool* h_isPivot;
  h_isPivot=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_isPivot;
  d_isPivot = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_isPivot[i], sizeof(bool)*(V+1));
  }

  int* h_scc;
  h_scc=(int*)malloc(sizeof(int)*(V+1));
  int** d_scc;
  d_scc = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_scc[i], sizeof(int)*(V+1));
  }

  int* h_range;
  h_range=(int*)malloc(sizeof(int)*(V+1));
  int** d_range;
  d_range = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_range[i], sizeof(int)*(V+1));
  }

  int* h_pivotField;
  h_pivotField=(int*)malloc(sizeof(int)*(V+1));
  int** d_pivotField;
  d_pivotField = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pivotField[i], sizeof(int)*(V+1));
  }

  int* h_pivotField_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_pivotField_temp1;
  hipMalloc(&d_pivotField_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_pivotField_temp2;
  hipMalloc(&d_pivotField_temp2,(V+1)*(devicecount)*sizeof(int));


  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(int)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified+h_vertex_partition[i],d_modified[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_outDeg[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_outDeg+h_vertex_partition[i],d_outDeg[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_inDeg[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_inDeg+h_vertex_partition[i],d_inDeg[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitFw+h_vertex_partition[i],d_visitFw[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitBw+h_vertex_partition[i],d_visitBw[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propFw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_propFw+h_vertex_partition[i],d_propFw[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propBw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_propBw+h_vertex_partition[i],d_propBw[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_isPivot[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_isPivot+h_vertex_partition[i],d_isPivot[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_scc+h_vertex_partition[i],d_scc[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_pivotField[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_pivotField+h_vertex_partition[i],d_pivotField[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_inDeg[i],d_outDeg[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_inDeg+h_vertex_partition[i],d_inDeg[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_inDeg[i],h_inDeg,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_outDeg+h_vertex_partition[i],d_outDeg[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_outDeg[i],h_outDeg,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  bool fpoint1 = false; // asst in .cu 
  bool** h_fpoint1;
  h_fpoint1 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint1[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint1;
  d_fpoint1 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  int** d_modified_next;
  d_modified_next = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(int)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint1[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel2<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint1[i],d_isPivot[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_scc+h_vertex_partition[i],d_scc[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_scc[i],h_scc,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_isPivot+h_vertex_partition[i],d_isPivot[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_isPivot[i],h_isPivot,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel3<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_outDeg[i],d_inDeg[i],d_pivotField[i],d_range[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }




  if(devicecount>1){
    //src index
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_pivotField_temp1+i*(V+1),d_pivotField[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_pivotField_temp1,h_pivotField_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_pivotField_temp2+i*(V+1),h_pivotField,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_pivotField_temp1,d_pivotField_temp2,V,devicecount);
    hipMemcpy(h_pivotField,d_pivotField_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_pivotField[i],h_pivotField,(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel4<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_pivotField[i],d_range[i],d_isPivot[i],d_visitBw[i],d_visitFw[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitFw+h_vertex_partition[i],d_visitFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitFw[i],h_visitFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitBw+h_vertex_partition[i],d_visitBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitBw[i],h_visitBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_isPivot+h_vertex_partition[i],d_isPivot[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_isPivot[i],h_isPivot,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  bool fpoint2 = false; // asst in .cu 
  bool** h_fpoint2;
  h_fpoint2 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint2[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint2;
  d_fpoint2 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint2[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!fpoint2) {

    fpoint2 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint2[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel5<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_visitBw[i],d_propFw[i],d_visitFw[i],d_scc[i],d_propBw[i],d_range[i],d_fpoint2[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    if(devicecount>1){
      //src dst
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
      hipMemcpy(h_visitFw,d_visitFw_temp1,(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitFw[i],h_visitFw,(V+1)*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_propFw+h_vertex_partition[i],d_propFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_propFw[i],h_propFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src par
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
      hipMemcpy(h_visitBw,d_visitBw_temp1,(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitBw[i],h_visitBw,(V+1)*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_propBw+h_vertex_partition[i],d_propBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_propBw[i],h_propBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint2&=h_fpoint2[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel6<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_visitFw[i],d_visitBw[i],d_range[i],d_propBw[i],d_propFw[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }



  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_range[i],h_range,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitFw+h_vertex_partition[i],d_visitFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitFw[i],h_visitFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitBw+h_vertex_partition[i],d_visitBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitBw[i],h_visitBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_propFw+h_vertex_partition[i],d_propFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_propFw[i],h_propFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_propBw+h_vertex_partition[i],d_propBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_propBw[i],h_propBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_scc+h_vertex_partition[i],d_scc[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_scc[i],h_scc,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  fpoint1 = false;
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    //printed here

    initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],(bool)fpoint1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint1[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel7<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint1[i],d_isPivot[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_scc+h_vertex_partition[i],d_scc[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_scc[i],h_scc,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_isPivot+h_vertex_partition[i],d_isPivot[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_isPivot[i],h_isPivot,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel8<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  if(devicecount>1){
    //src src
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_range[i],h_range,sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  bool fpoint4 = false; // asst in .cu 
  bool** h_fpoint4;
  h_fpoint4 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint4[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint4;
  d_fpoint4 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint4[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint4[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!fpoint4) {

    fpoint4 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint4[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel9<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint4[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_range[i],h_range,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel10<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_range[i],d_fpoint4[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_range[i],h_range,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint4[i], d_fpoint4[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint4&=h_fpoint4[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint4[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  bool fpoint5 = false; // asst in .cu 
  bool** h_fpoint5;
  h_fpoint5 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint5[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint5;
  d_fpoint5 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint5[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint5[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!fpoint5) {

    fpoint5 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint5[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel11<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_outDeg[i],d_inDeg[i],d_pivotField[i],d_range[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }




    if(devicecount>1){
      //src index
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_pivotField_temp1+i*(V+1),d_pivotField[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_pivotField_temp1,h_pivotField_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_pivotField_temp2+i*(V+1),h_pivotField,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_pivotField_temp1,d_pivotField_temp2,V,devicecount);
      hipMemcpy(h_pivotField,d_pivotField_temp1,(V+1)*sizeof(int),hipMemcpyDeviceToHost);
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_pivotField[i],h_pivotField,(V+1)*sizeof(int),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel12<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_pivotField[i],d_range[i],d_isPivot[i],d_visitBw[i],d_visitFw[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitFw+h_vertex_partition[i],d_visitFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitFw[i],h_visitFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitBw+h_vertex_partition[i],d_visitBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitBw[i],h_visitBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_isPivot+h_vertex_partition[i],d_isPivot[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_isPivot[i],h_isPivot,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    fpoint2 = false;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      //printed here

      initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],(bool)fpoint2);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    while(!fpoint2) {

      fpoint2 = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_fpoint2[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        vHong_kernel13<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_visitBw[i],d_propFw[i],d_visitFw[i],d_scc[i],d_propBw[i],d_range[i],d_fpoint2[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }

      if(devicecount>1){
        //src dst
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
        hipMemcpy(h_visitFw,d_visitFw_temp1,(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_visitFw[i],h_visitFw,(V+1)*sizeof(bool),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //src src
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_propFw+h_vertex_partition[i],d_propFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_propFw[i],h_propFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //src par
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        hipSetDevice(0);
        hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
        for(int i=0;i<devicecount;i++){
          hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
        hipMemcpy(h_visitBw,d_visitBw_temp1,(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_visitBw[i],h_visitBw,(V+1)*sizeof(bool),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      if(devicecount>1){
        //src src
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_propBw+h_vertex_partition[i],d_propBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_propBw[i],h_propBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        fpoint2&=h_fpoint2[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel14<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_visitFw[i],d_visitBw[i],d_range[i],d_propBw[i],d_propFw[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }



    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_range+h_vertex_partition[i],d_range[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_range[i],h_range,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitFw+h_vertex_partition[i],d_visitFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitFw[i],h_visitFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitBw+h_vertex_partition[i],d_visitBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitBw[i],h_visitBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_propFw+h_vertex_partition[i],d_propFw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_propFw[i],h_propFw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_propBw+h_vertex_partition[i],d_propBw[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_propBw[i],h_propBw,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    if(devicecount>1){
      //src src
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_scc+h_vertex_partition[i],d_scc[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_scc[i],h_scc,sizeof(int)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint5[i], d_fpoint5[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint5&=h_fpoint5[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint5[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  int cnt = 0;
  for(int i=0;i<=V;i+=1){
    if(h_isPivot[0][i]==true){
      cnt+=1;
    }
  }
  printf("SCC val %d device count %d\n",cnt,devicecount);


} //end FUN


int main(int argc,char* argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  vHong(g);
  return 0;
}
