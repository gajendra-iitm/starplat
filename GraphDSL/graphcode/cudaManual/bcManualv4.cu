#include "hip/hip_runtime.h"
//~ nvcc bcManualv4.cu -o bcManualv4-sm60.out -arch=sm_60 -I.

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "graph.hpp"

#include <random>
#include <sstream>
#include <fstream>
#include <string>
#include <string>

#define cudaCheckError() {                                             \
 hipError_t e=hipGetLastError();                                     \
 if(e!=hipSuccess) {                                                  \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
   exit(0);                                                            \
 }                                                                     \
}


template <typename T>
__global__ void initKernel(unsigned nSize,T* dArray,T initVal){
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < nSize){ 
    dArray[id]=initVal;
  }
}

__global__ void incHop(int* d_hops_from_source) {
    *d_hops_from_source = *d_hops_from_source + 1;
  }

  __global__ void bc_forward_pass(int* d_offset,int* d_edgeList,int* d_edgeLen, double* d_sigma, int* d_level, int* d_hops_from_source, unsigned n, bool* d_finished) {
    unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
    if(v >= n) return;

    // only processing the nodes at level '*d_hops_from_source' -- a level synchronous processing, though not work efficient
    if(d_level[v] == *d_hops_from_source) {
       unsigned end = d_offset[v+1];
       for(unsigned i = d_offset[v]; i < end; ++i) { // going over the neighbors of u
          unsigned w = d_edgeList[i];
          if(d_level[w] == -1) {  // v is seen for the first time
            d_level[w] = *d_hops_from_source + 1; // no atomics required since this is benign data race due to level synchronous implementation
            *d_finished = false;
          }
          if(d_level[w] == *d_hops_from_source + 1) { // 'v' is indeed the neighbor of u
            atomicAdd(&d_sigma[w], d_sigma[v]);
          }
       }
    }
  }


  __global__ void bc_backward_pass(int* d_offset,int* d_edgeList,int* d_edgeLen, double* d_sigma, double* d_delta, double* d_nodeBC, int* d_level, int* d_hops_from_source, unsigned n) {
    // TO replace - DONE
    // u --> v
    // v --> w
    unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
    if(v >= n) return;

    if(d_level[v] == *d_hops_from_source - 1) { // backward traversal of DAG, one level at a time

       unsigned end = d_offset[v+1];
       double sum = 0.0;
       for(unsigned i = d_offset[v]; i < end; ++i) { // going over the neighbors of u for which it is the predecessor in the DAG
          unsigned w = d_edgeList[i];
          if(d_level[w] == *d_hops_from_source) {
            //~ if(std::isnan(d_sigma[w])
            //~ assert(d_sigma[w]!=0);
            sum += (1.0 * d_sigma[v]) / d_sigma[w] * (1.0 + d_delta[w]);
          }
       }

       d_delta[v] += sum;
    }

  }

  __global__ void accumulate_bc(double * d_delta, double* d_nodeBC, int* d_level, unsigned s, unsigned n) {

    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n || tid == s || d_level[tid] == -1) return;


    d_nodeBC[tid] += d_delta[tid]/2.0;

  }


__global__ void initialize(double* d_sigma, double* d_delta, int* d_level, int* d_hops_from_source, unsigned s, unsigned n) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) {
      d_level[tid] = -1;
      d_delta[tid] = 0.0;
      d_sigma[tid] = 0;

      if(tid == s) { // for the source
        d_level[tid] = 0;
        d_sigma[tid] = 1;
        *d_hops_from_source = 0;
      }
    }
  }


void BC(int * OA , int * edgeList , int* edgeLength, int V, int E, std::set<unsigned> sourceSet, bool printAns=false) {

  int* d_offset;
  int* d_edgeList;
  int* d_edgeLen;  //why this for unweighted?

  // VAR for BC
  // G variables
  hipMalloc(&d_offset,sizeof(int) *(1+V));
  hipMalloc(&d_edgeList,sizeof(int) *(E));
  hipMalloc(&d_edgeLen,sizeof(int) *(E));
  // G's Memcpy
  hipMemcpy (d_offset, OA, sizeof(int) *(1+V) ,hipMemcpyHostToDevice);
  hipMemcpy (d_edgeList, edgeList, sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (d_edgeLen, edgeLength , sizeof(int) *(E) ,hipMemcpyHostToDevice);


  // Vars from DSL body
  double* d_sigma; hipMalloc(&d_sigma,  sizeof(double) * V);
  double* d_delta ;  hipMalloc(&d_delta,  sizeof(double)   * V);

  // Vars from DSL func param and output var
  double* d_nodeBC;  hipMalloc(&d_nodeBC, sizeof(double)   * V);

  // extra D vars
  bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
  int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));
  int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

  // extra H vars
  bool finished = false;
  int hops_from_source;

  //DEBUG ONLY
  std::cout<< "srcSet Size:"<< sourceSet.size() << '\n';

  hipEvent_t start, stop; ///TIMER START
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  // FOR LAUNCH CONFIG - V
  unsigned int block_size = V;
  unsigned int num_blocks = 1;
  if(V > 1024){
    block_size = 1024;                        // at some point when sh-mem comes in, it should be 512
    num_blocks = (V+block_size-1)/block_size; // modified without ceil fun call
  }

  for(auto src:sourceSet) {

    hops_from_source = 0; // keeps track of the number of hops from source in the current iteration.
    initialize<<<num_blocks, block_size>>>(d_sigma, d_delta, d_level, d_hops_from_source, src, V);
    //DEBUG
    long k=0;
    //FORWARD PASS
    do{

      finished=true;
      hipMemcpy(d_finished,&finished, sizeof(bool) ,hipMemcpyHostToDevice);

      //~ hipMemset(d_finished,true,sizeof(bool)); DONOT use MEMSET --rupesh

      bc_forward_pass<<<num_blocks, block_size>>>(d_offset, d_edgeList,d_edgeLen, d_sigma, d_level, d_hops_from_source, V, d_finished);

      hipDeviceSynchronize(); //MUST - rupesh

      ++hops_from_source; // updating the level to process in the next iteration

      incHop<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST -rupesh //this 2nd CDS may be removed because of following D2H
      //DEBUG
      k++;
      hipMemcpy(&finished,d_finished, sizeof(bool) ,hipMemcpyDeviceToHost);
      //~ std::cout<< "SRC:"<< src <<" Fin? "<< (finished?"True":"False") << '\n';
    }while(!finished);


    //DEBUG
    /*
    hipMemcpy(level,d_level , sizeof(int) * (V), hipMemcpyDeviceToHost);
    std::cout<< "SRC:"<< src << " iters:" << k << " Hops:"<<hops_from_source<< '\n';
    for (int i = 0; i <V; i++)
      printf("%d %d\n", i, level[i]);
    return;
    */

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {
      bc_backward_pass<<<num_blocks, block_size>>>(d_offset, d_edgeList,d_edgeLen, d_sigma, d_delta, d_nodeBC, d_level, d_hops_from_source, V);
      --hops_from_source;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice);
    }

    accumulate_bc<<<num_blocks, block_size>>>(d_delta, d_nodeBC, d_level, src, V);
    hipDeviceSynchronize();
  }// END FOR



  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  //~ printf("GPU Time: %.6f ms \nIterations:%d\n", milliseconds);

  if(printAns){

  //For printing result variable
  double* nodeBC;
  nodeBC = (double *)malloc( (V)*sizeof(double));
  hipMemcpy(nodeBC,d_nodeBC , sizeof(double) * (V), hipMemcpyDeviceToHost);

   for (int i = 0; i <9; i++)
     printf("%d %lf\n", i, nodeBC[i]);
  }

  //cudaCheckError();


  //~ char *outputfilename = "output_generated.txt";
  //~ FILE *outputfilepointer;
  //~ outputfilepointer = fopen(outputfilename, "w");
  //~ for (int i = 0; i <V; i++)
  //~ {
    //~ fprintf(outputfilepointer, "%d  %d\n", i, dist[i]);
  //~ }
  //~ Let's add fclose!

}


// driver program to test above function
int main(int argc , char ** argv)
{
  graph G(argv[1]);
  G.parseGraph();
  bool printAns = false;

  std::set<unsigned> src;

  if(argc>3) { // ./a.out inputfile srcFile -p
      printAns = true;
  }

  // Check and READ Src file =================

  std::string line;
  std::ifstream srcfile(argv[2]);
  if (!srcfile.is_open()) {
    std::cout << "Unable to open src file :" << argv[1] << std::endl;
    exit(1);
  }
  

  int nodeVal;
  while ( std::getline (srcfile,line) ) {
   std::stringstream ss(line);
   ss>> nodeVal;
   //~ std::cout << "src " << nodeVal << '\n';
   src.insert(nodeVal);
  }

  srcfile.close();
  //==========================================

  //---------------------------------------//
  int V = G.num_nodes();
  int E = G.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  //-------------------------------------//

  int* edgeLen = G.getEdgeLen();

  int *OA;
  int *edgeList;
  int *edgeLength;

   OA = (int *)malloc( (V+1)*sizeof(int));
   edgeList = (int *)malloc( (E)*sizeof(int));
   edgeLength = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = G.indexofNodes[i];
    OA[i] = temp;
  }
  //~ for(int i=1; i<=V;++i){
    //~ std::cout<< i << " " << OA[i]-OA[i-1] << '\n';
  //~ }
  for(int i=0; i< E; i++) {
    int temp = G.edgeList[i];
    edgeList[i] = temp;
    temp = edgeLen[i];
    edgeLength[i] = temp;
  }


    //~ hipEvent_t start, stop; // should not be here!
    //~ hipEventCreate(&start);
    //~ hipEventCreate(&stop);
    //~ float milliseconds = 0;
    //~ hipEventRecord(start,0);

    BC(OA,edgeList, edgeLength, V,E,src, printAns);
    hipDeviceSynchronize();

    //~ hipEventRecord(stop,0);
    //~ hipEventSynchronize(stop);
    //~ hipEventElapsedTime(&milliseconds, start, stop);
    //~ printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    cudaCheckError();

  return 0;

}
