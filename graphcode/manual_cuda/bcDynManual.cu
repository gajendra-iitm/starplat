#include "hip/hip_runtime.h"
//header flies
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#ifndef GENCPP_BC_DSLV2_H
#define GENCPP_BC_DSLV2_H
#include <hip/hip_cooperative_groups.h>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include "../graph.hpp"
//#include "../libcuda.cuh"
#include "../dynamic_mst_delete_cuda/libcuda.cuh"

//static bc kernels
//--------------------------------------------------------------------------------------------//
__global__ void fwd_pass(int n, int* d_meta,int* d_data,int* d_weight, float* d_delta, float* d_sigma, int* d_level, int* d_hops_from_source, bool* d_finished,float* d_BC) {
  unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
  if(v >= n) return;
  if(d_level[v] == *d_hops_from_source) {
    for (int edge = d_meta[v]; edge < d_meta[v+1]; edge++) { // FOR NBR ITR 
      int w = d_data[edge];
      if(d_level[w] == -1) {
        d_level[w] = *d_hops_from_source + 1;
        *d_finished = false;
      }
      if(d_level[w] == *d_hops_from_source + 1) {
        d_sigma[w] = d_sigma[w]+ d_sigma[v];

      }
    }
  } // end if d lvl
} // kernel end

__global__ void back_pass(int n, int* d_meta,int* d_data,int* d_weight, float* d_delta, float* d_sigma, int* d_level, int* d_hops_from_source, bool* d_finished,float* d_BC) {
  unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
  if(v >= n) return;
  auto grid = cooperative_groups::this_grid();
  if(d_level[v] == *d_hops_from_source-1) {
    for (int edge = d_meta[v]; edge < d_meta[v+1]; edge++) { // FOR NBR ITR 
      int w = d_data[edge];
      if(d_level[w] == *d_hops_from_source) {
        atomicAdd(&d_delta[v],  (d_sigma[v] / d_sigma[w]) * (1 + d_delta[w]));

      } // end IF  
    } // end FOR
  } // end if d lvl
} // kernel end

__global__ void Compute_BC_kernel(int V, int E,float* d_delta,float* d_BC){ // BEGIN KER FUN via ADDKERNEL
  float num_nodes  = V;
  unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
  if(v >= V) return;
  d_BC[v] = d_BC[v] + d_delta[v];
} // end KER FUNC

#endif


//here computeBcDyn() fn that makes use of above static bc kernels
//compute_bc uses static bc kernels
//--------------------------------------------------------------------------------------------//
void Compute_BC(graph& g,float* BC,std::set<int>& sourceSet)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_BC;
  hipMalloc(&d_BC, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_BC,(float)0);

  float* d_sigma;
  hipMalloc(&d_sigma, sizeof(float)*(V));

  float* d_delta;
  hipMalloc(&d_delta, sizeof(float)*(V));

  //FOR SIGNATURE of SET - Assumes set for on .cu only
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++) 
  {
    int src = *itr;
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_delta,(float)0);

    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_sigma,(float)0);

    initIndex<float><<<1,1>>>(V,d_sigma,src,(float)1); //InitIndexDevice

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;
    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));  hipMemset(d_hops_from_source,0,sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    //EXTRA vars INITIALIZATION
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_level,-1);
    initIndex<int><<<1,1>>>(V,d_level,src, 0);

    // long k =0 ;// For DEBUG
    do {
      finished = true;
      hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished,d_BC); ///DONE from varList

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST - rupesh
      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    }while(!finished);

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished
        ,d_BC); ///DONE from varList

      hops_from_source--;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
    }
    //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);
    Compute_BC_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_delta,d_BC);
    hipDeviceSynchronize();




  }

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_delta);
  hipFree(d_sigma);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(      BC,     d_BC, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN



// main fn () bc_dslV2mainCuda
//--------------------------------------------------------------------------------------------//
// main fn: reads Input and Calls `Compute_BC`
int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <graph_file> <source_nodes_file>" << std::endl;
        return 1;
    }

    char* graphFilePath = argv[1];
    char* sourceNodesFilePath = argv[2];

    // Load graph
    graph g(graphFilePath);
    g.parseGraph();

    std::cout << "Number of nodes: " << g.num_nodes() << std::endl;
    std::cout << "Number of edges: " << g.num_edges() << std::endl;

    // Read source nodes from file
    std::set<int> sourceSet;
    std::ifstream sourceFile(sourceNodesFilePath);
    if (!sourceFile) {
        std::cerr << "Error: Unable to open source nodes file: " << sourceNodesFilePath << std::endl;
        return 1;
    }

    int node;
    while (sourceFile >> node) {
        if (node >= 0 && node < g.num_nodes()) {
            sourceSet.insert(node);
        }
    }
    sourceFile.close();

    std::cout << "Source nodes loaded: ";
    for (int src : sourceSet) {
      std::cout << src << " ";
    }
    std::cout << std::endl;



    float* BC = new float[g.num_nodes()];
    std::fill_n(BC, g.num_nodes(), 0.0f);//to set the array BC values to 0

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//to list out all the edges of each node in the graph.
    for (int i = 0; i < g.num_nodes(); ++i) {
      std::cout << "Node " << i << " has edges: ";
        for (int j = g.indexofNodes[i]; j < g.indexofNodes[i + 1]; j++) {
          std::cout << g.edgeList[j] << " ";
        }
      std::cout << std::endl;//new line after listing out all the edges of the node i.
    }

    hipEventRecord(start, 0);
    printf("Calling Compute_BC now.\n");
    Compute_BC(g, BC, sourceSet);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "\nGPU Time: " << milliseconds / 1000.0 << " seconds" << std::endl;

    for (int i = 0; i < g.num_nodes(); ++i) {
        std::cout << "Node " << i << " BC: " << BC[i] << std::endl;
    }

    delete[] BC;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}





//--------------------------------------------------------------------------------------------//
//input graph handling-- most probably thru main fn only
//input updates incremental, decremental-- same file or different files? what does these update files has
//using sssp dyn main () fn code as reference
//main() fn
//--------------------------------------------------------------------------------------------//
// int main(int argc, char **argv)
// {
//   //take input graph file
//   graph G("../inputGraphs/inputs/pokec.txt", "cuda", true);
//   G.parseGraph();

//   int *dist = new int[G.num_nodes()];
//   int *parent = new int[G.num_nodes()];
//   int *edgeLen = G.getEdgeLen();
//   //compute_bc here
//   //Compute_SSSP(G, dist, parent, edgeLen, 0);

//   //boolean array to keep track of added and deleted edges (through nodes)  
//   bool *modified_add = new bool[G.num_nodes()];
//   bool *modified_del = new bool[G.num_nodes()];
//   int elements = 0;

//   std::vector<update> updateEdges = G.parseUpdates("../inputGraphs/updates/update_pokec.txt");

//   int percent = atoi(argv[1])
//   int updateSize = percent * 0.01 * G.num_edges();
//   if (!G.isGraphDirected() && updateSize & 1)
//     updateSize++;
//   int batchSize = updateSize;
//   // int batchSize = 1000000;

//   std::chrono::time_point<std::chrono::high_resolution_clock> startTime, endTime;
//   std::chrono::duration<double, std::milli> timeTaken;

//   startTime = std::chrono::high_resolution_clock::now();
//   for (int k = 0; k < updateSize; k += batchSize)
//   {
//     if ((k + batchSize) > updateSize)
//       elements = updateSize - k;
//     else
//       elements = batchSize;

//     for (int i = 0; i < G.num_nodes(); i++)
//     {
//       modified_add[i] = false;
//       modified_del[i] = false;
//     }

//     for (int i = k; i < (k + batchSize) && i < updateSize; i++)
//     {
//       if (updateEdges[i].type == 'd')
//       {
//         update u = updateEdges[i];
//         int src = u.source;
//         int dest = u.destination;

//         if (parent[dest] >= 0 && parent[dest] == src)
//         {
//           dist[dest] = INT_MAX;
//           parent[dest] = -1;
//           modified_del[dest] = true;
//         }
//       }
//     }

//     //update graph CSR with updated deleted edges
//     G.updateCSR_Del(updateEdges, k, elements);
//     //compute_dynBcDel() here
//     Compute_dynamicSSSPDel(G, dist, parent, modified_del);

//     for (int i = k; i < (k + batchSize) && i < updateSize; i++)
//     {
//       if (updateEdges[i].type == 'a')
//       {
//         update u = updateEdges[i];
//         int src = u.source;
//         int dest = u.destination;
//         int weight = u.weight;
//         if (dist[src] != INT_MAX && dist[src] + weight < dist[dest])
//           modified_add[src] = true;
//       }
//     }

//     //update graph CSR with added edges
//     G.updateCSR_Add(updateEdges, k, elements);
//     //compute_dynBcAdd()
//     Compute_dynamicSSSPAdd(G, dist, parent, modified_add);
//   }
//   endTime = std::chrono::high_resolution_clock::now();
//   timeTaken = endTime - startTime;
//   printf("Total time taken for %d percent updates: %.3f ms\n", percent, timeTaken.count());

//   unsigned long long sum = 0;
//   for (int i = 0; i < G.num_nodes(); i++)
//   {
//     if (dist[i] != INT_MAX)
//       sum += dist[i];
//   }
//   printf("Final graph, total path sum = %llu\n\n", sum);

//   free(dist);
//   free(parent);
//   free(modified_add);
//   free(modified_del);

//   return 0;
// }