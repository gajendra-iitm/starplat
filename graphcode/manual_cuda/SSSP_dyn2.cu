#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include "../graph.hpp"
#include "../libcuda.cuh"

// kernel for computing SSSP of static graph
__global__ void sssp_kernel(int *d_meta, int *d_data, int *d_weight, int *d_dist,
                            int *d_parent, int *d_locks, int V, bool *d_modified,
                            bool *d_modified_next, bool *d_finished)
{
  unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < V && d_modified[id] == true && d_dist[id] != INT_MAX)
  {
    int distToCurNode = d_dist[id];
    int v, newDist, lock;
    bool gotLock;
    for (int e = d_meta[id]; e < d_meta[id + 1]; e++)
    {
      gotLock = false;
      v = d_data[e];
      newDist = distToCurNode + d_weight[e];
      do
      {
        if (gotLock == false)
          lock = atomicCAS(&d_locks[v], 0, 1);
        if (lock == 0 && newDist < d_dist[v])
        {
          d_dist[v] = newDist;
          d_parent[v] = id;
          d_modified_next[v] = true;
          *d_finished = false;
        }
        if (lock == 0)
        {
          gotLock = true;
          lock = 1;
          atomicExch(&d_locks[v], 0);
        }
      } while (gotLock == false);
    }
  }
}

// kernel for marking all the descendants of modified nodes in SPT
__global__ void mark_descendants(int *d_dist, int *d_parent, bool *d_modified, int V,
                                 bool *d_finished)
{
  unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < V && d_modified[id] == false)
  {
    int parent = d_parent[id];
    if (parent >= 0 && d_modified[parent] == true)
    {
      d_modified[id] = true;
      *d_finished = false;
    }
  }
}

// kernel for setting distance and parent of marked nodes
__global__ void mark_not_reachable(int *d_dist, int *d_parent, int V, bool *d_modified)
{
  unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < V && d_modified[id] == true)
  {
    d_dist[id] = INT_MAX;
    d_parent[id] = -1;
  }
}

// kernel for updating the distance and parent of marked nodes (as a result of deletions)
__global__ void fetch_and_update(int *d_metaR, int *d_dataR, int *d_weightR,
                                 int *d_diffMetaR, int *d_diffDataR, int *d_diffWeightR,
                                 int *d_dist, int *d_parent, int V,
                                 bool *d_modified, bool *d_finished)
{
  unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < V && d_modified[id] == true)
  {
    int u;
    for (int e = d_metaR[id]; e < d_metaR[id + 1]; e++)
    {
      u = d_dataR[e];
      if (u != INT_MAX && d_dist[u] != INT_MAX)
      {
        if (d_dist[id] > d_dist[u] + d_weightR[e])
        {
          d_dist[id] = d_dist[u] + d_weightR[e];
          d_parent[id] = u;
          *d_finished = false;
        }
      }
    }
    for (int e = d_diffMetaR[id]; e < d_diffMetaR[id + 1]; e++)
    {
      u = d_diffDataR[e];
      if (u != INT_MAX && d_dist[u] != INT_MAX)
      {
        if (d_dist[id] > d_dist[u] + d_diffWeightR[e])
        {
          d_dist[id] = d_dist[u] + d_diffWeightR[e];
          d_parent[id] = u;
          *d_finished = false;
        }
      }
    }
  }
}

// kernel for updating the distance and parent of marked nodes (as a result of additions)
__global__ void push_and_update(int *d_meta, int *d_data, int *d_weight, int *d_dist, int *d_parent,
                                int *d_diffMeta, int *d_diffData, int *d_diffWeight,
                                int *d_locks, int V, bool *d_modified, bool *d_finished)
{
  unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < V && d_modified[id] == true && d_dist[id] != INT_MAX)
  {
    int distToCurNode = d_dist[id];
    int v, newDist, lock;
    bool gotLock;
    for (int e = d_meta[id]; e < d_meta[id + 1]; e++)
    {
      if (d_data[e] != INT_MAX)
      {
        gotLock = false;
        v = d_data[e];
        newDist = distToCurNode + d_weight[e];
        do
        {
          if (gotLock == false)
            lock = atomicCAS(&d_locks[v], 0, 1);
          if (lock == 0 && newDist < d_dist[v])
          {
            d_dist[v] = newDist;
            d_parent[v] = id;
            d_modified[v] = true;
            *d_finished = false;
          }
          if (lock == 0)
          {
            gotLock = true;
            lock = 1;
            atomicExch(&d_locks[v], 0);
          }
        } while (gotLock == false);
      }
    }
    for (int e = d_diffMeta[id]; e < d_diffMeta[id + 1]; e++)
    {
      if (d_diffData[e] != INT_MAX)
      {
        gotLock = false;
        v = d_diffData[e];
        newDist = distToCurNode + d_diffWeight[e];
        do
        {
          if (gotLock == false)
            lock = atomicCAS(&d_locks[v], 0, 1);
          if (lock == 0 && newDist < d_dist[v])
          {
            d_dist[v] = newDist;
            d_parent[v] = id;
            d_modified[v] = true;
            *d_finished = false;
          }
          if (lock == 0)
          {
            gotLock = true;
            lock = 1;
            atomicExch(&d_locks[v], 0);
          }
        } while (gotLock == false);
      }
    }
  }
}

// check for hipError_t
void checkCudaError()
{
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }
}

void Compute_SSSP(graph &g, int *dist, int *parent, int *weight, int src)
{
  // data on host
  int V = g.num_nodes();
  int E = g.num_edges();
  bool finished;
  int *h_meta = g.indexofNodes;
  int *h_data = g.edgeList;
  bool *h_modified = new bool[V];

  // data on device
  int *d_meta, *d_data, *d_weight;
  int *d_dist, *d_parent, *d_locks;
  bool *d_finished, *d_modified, *d_modified_next;

  // for recording the total time taken
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // initialize host data
  for (int i = 0; i < V; i++)
  {
    dist[i] = INT_MAX;
    parent[i] = -1;
    h_modified[i] = false;
  }
  dist[src] = 0;
  h_modified[src] = true;
  finished = false;

  // allocate memory on device
  hipMalloc(&d_meta, sizeof(int) * (V + 1));
  hipMalloc(&d_data, sizeof(int) * (E));
  hipMalloc(&d_weight, sizeof(int) * (E));
  hipMalloc(&d_dist, sizeof(int) * V);
  hipMalloc(&d_parent, sizeof(int) * V);
  hipMalloc(&d_locks, sizeof(int) * V);
  hipMalloc(&d_finished, sizeof(bool));
  hipMalloc(&d_modified, sizeof(bool) * V);
  hipMalloc(&d_modified_next, sizeof(bool) * V);

  // copy to device
  hipMemcpy(d_meta, h_meta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_data, h_data, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, weight, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_dist, dist, sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(d_parent, parent, sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(d_modified, h_modified, sizeof(bool) * (V), hipMemcpyHostToDevice);

  // launch config
  const int numThreads = 1024;
  const int numBlocks = (V + numThreads - 1) / numThreads;

  // call kernel to compute the SSSP
  int iter = 0;
  init_kernel<bool><<<numBlocks, numThreads>>>(d_modified_next, false, V);
  init_kernel<int><<<numBlocks, numThreads>>>(d_locks, 0, V);
  while (finished != true)
  {
    init_kernel<bool><<<1, 1>>>(d_finished, true, 1);
    sssp_kernel<<<numBlocks, numThreads>>>(d_meta, d_data, d_weight, d_dist, d_parent,
                                           d_locks, V, d_modified, d_modified_next, d_finished);
    init_kernel<bool><<<numBlocks, numThreads>>>(d_modified, false, V);
    hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);

    bool *tempPtr = d_modified_next;
    d_modified_next = d_modified;
    d_modified = tempPtr;

    if (++iter >= V - 1)
      break;
  }
  hipDeviceSynchronize();
  checkCudaError();

  // copy distances and parents back to host
  hipMemcpy(dist, d_dist, sizeof(int) * (V), hipMemcpyDeviceToHost);
  hipMemcpy(parent, d_parent, sizeof(int) * (V), hipMemcpyDeviceToHost);

  // print time taken
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Initial graph, SSSP using GPU - time taken: %.3f ms \n", milliseconds);

  // free up the memory
  free(h_modified);
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_weight);
  hipFree(d_dist);
  hipFree(d_parent);
  hipFree(d_locks);
  hipFree(d_finished);
  hipFree(d_modified);
  hipFree(d_modified_next);

  unsigned long long sum = 0;
  for (int i = 0; i < V; i++)
  {
    if (dist[i] != INT_MAX)
      sum += dist[i];
  }
  printf("Initial graph, total path sum = %llu\n", sum);
}

void Compute_dynamicSSSPDel(graph &g, int *dist, int *parent, bool *modified)
{
  // data on host
  int V = g.num_nodes();
  int E = g.num_edges_CSR();
  int diffE1 = g.num_edges_diffCSR();
  int diffE2 = g.num_edges_diffCSC();
  bool finished;
  int *h_metaR = g.rev_indexofNodes;
  int *h_dataR = g.srcList;
  int *h_weightR = g.getRev_edgeLen();
  int *h_diffMetaR = g.diff_rev_indexofNodes;
  int *h_diffDataR = g.diff_rev_edgeList;
  int *h_diffWeightR = g.getDiff_rev_edgeLen();

  // data on device
  int *d_metaR, *d_dataR, *d_weightR;
  int *d_diffMetaR, *d_diffDataR, *d_diffWeightR;
  int *d_dist, *d_parent;
  bool *d_finished, *d_modified;

  // for recording the total time taken
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // initialize host data
  finished = false;

  // allocate memory on device
  hipMalloc(&d_metaR, sizeof(int) * (V + 1));
  hipMalloc(&d_dataR, sizeof(int) * (E));
  hipMalloc(&d_weightR, sizeof(int) * (E));
  hipMalloc(&d_diffMetaR, sizeof(int) * (V + 1));
  hipMalloc(&d_diffDataR, sizeof(int) * (diffE2));
  hipMalloc(&d_diffWeightR, sizeof(int) * (diffE2));
  hipMalloc(&d_dist, sizeof(int) * V);
  hipMalloc(&d_parent, sizeof(int) * V);
  hipMalloc(&d_finished, sizeof(bool));
  hipMalloc(&d_modified, sizeof(bool) * V);

  // copy to device
  hipMemcpy(d_metaR, h_metaR, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_dataR, h_dataR, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_weightR, h_weightR, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_diffMetaR, h_diffMetaR, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_diffDataR, h_diffDataR, sizeof(int) * (diffE2), hipMemcpyHostToDevice);
  hipMemcpy(d_diffWeightR, h_diffWeightR, sizeof(int) * (diffE2), hipMemcpyHostToDevice);
  hipMemcpy(d_dist, dist, sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(d_parent, parent, sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(d_modified, modified, sizeof(bool) * (V), hipMemcpyHostToDevice);

  // launch config
  const int numThreads = 1024;
  const int numBlocks = (V + numThreads - 1) / numThreads;

  // mark the descendants of modified nodes as modified
  int iter = 0;
  finished = false;
  while (finished != true)
  {
    init_kernel<bool><<<1, 1>>>(d_finished, true, 1);
    mark_descendants<<<numBlocks, numThreads>>>(d_dist, d_parent, d_modified,
                                                V, d_finished);
    hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);
    if (++iter >= V - 1)
      break;
  }
  hipDeviceSynchronize();
  checkCudaError();

  // set the distance and parent of marked nodes (INT_MAX, -1)
  mark_not_reachable<<<numBlocks, numThreads>>>(d_dist, d_parent, V, d_modified);
  hipDeviceSynchronize();
  checkCudaError();

  // update the distances and parents (pull based approach)
  iter = 0;
  finished = false;
  while (finished != true)
  {
    init_kernel<bool><<<1, 1>>>(d_finished, true, 1);
    fetch_and_update<<<numBlocks, numThreads>>>(d_metaR, d_dataR, d_weightR,
                                                d_diffMetaR, d_diffDataR, d_diffWeightR,
                                                d_dist, d_parent, V, d_modified, d_finished);
    hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);
    if (++iter >= V - 1)
      break;
  }
  hipDeviceSynchronize();
  checkCudaError();

  // copy distances and parents back to host
  hipMemcpy(dist, d_dist, sizeof(int) * (V), hipMemcpyDeviceToHost);
  hipMemcpy(parent, d_parent, sizeof(int) * (V), hipMemcpyDeviceToHost);

  // print time taken
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Compute DynSSSP-Del - time taken: %.3f ms \n", milliseconds);

  // free up the memory
  hipFree(d_metaR);
  hipFree(d_dataR);
  hipFree(d_weightR);
  hipFree(d_diffMetaR);
  hipFree(d_diffDataR);
  hipFree(d_diffWeightR);
  hipFree(d_dist);
  hipFree(d_parent);
  hipFree(d_finished);
  hipFree(d_modified);
}

void Compute_dynamicSSSPAdd(graph &g, int *dist, int *parent, bool *modified)
{
  // data on host
  int V = g.num_nodes();
  int E = g.num_edges_CSR();
  int diffE1 = g.num_edges_diffCSR();
  int diffE2 = g.num_edges_diffCSC();
  bool finished;
  int *h_meta = g.indexofNodes;
  int *h_data = g.edgeList;
  int *h_weight = g.getEdgeLen();
  int *h_diffMeta = g.diff_indexofNodes;
  int *h_diffData = g.diff_edgeList;
  int *h_diffWeight = g.getDiff_edgeLen();

  // data on device
  int *d_meta, *d_data, *d_weight;
  int *d_diffMeta, *d_diffData, *d_diffWeight;
  int *d_dist, *d_parent, *d_locks;
  bool *d_finished, *d_modified;

  // for recording the total time taken
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // initialize host data
  finished = false;

  // allocate memory on device
  hipMalloc(&d_meta, sizeof(int) * (V + 1));
  hipMalloc(&d_data, sizeof(int) * (E));
  hipMalloc(&d_weight, sizeof(int) * (E));
  hipMalloc(&d_diffMeta, sizeof(int) * (V + 1));
  hipMalloc(&d_diffData, sizeof(int) * (diffE1));
  hipMalloc(&d_diffWeight, sizeof(int) * (diffE1));
  hipMalloc(&d_dist, sizeof(int) * V);
  hipMalloc(&d_parent, sizeof(int) * V);
  hipMalloc(&d_locks, sizeof(int) * V);
  hipMalloc(&d_finished, sizeof(bool));
  hipMalloc(&d_modified, sizeof(bool) * V);

  // copy to device
  hipMemcpy(d_meta, h_meta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_data, h_data, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_diffMeta, h_diffMeta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_diffData, h_diffData, sizeof(int) * (diffE1), hipMemcpyHostToDevice);
  hipMemcpy(d_diffWeight, h_diffWeight, sizeof(int) * (diffE1), hipMemcpyHostToDevice);
  hipMemcpy(d_dist, dist, sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(d_parent, parent, sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(d_modified, modified, sizeof(bool) * (V), hipMemcpyHostToDevice);

  // launch config
  const int numThreads = 1024;
  const int numBlocks = (V + numThreads - 1) / numThreads;

  // update the distances and parents (push based approach)
  int iter = 0;
  init_kernel<int><<<numBlocks, numThreads>>>(d_locks, 0, V);
  while (finished != true)
  {
    init_kernel<bool><<<1, 1>>>(d_finished, true, 1);
    push_and_update<<<numBlocks, numThreads>>>(d_meta, d_data, d_weight, d_dist, d_parent,
                                               d_diffMeta, d_diffData, d_diffWeight,
                                               d_locks, V, d_modified, d_finished);
    hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);

    if (++iter >= V - 1)
      break;
  }
  hipDeviceSynchronize();
  checkCudaError();

  // copy distances and parents back to host
  hipMemcpy(dist, d_dist, sizeof(int) * (V), hipMemcpyDeviceToHost);
  hipMemcpy(parent, d_parent, sizeof(int) * (V), hipMemcpyDeviceToHost);

  // print time taken
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Compute DynSSSP-Add - time taken: %.3f ms \n", milliseconds);

  // free up the memory
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_weight);
  hipFree(d_diffMeta);
  hipFree(d_diffData);
  hipFree(d_diffWeight);
  hipFree(d_dist);
  hipFree(d_parent);
  hipFree(d_locks);
  hipFree(d_finished);
  hipFree(d_modified);
}

int main(int argc, char **argv)
{
  graph G("../inputGraphs/inputs/pokec.txt", "cuda", true);
  G.parseGraph();

  int *dist = new int[G.num_nodes()];
  int *parent = new int[G.num_nodes()];
  int *edgeLen = G.getEdgeLen();
  Compute_SSSP(G, dist, parent, edgeLen, 0);

  bool *modified_add = new bool[G.num_nodes()];
  bool *modified_del = new bool[G.num_nodes()];
  int elements = 0;

  std::vector<update> updateEdges = G.parseUpdates("../inputGraphs/updates/update_pokec.txt");

  int percent = atoi(argv[1]);
  int updateSize = percent * 0.01 * G.num_edges();
  if (!G.isGraphDirected() && updateSize & 1)
    updateSize++;
  int batchSize = updateSize;
  // int batchSize = 1000000;

  std::chrono::time_point<std::chrono::high_resolution_clock> startTime, endTime;
  std::chrono::duration<double, std::milli> timeTaken;

  startTime = std::chrono::high_resolution_clock::now();
  for (int k = 0; k < updateSize; k += batchSize)
  {
    if ((k + batchSize) > updateSize)
      elements = updateSize - k;
    else
      elements = batchSize;

    for (int i = 0; i < G.num_nodes(); i++)
    {
      modified_add[i] = false;
      modified_del[i] = false;
    }

    for (int i = k; i < (k + batchSize) && i < updateSize; i++)
    {
      if (updateEdges[i].type == 'd')
      {
        update u = updateEdges[i];
        int src = u.source;
        int dest = u.destination;

        if (parent[dest] >= 0 && parent[dest] == src)
        {
          dist[dest] = INT_MAX;
          parent[dest] = -1;
          modified_del[dest] = true;
        }
      }
    }

    G.updateCSR_Del(updateEdges, k, elements);
    Compute_dynamicSSSPDel(G, dist, parent, modified_del);

    for (int i = k; i < (k + batchSize) && i < updateSize; i++)
    {
      if (updateEdges[i].type == 'a')
      {
        update u = updateEdges[i];
        int src = u.source;
        int dest = u.destination;
        int weight = u.weight;
        if (dist[src] != INT_MAX && dist[src] + weight < dist[dest])
          modified_add[src] = true;
      }
    }

    G.updateCSR_Add(updateEdges, k, elements);
    Compute_dynamicSSSPAdd(G, dist, parent, modified_add);
  }
  endTime = std::chrono::high_resolution_clock::now();
  timeTaken = endTime - startTime;
  printf("Total time taken for %d percent updates: %.3f ms\n", percent, timeTaken.count());

  unsigned long long sum = 0;
  for (int i = 0; i < G.num_nodes(); i++)
  {
    if (dist[i] != INT_MAX)
      sum += dist[i];
  }
  printf("Final graph, total path sum = %llu\n\n", sum);

  free(dist);
  free(parent);
  free(modified_add);
  free(modified_del);

  return 0;
}