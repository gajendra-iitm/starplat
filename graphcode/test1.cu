#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

class Layer{
	public:
		int32_t num_features;
		double *weights;
		double *bias;
		double epsilon = 0.01;
		double grad_epsilon;
		double *aggregatedFeatures;
		double *preActivatedFeatures;
		double *postActivatedFeatures;
		double *grad_pre_act_output;
		double *grad_weights;
		double *grad_bias;
		double *m_weights;
		double *m_biases;
		double m_epsilon = 0.0f;
		double v_epsilon = 0.0f;
		double *v_weights;
		double *v_biases;

		//Constructor to initialize layers
		__device__ Layer() : num_features(0), weights(nullptr), bias(nullptr), epsilon(0.01), grad_epsilon(0.0), aggregatedFeatures(nullptr), preActivatedFeatures(nullptr), postActivatedFeatures(nullptr),grad_pre_act_output(nullptr), grad_weights(nullptr), grad_bias(nullptr), m_weights(nullptr), m_biases(nullptr), m_epsilon(0.0f), v_epsilon(0.0f), v_weights(nullptr), v_biases(nullptr) {}


		__device__ void initializeLayers(int32_t layer_num, int32_t num_feat_current, int32_t num_feat_prev, int32_t num_nodes){
			weights = new double[num_feat_prev * num_feat_current];
			bias = new double[ num_feat_current];
			grad_weights = new double[num_feat_prev * num_feat_current];
			grad_bias = new double[num_feat_current];
			m_weights = new double[num_feat_prev * num_feat_current];
			m_biases = new double[num_feat_current];
			v_weights = new double[num_feat_prev * num_feat_current];
			v_biases = new double[num_feat_current];

			aggregatedFeatures = new double[num_nodes * num_feat_current];
			preActivatedFeatures = new double[num_nodes * num_feat_current];
			postActivatedFeatures = new double[num_nodes * num_feat_current];
			grad_pre_act_output = new double[num_nodes * num_feat_current];
		}
};

__global__ void initializeLayers_cuda(Layer *layers, int *numFeaturesPerLayer, int num_nodes){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx == 0){
		layers[idx].num_features =  numFeaturesPerLayer[idx];
		//  layers[0].preActivatedFeatures = new double *[gnn.getGraph().num_nodes()];
		//   layers[0].postActivatedFeatures = new double *[gnn.getGraph().num_nodes()];
		layers[idx].preActivatedFeatures = new double[num_nodes * numFeaturesPerLayer[idx]];
		layers[idx].postActivatedFeatures = new double[num_nodes * numFeaturesPerLayer[idx]];
	}
	layers[idx].initializeLayers(idx, numFeaturesPerLayer[idx], numFeaturesPerLayer[idx-1], num_nodes);
}

void initializeLayers_cuda(GNN &gnn, vector<int> numFeaturesPerLayer, char *transformation){
	int num_layers = numNodespreLayer.size() - 1;
	Layer *layers;
	if(strcmp(transformation, "xaviers") == 0){
		transformType = 1;
	}
	else if(strcmp(transformation, "he") == 0){
		transformType = 2;
	}
	hipMallocManaged(&layers, num_layers * sizeof(Layer));
	//create numFeaturesperLayer in cuda and copy
	int *d_numFeaturesperLayer;
	hipMallocManaged(&d_numFeaturesperLayer, num_layers * sizeof(int));
	hipMemcpy(h_numFeaturesperLayer, numFeaturesperLayer.data(), num_layers * sizeof(int), hipMemcpyHostToDevice);
	//initialize layers
	initializeLayers_cuda<<<1, num_layers>>>(layers, h_numFeaturesperLayer, num_nodes);
	hipDeviceSynchronize();

}

gnn.forward(a,b,c) -> forward_cuda(gnn, a ,b, c)

void forward(){


	aggregation<<<1,1>>>();

	NNoperation<<<1,1>>>();



}



void optimizer(char *algorithm, char * optimizer_type, int batch_size, double learning_rate, double weight_decay){

	
	if(strcmp(algorithm,"GraphSage")==0){
		forward_Sage(batch_size);
	}
	else if(strcmp(algorithm,"GCN")==0){
		forward_GCN();
	}
	else if(strcmp(algorithm, "GIN")==0){
		forward_GIN();
	}

	if(strcmp(optimizer_type, "adam") == 0){
		adam_optimizer(learning_rate, weight_decay);
	}
	else if(strcmp(optimizer_type, "sgd") == 0){
		sgd_optimizer(learning_rate, weight_decay);
	}
	else if(strcmp(optimizer_type, "rmsprop") == 0){
		rmsprop_optimizer(learning_rate, weight_decay);
	}


}


__global__ void GCN_aggregate_cuda(int numNodes, int numFeatures, int* rowPtr, int* colIdx, double* edgeWeight, double* postActivatedFeatures, double* aggregatedFeatures)
{
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= numNodes) return; 

    for (int i = 0; i < numFeatures; ++i) {
        aggregatedFeatures[node * numFeatures + i] = 0.0f;
    }

    for (int edge = rowPtr[node]; edge < rowPtr[node + 1]; ++edge) {		    // Iterate over the neighbors of the node

        int neighbor = colIdx[edge];
        double weight = edgeWeight[edge];
        
        for (int i = 0; i < numFeatures; ++i) {				        // Aggregate features from neighbor node
            atomicAdd(&aggregatedFeatures[node * numFeatures + i], 
                      postActivatedFeatures[neighbor * numFeatures + i] * weight);
        }
    }
}

__global__ void softmaxKernel(double* data, int numFeatures)
{
    extern __shared__ double temp[];
    int tid = threadIdx.x;

    
    temp[tid] = expf(data[tid]);
    __syncthreads();

    
    double sum = 0.0f;
    for (int i = 0; i < numFeatures; ++i)
        sum += temp[i];
    
    __syncthreads();
    
    data[tid] = temp[tid] / sum;
}


__global__ void Activation_cuda(double* preActivated, double* postActivated, double* bias, int numFeatures, int initType, int totalLayers, int currentLayer)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numFeatures)
    {
        // Add bias
        preActivated[i] += bias[i];

        // Apply activation
        if (initType == 2 && currentLayer<totalLayers)
            postActivated[i] = fmaxf(0.0f, preActivated[i]);  // ReLU
        if (initType == 1 && currentLayer<totalLayers)
            postActivated[i] = tanhf(preActivated[i]);  // Tanh
	}
}


void forward(){

	double* d_aggregatedFeatures = layers[layerNumber].aggregatedFeatures[node];  // Aggregated features (GPU memory)
    double* d_weights = layers[layerNumber].weights;  // Weights matrix 
    double* d_preActivatedFeatures = layers[layerNumber].preActivatedFeatures[node];  // Pre-activation features
    double* d_bias = layers[layerNumber].bias;  // Bias
    double* d_postActivatedFeatures = layers[layerNumber].postActivatedFeatures[node];  // Post-activation features
	int numBlocks = (numNodes + 1024 - 1) / 1024;
	GCN_aggregate_cuda<<<numBlocks, 1024>>>(numNodes,numFeatures, d_rowPtr, d_colIdx, d_edgeWeight, d_postActivatedFeatures, d_aggregatedFeatures)
	hipDeviceSynchronize();
//	cusparse()
	int blockSize = 256;
    int gridSize = (numFeatures + blockSize - 1) / blockSize;
	Activation_cuda<<<gridSize, blockSize>>>(d_preActivatedFeatures, d_postActivatedFeatures, d_bias, numFeatures,activationType);
	 if (layerNumber == layers.size() - 1)
    {
        softmaxKernel<<<1, numFeatures, numFeatures * sizeof(double)>>>(d_postActivatedFeatures, numFeatures);
    }
}


__global__ void computeGradientsOutputLayer(double *y_pred, int *y_true, double *grad_output, int num_nodes, int num_classes)
{
    int nodeIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (nodeIdx < num_nodes) {
        int label = y_true[nodeIdx];  
        for (int j = 0; j < num_classes; ++j) {
            grad_output[nodeIdx * num_classes + j] = y_pred[nodeIdx * num_classes + j] - (label == j ? 1.0f : 0.0f);
        }
    }
}

__global__ void computeGradientsIntermediateLayer(double *grad_next_layer, double *weights_next_layer, double *grad_current_layer, double *pre_activated_features, int num_nodes, int num_features, int next_layer_features, int activation_type)
{
    int nodeIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (nodeIdx < num_nodes) {
        for (int i = 0; i < num_features; ++i) {
            double grad_sum = 0.0f;
            for (int j = 0; j < next_layer_features; ++j) {
                grad_sum += grad_next_layer[nodeIdx * next_layer_features + j] * weights_next_layer[i * next_layer_features + j];
            }

            if (activation_type == 1) {  // Tanh
                grad_current_layer[nodeIdx * num_features + i] = grad_sum * (1 - pre_activated_features[nodeIdx * num_features + i] * pre_activated_features[nodeIdx * num_features + i]);
            } else if (activation_type == 2) {  // ReLU
                grad_current_layer[nodeIdx * num_features + i] = (pre_activated_features[nodeIdx * num_features + i] > 0) ? grad_sum : 0;
            }
        }
    }
}
__global__ void computeWeightBiasGradients(double *aggregated_features, double *grad_pre_act_output, double *grad_weights, double *grad_bias, int num_nodes, int num_input_features, int num_output_features)
{
    int inputIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (inputIdx < num_input_features) {
        for (int j = 0; j < num_output_features; ++j) {
            double weight_grad_sum = 0.0f;
            double bias_grad_sum = 0.0f;
            
            for (int nodeIdx = 0; nodeIdx < num_nodes; ++nodeIdx) {
                weight_grad_sum += aggregated_features[nodeIdx * num_input_features + inputIdx] * grad_pre_act_output[nodeIdx * num_output_features + j];
                bias_grad_sum += grad_pre_act_output[nodeIdx * num_output_features + j];
            }

            grad_weights[inputIdx * num_output_features + j] = weight_grad_sum;
            if (inputIdx == 0) {  
                grad_bias[j] = bias_grad_sum;
            }
        }
    }
}
__global__ void clipGradients(double *grad_weights, double *grad_bias, int num_input_features, int num_output_features, double clip_value)
{
    int weightIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (weightIdx < num_input_features * num_output_features) {
        if (abs(grad_weights[weightIdx]) > clip_value) {
            grad_weights[weightIdx] = clip_value * (grad_weights[weightIdx] > 0 ? 1.0f : -1.0f);
        }
    }

    int biasIdx = threadIdx.x;
    if (biasIdx < num_output_features) {
        if (abs(grad_bias[biasIdx]) > clip_value) {
            grad_bias[biasIdx] = clip_value * (grad_bias[biasIdx] > 0 ? 1.0f : -1.0f);
        }
    }
}


void backPropagation_cuda(GNN &gnn, int layerNumber, int numNodes, int numInputFeatures, int numOutputFeatures) {
    double *d_y_pred, *d_grad_output, *d_weights, *d_preActivatedFeatures, *d_aggregatedFeatures;
    int *d_y_true;
    
    if (layerNumber == gnn.numLayers() - 1) {
        int threads = 1024;
        int blocks = (numNodes + threads - 1) / threads;
        computeGradientsOutputLayer<<<blocks, threads>>>(d_y_pred, d_y_true, d_grad_output, numNodes, gnn.numClasses());
    } else {
        int threads = 1024;
        int blocks = (numNodes + threads - 1) / threads;
        computeGradientsIntermediateLayer<<<blocks, threads>>>(d_grad_output, d_weights, d_grad_output, d_preActivatedFeatures, numNodes, numInputFeatures, numOutputFeatures, gnn.initType());
    }

    int threads = 1024;
    int blocks = (numInputFeatures + threads - 1) / threads;
    computeWeightBiasGradients<<<blocks, threads>>>(d_aggregatedFeatures, d_grad_output, d_weights, d_bias, numNodes, numInputFeatures, numOutputFeatures);
    clipGradients<<<blocks, threads>>>(d_weights, d_bias, numInputFeatures, numOutputFeatures, 3.0);
}
