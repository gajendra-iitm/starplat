#include <iostream>
#include <hip/hip_runtime.h>

// Define the Layer struct
struct Layer {
    int num_features;
    float *weights;
    float *aggregatedFeatures;
    float *preActivatedFeatures;
};

// Kernel function
__global__ void NeuralNetworkOperation(Layer *layer, int numNodes, int layerNumber)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numNodes)
    {
        for (int i = 0; i < layer[layerNumber + 1].num_features; i++) // For each feature in the next layer
        {
            float sum = 0.0f;
            for (int j = 0; j < layer[layerNumber].num_features; j++) // For each feature in the current layer
            {
                sum += layer[layerNumber].weights[i * layer[layerNumber].num_features + j] *
                       layer[layerNumber].aggregatedFeatures[idx * layer[layerNumber].num_features + j];
                
                // Debugging output
                if (idx == 0 && j < 3) // Print for the first node and a few weights/features
                {
                    printf("Node %d: weight[%d] = %f, feature[%d] = %f, partial sum = %f\n",
                           idx, i * layer[layerNumber].num_features + j, 
                           layer[layerNumber].weights[i * layer[layerNumber].num_features + j], 
                           idx * layer[layerNumber].num_features + j, 
                           layer[layerNumber].aggregatedFeatures[idx * layer[layerNumber].num_features + j], 
                           sum);
                }
            }
            layer[layerNumber].preActivatedFeatures[idx * layer[layerNumber + 1].num_features + i] = sum;
        }
    }
}

int main()
{
    // Define the sizes and number of layers (example values)
    const int numLayers = 2; // Number of layers
    const int numNodes = 4; // Number of nodes in the current layer
    const int numFeatures = 3; // Number of features in each layer
    const int numFeaturesNextLayer = 2; // Number of features in the next layer

    // Allocate memory for layers
    Layer *d_layer;
    hipMalloc(&d_layer, sizeof(Layer) * numLayers);

    // Initialize Layer structs and allocate device memory for weights and features
    Layer h_layer[numLayers];
    // Example Initialization (make sure to use realistic values)
float h_weights[] = {0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f}; // Example weights
float h_features[] = {1.0f, 2.0f, 3.0f, 4.0f}; // Example features

hipMemcpy(d_layer[0].weights, h_weights, sizeof(h_weights), hipMemcpyHostToDevice);
hipMemcpy(d_layer[0].aggregatedFeatures, h_features, sizeof(h_features), hipMemcpyHostToDevice);

    for (int i = 0; i < numLayers; ++i) {
        h_layer[i].num_features = (i == numLayers - 1) ? numFeaturesNextLayer : numFeatures;
        hipMalloc(&h_layer[i].weights, h_layer[i].num_features * (i == numLayers - 1 ? numFeatures : numFeatures) * sizeof(float));
        hipMalloc(&h_layer[i].aggregatedFeatures, numNodes * h_layer[i].num_features * sizeof(float));
        hipMalloc(&h_layer[i].preActivatedFeatures, numNodes * (i == numLayers - 1 ? numFeaturesNextLayer : numFeatures) * sizeof(float));
        hipMemcpy(&d_layer[i], &h_layer[i], sizeof(Layer), hipMemcpyHostToDevice);
    }

    // Launch the kernel
    int blockSize = 4;
    int gridSize = (numNodes + blockSize - 1) / blockSize;
    NeuralNetworkOperation<<<gridSize, blockSize>>>(d_layer, numNodes, 0);

    // Check for CUDA errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    // Copy results back and print
    float *h_preActivatedFeatures = new float[numNodes * numFeaturesNextLayer];
    hipMemcpy(h_preActivatedFeatures, h_layer[0].preActivatedFeatures, numNodes * numFeaturesNextLayer * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Pre-activated features after kernel execution:\n";
    for (int i = 0; i < numNodes; i++)
    {
        for (int j = 0; j < numFeaturesNextLayer; j++)
        {
            std::cout << h_preActivatedFeatures[i * numFeaturesNextLayer + j] << " ";
        }
        std::cout << "\n";
    }

    delete[] h_preActivatedFeatures;

    // Free device memory
    for (int i = 0; i < numLayers; ++i) {
        hipFree(h_layer[i].weights);
        hipFree(h_layer[i].aggregatedFeatures);
        hipFree(h_layer[i].preActivatedFeatures);
    }
    hipFree(d_layer);

    return 0;
}
