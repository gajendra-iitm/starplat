// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "dyn4_ud_simple.h"

void staticMaxFlow(graph& g,int source,int sink,int* d_residual_capacity,
  int* d_rev_residual_capacity,int* d_reverse_edge,int* d_excess,int* d_parallel_edge,
  int kernel_cycles)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  int V = g.num_nodes(); // asst in .cu

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  staticMaxFlow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity,d_parallel_edge);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  staticMaxFlow_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_reverse_edge,d_residual_capacity,d_excess,d_rev_residual_capacity,d_parallel_edge);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  bool flag1 = true; // asst in .cu

  do{
    flag1 = false;
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)V);

    d_height[sink] = 0;

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;
    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));  hipMemset(d_hops_from_source,0,sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    //EXTRA vars INITIALIZATION
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_level,-1);
    initIndex<int><<<1,1>>>(V,d_level,sink, 0);

    // long k =0 ;// For DEBUG
    do {
      finished = true;
      hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_parallel_edge); ///DONE from varList

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST - rupesh
      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    }while(!finished);

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished
        ,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_parallel_edge); ///DONE from varList

      hops_from_source--;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
    }
    //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::V), &V, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::kernel_cycles), &kernel_cycles, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    staticMaxFlow_kernel_12<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_residual_capacity,d_height,d_rev_residual_capacity,d_parallel_edge,d_reverse_edge,d_excess);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&V, HIP_SYMBOL(::V), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&kernel_cycles, HIP_SYMBOL(::kernel_cycles), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag1);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_height);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
void Incremental(graph& g, int V,int E,int* d_meta,int* d_data,int* d_rev_meta,int* d_src,int* d_weight, int source, int sink, int* d_parallel_edge, 
  int* d_residual_capacity, int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess, 
  int* d_height, int kernel_cycles)
{

  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  int V = g.num_nodes(); // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  incremental_kernel_15<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_residual_capacity,d_reverse_edge,d_excess,d_rev_residual_capacity,d_parallel_edge);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  bool flag1 = true; // asst in .cu

  do{
    flag1 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::kernel_cycles), &kernel_cycles, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::V), &V, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    incremental_kernel_18<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_rev_residual_capacity,d_excess,d_reverse_edge,d_height,d_residual_capacity,d_parallel_edge);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&kernel_cycles, HIP_SYMBOL(::kernel_cycles), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&V, HIP_SYMBOL(::V), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag1);


}
void Decremental(graph& g, int V,int E,int* d_meta,int* d_data,int* d_rev_meta,int* d_src,int* d_weight, int source, int sink, int* d_parallel_edge, 
  int* d_residual_capacity, int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess, 
  int* d_height, int cycle)
{

  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,sink,(int)g.num_nodes()); //InitIndexDevice
  int V = g.num_nodes(); // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
  decremental_kernel_21<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_excess,d_residual_capacity,d_parallel_edge);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  bool flag1 = true; // asst in .cu

  do{
    flag1 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::V), &V, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    decremental_kernel_24<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_excess,d_residual_capacity,d_height,d_parallel_edge,d_reverse_edge,d_rev_residual_capacity);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&V, HIP_SYMBOL(::V), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag1);


}
void recalculate_max_flow(graph& g, std::vector<update> updateBatch, int batchSize, int source, 
  int sink, int cycle)
{
   int V = g.num_nodes();
   int E = g.num_edges();
  int* edgeLen = g.getEdgeLen();
  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;
  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  for(int i=0; i<= V; i++) {
       int temp;
      temp = g.indexofNodes[i];
      h_meta[i] = temp;
     temp = g.rev_indexofNodes[i];
      h_rev_meta[i] = temp;
  }
   for(int i=0; i< E; i++) {
     int temp;
    temp = g.edgeList[i];
      h_data[i] = temp;
      temp = g.srcList[i];
      h_src[i] = temp;
      temp = edgeLen[i];
    h_weight[i] = temp;
  }
  int *d_meta;
  int *d_data;
  int *d_src;
  int *d_weight;
  int *d_rev_meta;
  hipMalloc(&d_meta,sizeof(int)*(V+1));
  hipMalloc(&d_data,sizeof(int)*(E));
  hipMalloc(&d_src,sizeof(int)*(E));
  hipMalloc(&d_weight,sizeof(int)*(E));
  hipMalloc(&d_rev_meta,sizeof(int)*(V+1));
  hipMemcpy(d_meta,h_meta,sizeof(int)*(V+1),hipMemcpyHostToDevice);
  hipMemcpy(d_data,h_data,sizeof(int)*(E),hipMemcpyHostToDevice);
  hipMemcpy(d_src,h_src,sizeof(int)*(E),hipMemcpyHostToDevice);
  hipMemcpy(d_weight,h_weight,sizeof(int)*(E),hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta,h_rev_meta,sizeof(int)*(V+1),hipMemcpyHostToDevice);
  unsigned threadsPerBlock   = (V < THREADS_PER_BLOCK)? V: THREADS_PER_BLOCK;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  int* d_parallel_edge;
  hipMalloc(&d_parallel_edge, sizeof(int)*(E));

  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  staticMaxFlow(g,source,sink,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_parallel_edge,cycle);

  int _batchSize = batchSize;
  update *d_updateBatch;
  hipMalloc(&d_updateBatch,sizeof(update)*_batchSize);
  int batchElements = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  for( int updateIndex = 0 ; updateIndex < updateBatch.size() ; updateIndex += _batchSize){
    if((updateIndex + _batchSize) > updateBatch.size())
    {
      batchElements = updateBatch.size() - updateIndex ;
    }
    else
    batchElements = _batchSize ;
    hipMemcpy(d_updateBatch,&updateBatch[updateIndex],batchElements*sizeof(update),hipMemcpyHostToDevice);
    unsigned updateThreads = (batchElements < THREADS_PER_BLOCK)? batchElements: THREADS_PER_BLOCK;
    unsigned updateBlocks = (batchElements+updateThreads-1)/updateThreads;

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    OnAdd_kernel<<<updateBlocks,updateThreads>>>(d_updateBatch,batchElements, d_meta, d_data, d_weight, d_rev_residual_capacity, d_parallel_edge, d_residual_capacity);
    recalculate_max_flow_kernel_28<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_excess);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    Decremental(g,V,E,d_meta,d_data,d_rev_meta,d_src,d_weight,source,sink,d_parallel_edge,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_height,cycle);

    Incremental(g,V,E,d_meta,d_data,d_rev_meta,d_src,d_weight,source,sink,d_parallel_edge,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_height,cycle);



  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
  recalculate_max_flow_kernel_32<<<numBlocks, threadsPerBlock>>>(V,E,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);





  hipMemcpy(h_meta,d_meta,sizeof(int)*(V+1),hipMemcpyDeviceToHost);
  hipMemcpy(h_data,d_data,sizeof(int)*(E),hipMemcpyDeviceToHost);
  hipMemcpy(h_src,d_src,sizeof(int)*(E),hipMemcpyDeviceToHost);
  hipMemcpy(h_weight,d_weight,sizeof(int)*(E),hipMemcpyDeviceToHost);
  hipMemcpy(h_rev_meta,d_rev_meta,sizeof(int)*(V+1),hipMemcpyDeviceToHost);
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_src);
  hipFree(d_weight);
  hipFree(d_rev_meta);
}
