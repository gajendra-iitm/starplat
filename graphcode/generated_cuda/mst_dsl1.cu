// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_dsl1.h"

void Boruvka(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V > threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_nodeId;
  hipMalloc(&d_nodeId, sizeof(int)*(V));

  int* d_color;
  hipMalloc(&d_color, sizeof(int)*(V));

  bool* d_isMSTEdge;
  hipMalloc(&d_isMSTEdge, sizeof(bool)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_nodeId,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color,(int)-1);

  Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_color,d_nodeId);
  hipDeviceSynchronize();



  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  int* d_minEdgeOfComp;
  hipMalloc(&d_minEdgeOfComp, sizeof(int)*(V));

  int* d_minEdge;
  hipMalloc(&d_minEdge, sizeof(int)*(V));

  bool noNewComp = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!noNewComp) {

    noNewComp = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdge,(int)-1);

    Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_color,d_minEdge);
    hipDeviceSynchronize();



    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdgeOfComp,(int)-1);

    bool finishedMinEdge = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finishedMinEdge) {

      finishedMinEdge = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finishedMinEdge), &finishedMinEdge, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finishedMinEdge), &finishedMinEdge, sizeof(bool), 0, hipMemcpyHostToDevice);
      Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_color,d_minEdge,d_minEdgeOfComp);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finishedMinEdge, HIP_SYMBOL(::finishedMinEdge), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu


      hipMemcpyFromSymbol(&finishedMinEdge, HIP_SYMBOL(::finishedMinEdge), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_color,d_nodeId,d_minEdgeOfComp);
    hipDeviceSynchronize();



    Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_color,d_nodeId,d_minEdgeOfComp,d_isMSTEdge);
    hipDeviceSynchronize();



    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_color,d_nodeId,d_minEdgeOfComp);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);



    bool finished = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      Boruvka_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_color);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_minEdge);
  hipFree(d_minEdgeOfComp);
  hipFree(d_modified);
  hipFree(d_nodeId);
  hipFree(d_color);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
