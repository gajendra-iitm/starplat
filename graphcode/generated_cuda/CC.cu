// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "CC.h"

void Compute_CC(graph& g,float* CC,std::set<int>& sourceSet)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_CC;
  hipMalloc(&d_CC, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_CC,(float)0);

  int numNodes = g.num_nodes( ); // asst in .cu

  int* d_dist;
  hipMalloc(&d_dist, sizeof(int)*(V));

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  //FOR SIGNATURE of SET - Assumes set for on .cu only
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++) 
  {
    int src = *itr;
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,(int)INT_MAX);

    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

    initIndex<bool><<<1,1>>>(V,d_modified,src,(bool)true); //InitIndexDevice
    initIndex<int><<<1,1>>>(V,d_dist,src,(int)0); //InitIndexDevice
    bool finished = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      Compute_CC_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_dist,d_modified);
      hipDeviceSynchronize();




      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    int temp = 0; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::temp), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
    Compute_CC_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_dist);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&temp, HIP_SYMBOL(::temp), sizeof(int), 0, hipMemcpyDeviceToHost);



    initIndex<float><<<1,1>>>(V,d_CC,src,(float)1.000000 / temp); //InitIndexDevice

  }

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);
  hipFree(d_dist);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(      CC,     d_CC, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
