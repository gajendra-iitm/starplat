// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRank_DSL_V2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_pageRank;
  hipMalloc(&d_pageRank, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  float* d_pageRank_nxt;
  hipMalloc(&d_pageRank_nxt, sizeof(float)*(V));

  float num_nodes = (float)g.num_nodes( ); // asst in .cu

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,(float)1 / num_nodes);

  int iterCount = 0; // asst in .cu

  float diff; // asst in .cu

  bool tempVar_0 = false; // asst in .cu

  do{
    if (tempVar_0){ // if filter begin 

    } // if filter end
    tempVar_0 = true;
    diff = 0.000000;
    hipMemcpyToSymbol(HIP_SYMBOL(::diff), &diff, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::num_nodes), &num_nodes, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::delta), &delta, sizeof(float), 0, hipMemcpyHostToDevice);
    Compute_PR_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_pageRank,d_pageRank_nxt);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    hipMemcpy(d_pageRank, d_pageRank_nxt, sizeof(float)*V, hipMemcpyDeviceToDevice);
    iterCount++;
    hipMemcpyFromSymbol(&diff, HIP_SYMBOL(::diff), sizeof(float), 0, hipMemcpyDeviceToHost);

  }while((diff > beta) && (iterCount < maxIter));

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_pageRank_nxt);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
