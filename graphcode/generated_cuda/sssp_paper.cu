// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "sssp_paper.h"

void computeSSSP(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int src = 0; // asst in .cu

  int* d_dist;
  hipMalloc(&d_dist, sizeof(int)*(V));

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,(int)INT_MAX);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  initIndex<bool><<<1,1>>>(V,d_modified,src,(bool)true); //InitIndexDevice
  initIndex<int><<<1,1>>>(V,d_dist,src,(int)0); //InitIndexDevice
  bool finished = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!finished) {

    finished = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
    computeSSSP_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_modified);
    hipDeviceSynchronize();




    hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);
  hipFree(d_dist);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
