// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "push_relabel_check.h"

void relabel(graph& g,int u,int* residue,int* label,
  thrust::host_vector<int> count)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();







  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  int* d_residue;
  hipMalloc(&d_residue, sizeof(int)*(E));

  int* d_label;
  hipMalloc(&d_label, sizeof(int)*(V));


  //BEGIN DSL PARSING 
  int x = label[u]; // asst in .cu

  int new_label = g.num_nodes() + 2; // asst in .cu

  int residual_capacity = edge;
  if (residue[residual_capacity] > 0){ // if filter begin 
    if (new_label < v){ // if filter begin 
      new_label = v;

    } // if filter end

  } // if filter end

} //  end FOR NBR ITR. TMP FIX!
if (new_label < g.num_nodes() + 1 && new_label + 1 > x){ // if filter begin 
  d_label[u] = new_label + 1;

} // if filter end

//TIMER STOP
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("GPU Time: %.6f ms\n", milliseconds);

hipMemcpy( residue, d_residue, sizeof(int)*(E), hipMemcpyDeviceToHost);
hipMemcpy(   label,  d_label, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN
void fixGap(thrust::host_vector<int> count,int* label)

{
// CSR BEGIN
int V = label.num_nodes();
int E = label.num_edges();

printf("#nodes:%d\n",V);
printf("#edges:%d\n",E);
int* edgeLen = label.getEdgeLen();







// CSR END
//LAUNCH CONFIG
const unsigned threadsPerBlock = 512;
unsigned numThreads   = (V < threadsPerBlock)? 512: V;
unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


// TIMER START
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float milliseconds = 0;
hipEventRecord(start,0);


//DECLAR DEVICE AND HOST vars in params
int* d_label;
hipMalloc(&d_label, sizeof(int)*(V));


//BEGIN DSL PARSING 
int gap = count.getIdx(0); // asst in .cu

if (gap == -1){ // if filter begin 
} // if filter end
hipMemcpyToSymbol(HIP_SYMBOL(::gap), &gap, sizeof(int), 0, hipMemcpyHostToDevice);
fixGap_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_label);
hipDeviceSynchronize();
hipMemcpyFromSymbol(&gap, HIP_SYMBOL(::gap), sizeof(int), 0, hipMemcpyDeviceToHost);



()
//TIMER STOP
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("GPU Time: %.6f ms\n", milliseconds);

hipMemcpy(   label,  d_label, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN
