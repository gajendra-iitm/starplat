// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "APFB_Matching.h"

void APFB(graph& g,int nc)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V > threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  int* d_rmatch;
  hipMalloc(&d_rmatch, sizeof(int)*(V));

  int* d_cmatch;
  hipMalloc(&d_cmatch, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_rmatch,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_cmatch,(int)-1);

  bool noNewPaths = false; // asst in .cu

  int* d_bfsArray;
  hipMalloc(&d_bfsArray, sizeof(int)*(V));

  int* d_predeccesor;
  hipMalloc(&d_predeccesor, sizeof(int)*(V));

  bool* d_compress;
  hipMalloc(&d_compress, sizeof(bool)*(V));

  bool* d_compress_next;
  hipMalloc(&d_compress_next, sizeof(bool)*(V));

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!noNewPaths) {

    noNewPaths = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewPaths), &noNewPaths, sizeof(bool), 0, hipMemcpyHostToDevice);
    int L0 = 0; // asst in .cu

    int NOT_VISITED = L0 - 1; // asst in .cu

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_bfsArray,(int)NOT_VISITED);

    hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::L0), &L0, sizeof(int), 0, hipMemcpyHostToDevice);
    APFB_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_cmatch,d_bfsArray);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&L0, HIP_SYMBOL(::L0), sizeof(int), 0, hipMemcpyDeviceToHost);



    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_predeccesor,(int)-1);

    int bfsLevel = L0; // asst in .cu

    bool noNewVertices = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!noNewVertices) {

      noNewVertices = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::noNewVertices), &noNewVertices, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::NOT_VISITED), &NOT_VISITED, sizeof(int), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::bfsLevel), &bfsLevel, sizeof(int), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::noNewPaths), &noNewPaths, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::noNewVertices), &noNewVertices, sizeof(bool), 0, hipMemcpyHostToDevice);
      APFB_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_bfsArray,d_rmatch,d_predeccesor);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&NOT_VISITED, HIP_SYMBOL(::NOT_VISITED), sizeof(int), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&bfsLevel, HIP_SYMBOL(::bfsLevel), sizeof(int), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&noNewPaths, HIP_SYMBOL(::noNewPaths), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&noNewVertices, HIP_SYMBOL(::noNewVertices), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      bfsLevel = bfsLevel + 1;

      hipMemcpyFromSymbol(&noNewVertices, HIP_SYMBOL(::noNewVertices), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_compress,(bool)false);

    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_compress_next,(bool)false);

    hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
    APFB_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_rmatch,d_compress);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);



    bool compressed = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!compressed) {

      compressed = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::compressed), &compressed, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::compressed), &compressed, sizeof(bool), 0, hipMemcpyHostToDevice);
      APFB_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_compress,d_predeccesor,d_cmatch,d_compress_next,d_rmatch);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&compressed, HIP_SYMBOL(::compressed), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
      APFB_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_compress_next,d_compress);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);




      hipMemcpyFromSymbol(&compressed, HIP_SYMBOL(::compressed), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
    APFB_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_rmatch,d_cmatch);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu


    hipMemcpyFromSymbol(&noNewPaths, HIP_SYMBOL(::noNewPaths), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_compress_next);
  hipFree(d_compress);
  hipFree(d_rmatch);
  hipFree(d_cmatch);
  hipFree(d_modified);
  hipFree(d_bfsArray);
  hipFree(d_predeccesor);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
