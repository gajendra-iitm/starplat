// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRankDSLV2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_src;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.srcList[i];
    h_src[i] = temp;
  }


  int* d_meta;
  int* d_src;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_pageRank;
  hipMalloc(&d_pageRank, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  float* d_pageRank_nxt;
  hipMalloc(&d_pageRank_nxt, sizeof(float)*(V));

  float num_nodes = (float)g.num_nodes( ); // asst in .cu

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,(float)1 / num_nodes);

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank_nxt,(float)0);

  int iterCount = 0; // asst in .cu

  float diff; // asst in .cu

  bool tempVar_0 = false; // asst in .cu

  do{
    if (tempVar_0){ // if filter begin 

    } // if filter end
    tempVar_0 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::delta), &delta, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::num_nodes), &num_nodes, sizeof(float), 0, hipMemcpyHostToDevice);
    Compute_PR_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_src,d_rev_meta,d_pageRank,d_pageRank_nxt);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    hipMemcpy(d_pageRank, d_pageRank_nxt, sizeof(float)*V, hipMemcpyDeviceToDevice);
    iterCount++;

  }while((diff > beta) && (iterCount < maxIter));

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_pageRank_nxt);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
