// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_delete-v2.h"

void recolor(bool* isCovered,graph& g,int* parent,int* destination
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));

  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));


  //BEGIN DSL PARSING 
  recolor_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
  hipDeviceSynchronize();



  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)true);

  bool rfinish1 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!rfinish1) {

    rfinish1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::rfinish1), &rfinish1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::rfinish1), &rfinish1, sizeof(bool), 0, hipMemcpyHostToDevice);
    recolor_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_destination,d_parent);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&rfinish1, HIP_SYMBOL(::rfinish1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    bool rfinish2 = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!rfinish2) {

      rfinish2 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::rfinish2), &rfinish2, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::rfinish2), &rfinish2, sizeof(bool), 0, hipMemcpyHostToDevice);
      recolor_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&rfinish2, HIP_SYMBOL(::rfinish2), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&rfinish2, HIP_SYMBOL(::rfinish2), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


    hipMemcpyFromSymbol(&rfinish1, HIP_SYMBOL(::rfinish1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(isCovered, d_isCovered, sizeof(bool)*(E), hipMemcpyDeviceToHost);
  hipMemcpy(  parent, d_parent, sizeof(int)*(V), hipMemcpyDeviceToHost);
  hipMemcpy(destination, d_destination, sizeof(int)*(E), hipMemcpyDeviceToHost);
} //end FUN
