// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "dyn4_test.h"

void staticMaxFlow(graph& g,int source2,int sink2,int* d_residual_capacity,
  int* d_rev_residual_capacity,int* d_reverse_edge,int* d_excess,int cycle
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  staticMaxFlow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
  staticMaxFlow_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag1 = true; // asst in .cu

  int kernel = 1000; // asst in .cu

  do{
    flag1 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::cycle), &cycle, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    staticMaxFlow_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&cycle, HIP_SYMBOL(::cycle), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag1);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_height);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
void Incremental(graph& g, int V,int E,int* d_meta,int* d_data,int* d_rev_meta,int* d_src,int* d_weight, int source0, int sink0, int* d_residual_capacity, 
  int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess, int cycle
)
{

  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source0,(int)g.num_nodes()); //InitIndexDevice
  hipMemcpyToSymbol(HIP_SYMBOL(::source0), &source0, sizeof(int), 0, hipMemcpyHostToDevice);
  incremental_kernel_10<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source0, HIP_SYMBOL(::source0), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag2 = true; // asst in .cu

  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source0), &source0, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink0), &sink0, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::cycle), &cycle, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    incremental_kernel_12<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_excess,d_height,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source0, HIP_SYMBOL(::source0), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink0, HIP_SYMBOL(::sink0), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&cycle, HIP_SYMBOL(::cycle), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag2);


}
void Decremental(graph& g, int V,int E,int* d_meta,int* d_data,int* d_rev_meta,int* d_src,int* d_weight, int source1, int sink1, int* d_residual_capacity, 
  int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess, int cycle
)
{

  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  int* d_height2;
  hipMalloc(&d_height2, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height2,(int)0);

  initIndex<int><<<1,1>>>(V,d_height2,sink1,(int)g.num_nodes()); //InitIndexDevice
  decremental_kernel_15<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_excess);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::sink1), &sink1, sizeof(int), 0, hipMemcpyHostToDevice);
  decremental_kernel_18<<<numBlocks, threadsPerBlock>>>(V,E,d_src,d_rev_meta,d_reverse_edge,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&sink1, HIP_SYMBOL(::sink1), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag3 = true; // asst in .cu

  do{
    flag3 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source1), &source1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink1), &sink1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::cycle), &cycle, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag3), &flag3, sizeof(bool), 0, hipMemcpyHostToDevice);
    decremental_kernel_20<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_excess,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity,d_height2);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source1, HIP_SYMBOL(::source1), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink1, HIP_SYMBOL(::sink1), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&cycle, HIP_SYMBOL(::cycle), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag3, HIP_SYMBOL(::flag3), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag3);


}
