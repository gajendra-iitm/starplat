// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "sssp_dslV3.h"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  int* d_dist;
  hipMalloc(&d_dist, sizeof(int)*(V));


  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,(int)INT_MAX);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  initIndex<bool><<<1,1>>>(V,d_modified,src,(bool)true); //InitIndexDevice
  initIndex<int><<<1,1>>>(V,d_dist,src,(int)0); //InitIndexDevice
  bool finished = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!finished) {

    finished = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
    Compute_SSSP_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_dist,d_modified);
    hipDeviceSynchronize();




    hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(    dist,   d_dist, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN

int main(int argc,char* argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  int *distance = (int *)malloc((g.num_nodes() + 1) * sizeof(int));
  int src = 1;
  Compute_SSSP(g, distance, src);
  for (int i = 0; i <= g.num_nodes(); i++)
  {
    std::cout << distance[i] << " ";
  }
  std::cout << std::endl;
  // std::cout << INT_MAX << std::endl;
  return 0;
}
