#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "dyn4_modify.h"
#define THREADS_PER_BLOCK 1024
void checkCudaError( int  i)
{       
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)  
  {   
    printf("%d , CUDA error: %s\n", i, hipGetErrorString(error));
    exit(0);
  } 
} 
__global__ void print_excess(int V,int*d_excess){
  unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
  if(v >= V) return;
  if(d_excess[v]!=0){
    printf("Active vertex:%d excess[%d]=%d\n",v,v,d_excess[v]);
  }
}

void staticMaxFlow(graph& g,int source2,int sink2,int* d_residual_capacity,
  int* d_rev_residual_capacity,int* d_reverse_edge,int* d_excess,int cycle
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = THREADS_PER_BLOCK;
  unsigned numThreads   = (V < threadsPerBlock)? V: THREADS_PER_BLOCK;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  staticMaxFlow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
  staticMaxFlow_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag1 = true; // asst in .cu

  int kernel = 1000; // asst in .cu

  do{
    flag1 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    staticMaxFlow_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height,cycle);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag1);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_height);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("STATIC GPU Time: %.6f ms\n", milliseconds);

} //end FUN
void Incremental(graph& g, int V,int E,int* d_meta,int* d_data,int* d_rev_meta,int* d_src,int* d_weight, int source0, int sink0, int* d_residual_capacity, 
  int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess, int cycle
)
{

  const unsigned threadsPerBlock = THREADS_PER_BLOCK;
  unsigned numThreads   = (V < threadsPerBlock)? V: THREADS_PER_BLOCK;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source0,(int)g.num_nodes()); //InitIndexDevice
  hipMemcpyToSymbol(HIP_SYMBOL(::source0), &source0, sizeof(int), 0, hipMemcpyHostToDevice);
  incremental_kernel_10<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source0, HIP_SYMBOL(::source0), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag2 = true; // asst in .cu

  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source0), &source0, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink0), &sink0, sizeof(int), 0, hipMemcpyHostToDevice);
    
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    incremental_kernel_12<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_excess,d_height,cycle);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source0, HIP_SYMBOL(::source0), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink0, HIP_SYMBOL(::sink0), sizeof(int), 0, hipMemcpyDeviceToHost);
  
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag2);


}
void Decremental(graph& g, int V,int E,int* d_meta,int* d_data,int* d_rev_meta,int* d_src,int* d_weight, int source1, int sink1, int* d_residual_capacity, 
  int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess, int cycle
)
{

  const unsigned threadsPerBlock = THREADS_PER_BLOCK;
  unsigned numThreads   = (V < threadsPerBlock)? V: THREADS_PER_BLOCK;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  int* d_height2;
  hipMalloc(&d_height2, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height2,(int)0);

  initIndex<int><<<1,1>>>(V,d_height2,sink1,(int)g.num_nodes()); //InitIndexDevice
  decremental_kernel_15<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_excess);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::sink1), &sink1, sizeof(int), 0, hipMemcpyHostToDevice);
  decremental_kernel_18<<<numBlocks, threadsPerBlock>>>(V,E,d_src,d_rev_meta,d_reverse_edge,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&sink1, HIP_SYMBOL(::sink1), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag3 = true; // asst in .cu

  do{
    flag3 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::sink1), &sink1, sizeof(int), 0, hipMemcpyHostToDevice);
  
    hipMemcpyToSymbol(HIP_SYMBOL(::source1), &source1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag3), &flag3, sizeof(bool), 0, hipMemcpyHostToDevice);
    decremental_kernel_20<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_excess,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity,d_height2,cycle);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&sink1, HIP_SYMBOL(::sink1), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    hipMemcpyFromSymbol(&source1, HIP_SYMBOL(::source1), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag3, HIP_SYMBOL(::flag3), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag3);


}
void recalculate_max_flow(graph& g, std::vector<update> updateBatch, int batchSize, int source, 
  int sink, int cycle)
{
   int V = g.num_nodes();
   int E = g.num_edges();
  int* edgeLen = g.getEdgeLen();
  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;
  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  for(int i=0; i<= V; i++) {
       int temp;
      temp = g.indexofNodes[i];
      h_meta[i] = temp;
     temp = g.rev_indexofNodes[i];
      h_rev_meta[i] = temp;
  }
   for(int i=0; i< E; i++) {
     int temp;
    temp = g.edgeList[i];
      h_data[i] = temp;
      temp = g.srcList[i];
      h_src[i] = temp;
      temp = edgeLen[i];
    h_weight[i] = temp;
  }
  int *d_meta;
  int *d_data;
  int *d_src;
  int *d_weight;
  int *d_rev_meta;
  hipMalloc(&d_meta,sizeof(int)*(V+1));
  hipMalloc(&d_data,sizeof(int)*(E));
  hipMalloc(&d_src,sizeof(int)*(E));
  hipMalloc(&d_weight,sizeof(int)*(E));
  hipMalloc(&d_rev_meta,sizeof(int)*(V+1));
  hipMemcpy(d_meta,h_meta,sizeof(int)*(V+1),hipMemcpyHostToDevice);
  hipMemcpy(d_data,h_data,sizeof(int)*(E),hipMemcpyHostToDevice);
  hipMemcpy(d_src,h_src,sizeof(int)*(E),hipMemcpyHostToDevice);
  hipMemcpy(d_weight,h_weight,sizeof(int)*(E),hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta,h_rev_meta,sizeof(int)*(V+1),hipMemcpyHostToDevice);
  unsigned threadsPerBlock   = (V < THREADS_PER_BLOCK)? V : THREADS_PER_BLOCK;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  staticMaxFlow(g,source,sink,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,cycle);

  int _batchSize = batchSize;
  update *d_updateBatch;
  hipMalloc(&d_updateBatch,sizeof(update)*_batchSize);
  int batchElements = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  for( int updateIndex = 0 ; updateIndex < updateBatch.size() ; updateIndex += _batchSize){
    if((updateIndex + _batchSize) > updateBatch.size())
    {
      batchElements = updateBatch.size() - updateIndex ;
    }
    else
    batchElements = _batchSize ;
    hipMemcpy(d_updateBatch,&updateBatch[updateIndex],batchElements*sizeof(update),hipMemcpyHostToDevice);
    unsigned updateThreads = (batchElements < THREADS_PER_BLOCK)? batchElements: THREADS_PER_BLOCK;
    unsigned updateBlocks = (batchElements+updateThreads-1)/updateThreads;

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    OnAdd_kernel<<<updateBlocks,updateThreads>>>(d_updateBatch,batchElements, d_meta, d_data, d_weight, d_rev_residual_capacity, d_residual_capacity);
    Decremental(g,V,E,d_meta,d_data,d_rev_meta,d_src,d_weight,source,sink,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,cycle);

    Incremental(g,V,E,d_meta,d_data,d_rev_meta,d_src,d_weight,source,sink,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,cycle);



  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

   print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
  int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);
  hipMemcpy(h_excess,d_excess,sizeof(int)*V,hipMemcpyDeviceToHost);
  int val = h_excess[sink];
  free(h_excess);
  hipFree(d_excess);
  printf("DYNAMIC FLOW: %d GPU Time: %.6f ms\n",val, milliseconds);
  hipMemcpy(h_meta,d_meta,sizeof(int)*(V+1),hipMemcpyDeviceToHost);
  hipMemcpy(h_data,d_data,sizeof(int)*(E),hipMemcpyDeviceToHost);
  hipMemcpy(h_src,d_src,sizeof(int)*(E),hipMemcpyDeviceToHost);
  hipMemcpy(h_weight,d_weight,sizeof(int)*(E),hipMemcpyDeviceToHost);
  hipMemcpy(h_rev_meta,d_rev_meta,sizeof(int)*(V+1),hipMemcpyDeviceToHost);
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_src);
  hipFree(d_weight);
  hipFree(d_rev_meta);
}
int  main( int  argc, char** argv) {
  char* originalgraph=argv[1];
  char* updatesinp = argv[2];
  int source = atoi(argv[3]);
  int sink = atoi(argv[4]);
  int kernel = atoi(argv[5]);
  int percent = atoi(argv[6]);
  graph G1(originalgraph,"cuda",true);
  G1.parseGraph();
  int batchsize = (percent*G1.num_edges())/100;

  std::vector<update> updateEdges=G1.parseUpdates(updatesinp);
  printf("batchsize:%d\n, updatesSize:%d\n",batchsize,updateEdges.size());
  recalculate_max_flow(G1,updateEdges,batchsize,source,sink,kernel);
  return 0;
}