#include "hip/hip_runtime.h"
/* v1.0
 * TODO: 1 kernel SSSP and CPU function - Thu, 15-Jul-2021, 12:21:20 IST
 * DONE: 1 kernel SSSP and CPU function - Thu, 15-Jul-2021, 22:24:12 IST
 * TODO: DUBUG LEVEL - Thu, 15-Jul-2021, 22:24:12 IST
 * DONE: DUBUG LEVEL - Thu, 15-Jul-2021, 23:01:02 IST
 * TODO: SEQ BF ALGO - Thu, 15-Jul-2021, 23:01:02 IST
 * DONE: SEQ BF ALGO - Thu, 15-Jul-2021, 23:24:02 IST
 * TODO: SEQ DIJKSTRA ALGO - Fri, 16-Jul-2021, 00:04:43 IST
 * DONE: SEQ DIJKSTRA ALGO - Fri, 16-Jul-2021, 00:22:34 IST
 F* TODO: READ other graph  - Wed, 21-Jul-2021, 01:17:56 IST
 *
 *
 * TODO
 * CLARIFY should wrap into Graph class?
 * CLARIFY should take file as param?
 * IMP should abstract for user
 * IMP should add timers
 */
#include "sssp_dsl.h"

#define DEFAULT 0

#define DEBUG1 if(DEBUGCODE >= 1)
#define DEBUG2 if(DEBUGCODE >= 2)

unsigned DEBUGCODE = 0;

#define cudaCheckError() {                                             \
 hipError_t e=hipGetLastError();                                     \
 if(e!=hipSuccess) {                                                  \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
   exit(0);                                                            \
 }                                                                     \
}

template <typename T>
__global__ void initKernel(unsigned nSize,T* dArray,T initVal){
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < nSize){
    dArray[id]=initVal;
  }
}

// NO Need!
template <typename T>
__global__ void swapPointersKernel(T *gModifiedPrev, T* gModifiedNext){
  T *tempModPtr  = gModifiedNext;
  gModifiedNext = gModifiedPrev;
  gModifiedPrev = tempModPtr;
}


__global__ void SSSPKernel(unsigned nSize, int* csrMeta, int* csrData,int* csrWeight, int* distance, bool* modifiedPrev, bool* modifiedNext, bool* finished){
  unsigned u = threadIdx.x + blockDim.x*blockIdx.x; // tid
  if(u < nSize){ //Only modified u's //&& modified[u]
    if(modifiedPrev[u]) {
      for(int ii=csrMeta[u], end = csrMeta[u+1]; ii < end; ++ii){ //PUSH
        unsigned v = csrData[ii];
        int newDistance = distance[u] + csrWeight[ii] /*edgeweight(uv)*/ ;
        if(newDistance < distance[v]){
          atomicMin(&distance[v],newDistance);
          modifiedNext[v]=true;
          finished[0] = false;
        }
      }
    }
  }
  //~ if(u == 0) {
    //~ for(int i=0; i< 4;++i){
      //~ printf("threadID %d finished[%d]:%s\n",u,i, (finished[i]?"true":"false"));
    //~ }
  //~ }
}
__global__ void printKernel(unsigned nSize, int* distance, bool* modified){
  for(int ii=0; ii < nSize; ++ii){
    printf("modified=%s d[%d]=%d \n", (modified[ii]? " Yes": "No"),ii, distance[ii]);
  }

}

unsigned long long int processSSSP(const unsigned nSize, const unsigned mSize,
int*  csrM,
int*  csrD,
int*  csrW,
int*  distance,
bool* modified)
{
  DEBUG1 std::cout<< "Process SSSP..start" << '\n';
  /**************
   * INITIALISE
   **************/
  // OPT we can do this in kernel
  for(int ii=0; ii< nSize; ++ii){
    distance[ii] = INT_MAX/2;
    modified[ii] = false;
  }

  int source = 0;
  distance[source] = 0    ;
  modified[source] = true ;



  /**********************
   * ALLOCATE on GPU
   **********************/
  int*  gcsrM;
  int*  gcsrD;
  int*  gcsrW;

  int*  gDistance;
  bool* gModifiedNext;
  bool* gModifiedPrev;

  bool* gFinished;

  unsigned nSizeIntPlus1  = sizeof(int) *(nSize+1);
  unsigned nSizeIntBytes  = sizeof(int) *( nSize );
  unsigned mSizeIntBytes  = sizeof(int) *( mSize );
  unsigned nSizeBoolBytes = sizeof(bool)*( nSize );
  unsigned oneBoolBytes   = sizeof(bool)*( 1    );

  bool* finished = (bool*) malloc(oneBoolBytes); //ON CPU

  hipMalloc(&gcsrM, nSizeIntPlus1);
  hipMalloc(&gcsrD, mSizeIntBytes);
  hipMalloc(&gcsrW, mSizeIntBytes);

  hipMalloc(&gDistance     , nSizeIntBytes  );
  hipMalloc(&gModifiedNext , nSizeBoolBytes );
  hipMalloc(&gModifiedPrev , nSizeBoolBytes );

  hipMalloc(&gFinished, oneBoolBytes   );

  /***************
   * COPY TO GPU
   ***************/
  hipMemcpy( gcsrM, csrM, nSizeIntPlus1, hipMemcpyHostToDevice);
  hipMemcpy( gcsrD, csrD, mSizeIntBytes, hipMemcpyHostToDevice);
  hipMemcpy( gcsrW, csrW, mSizeIntBytes, hipMemcpyHostToDevice);

  hipMemcpy( gDistance, distance, nSizeIntBytes , hipMemcpyHostToDevice);
  hipMemcpy( gModifiedPrev, modified, nSizeBoolBytes, hipMemcpyHostToDevice);

  cudaCheckError()

  /***************
   * LAUNCH CONFIG
   ***************/
  const unsigned numOfThreads = 1024;
  const unsigned numOfBlocks  = (nSize+numOfThreads-1)/numOfThreads; //Ceil(n/#threads) with function call
  //hipSetDevice(1);

  //~ hipSetDevice(1);
  DEBUG1 std::cout<< "FPLOCK..start.." << '\n';
  /***************
   * FIX PT COMPUTATION
   ***************/
  int k=0;
  finished[0]= false;
  while(!finished[0]){
    DEBUG2 printf("BEFORE\n");
    DEBUG2 printKernel<<<1,1>>>(nSize, gDistance,gModifiedNext);
    DEBUG2 hipDeviceSynchronize();
    //~ finished[0] = true;
    //~ hipMemcpy( gFinished, finished, oneBoolBytes , hipMemcpyHostToDevice);
    //~ initKernel<bool> <<< 1, 1>>>(1, gFinished, true);
    //~ initKernel<bool> <<<numOfBlocks, numOfThreads>>>(nSize, gModifiedPrev, false); //HAPPEND INSIDE KERNEL
    SSSPKernel<<<numOfBlocks, numOfThreads>>>(nSize, gcsrM, gcsrD, gcsrW, gDistance, gModifiedPrev, gModifiedNext, gFinished);
    //~ hipDeviceSynchronize();
    cudaCheckError()

    //~ thrust::device_ptr<bool> gModPtr = thrust::device_pointer_cast(gModifiedNext);
    //~ finished = (thrust::reduce(gModPtr, gModPtr + nSize, true, thrust::logical_and<bool>())); // NOTE: this go into default stream!!  Thanks Rupesh!

    //~ finished[0] = (thrust::reduce(gModPtr, gModPtr + nSize, false, thrust::logical_or<bool>()));
    //~ finished[0] = !finished;

    /*
     * device_pointer_cast https://stackoverflow.com/questions/33156534/pass-cuda-array-to-thrustinclusive-scan/33156815#33156815
     * https://thrust.github.io/doc/functional_8h_source.html#l00798
     * finished = neg of all ORs   ~(a | b | c ) //not working
     * finished = all AND = (1 1 1 1)
     */

    //~ hipDeviceSynchronize();

    initKernel<bool><<<numOfBlocks, numOfThreads>>>(nSize, gModifiedPrev, false);
    //cudaCheckError()
    hipDeviceSynchronize();
    cudaCheckError()

    //~ swapPointersKernel<bool><<<1,1>>>(gModifiedPrev, gModifiedNext);
    bool *tempModPtr  = gModifiedNext;
    gModifiedNext = gModifiedPrev;
    gModifiedPrev = tempModPtr;
    //~ initKernel<bool> <<<numOfBlocks, numOfThreads>>>(nSize, gModifiedNext, false);
    //~ hipMemcpy( finished, gFinished,  oneBoolBytes , hipMemcpyDeviceToHost);
    cudaCheckError()
    //~ hipDeviceSynchronize();

    // DEBUGGING PURPOSE*******
    ++k;
    DEBUG2 printf("k:%d Finish:%s\n",k, (finished[0]?"True":"False"));
    if(k==nSize){
      DEBUG1 std::cout<< "TERMINATED!";
      break;
    }
    //*************************
  }
  DEBUG1 std::cout<< "FPLOCK..END.." << '\n';

  hipMemcpy( distance, gDistance, nSizeIntBytes , hipMemcpyDeviceToHost);

  DEBUG1 std::cout<< "Process SSSP..Completed" << '\n';
  // TODO: TIMER END
  // TODO: Check with CPU

  unsigned long long int sum=0;
  /***************
   * PRINTING DIST+FINDING HASH
   ***************/
  // TODO comment out in production!
  for(int ii=0; ii < nSize; ++ii){
    DEBUG2 printf("d[%d]=%d\n",ii, distance[ii]);
    sum+=distance[ii];
  }
  //~ DEBUG1 printf("HASH:%lld\n",sum);

  cudaCheckError()
  /***************
   * GPU HOUSEKEEPING
   ***************/
  hipFree(gcsrM);
  hipFree(gcsrD);
  hipFree(gcsrW);

  hipFree(gDistance);
  hipFree(gModifiedPrev);
  hipFree(gModifiedNext);

  hipFree(gFinished);

  free(finished);

  return sum;
}

/**************
 * GRAPH ATTRIBUTE
 **************/
short SHIFT       = 0 ; // starts from 0
short UNDIRECTED  = 1 ;

void readInputGr(
unsigned &nSize,
unsigned &mSize,
std::vector < std::vector <int>> &adjList,
std::map< std::pair<int,int>, int> &W){
  DEBUG1 std::cout<< "Reading inputs.." << '\n';

  std::string line, code, dummy;
  unsigned countEdge = 0;

  while(std::getline(std::cin, line) && !line.empty()){
    std::istringstream iStrSteam(line);
    iStrSteam >> code;
    if(code == "c") //Ignore this line
      continue;
    if(code == "p"){
      //~ p sp 264346 733846
      iStrSteam >> dummy >> nSize >> mSize;
      DEBUG1 std::cout<< nSize << " "<< mSize << '\n';
      adjList.resize(nSize);
    }
    else if( code == "a") {
      //~ a 1 2 803
      unsigned u, v;
      int w;

      iStrSteam >> u >> v >> w;
      //~ std::cout<< u << " " << v << " " << w << '\n';
      adjList[u-1].push_back(v-1);    // NOTE MINUS 1
      W[{u-1,v-1}]=w;                 // NOTE MINUS 1
      countEdge++;
    }
  }
  if(countEdge != mSize){
    std::cout<< "WARN: possibly different #edges" << '\n';
  }
  DEBUG1 std::cout<< "Loaded inputs.." << '\n';
}

void readInputTxt(
unsigned &nSize,
unsigned &mSize,
std::vector < std::vector <int>> &adjList,
std::map< std::pair<int,int>, int> &W){
  DEBUG1 std::cout<< "Reading inputs.." << '\n';

  std::string line, code, dummy;
  unsigned countEdge = 0;


  while(std::getline(std::cin, line) && !line.empty()){
    std::istringstream iStrSteam(line);
    iStrSteam >> dummy >> code ;
    if(dummy == "#" && code != "Nodes:") {
      continue; //Ignore any line with # and no node size
    }
    else if(dummy == "#" && code == "Nodes:"){
      //~ # Nodes: 1965206 Edges: 5533214
      iStrSteam >>  nSize >> dummy >> mSize;
      DEBUG1 std::cout<< nSize << " "<< mSize << '\n';
      adjList.resize(nSize);
    }
    else { //assuming rest of the lines are edges and nSize is comp
      //~ 0	1
      //~ 0	2
      unsigned u, v;

      iStrSteam >> u >> v ;

      //~ std::cout<< u << " " << v << " " << w << '\n';
      adjList[u-SHIFT].push_back(v-SHIFT);        // NOSHIFT
      W[{u-SHIFT,v-SHIFT}]= UNDIRECTED ;          // NOSHIFT

      if(UNDIRECTED) { // added the bi-directional edge also.
        adjList[v-SHIFT].push_back(u-SHIFT);      // NOSHIFT
        W[{v-SHIFT,u-SHIFT}]= UNDIRECTED ;        // NOSHIFT
      }
      countEdge++;
    }
  }
  if(countEdge != mSize){
    std::cout<< "WARN: possibly different #edges" << '\n';
  }
  if(nSize == DEFAULT || mSize == DEFAULT)
    std::cout<< "WARN: possibly Nodes/nSize is not read from files" << '\n';
  DEBUG1 std::cout<< "Loaded inputs.." << '\n';
}

void computeCSR(const unsigned nSize, const unsigned mSize,
const std::vector < std::vector <int>> &adjList,
const std::map< std::pair<int,int>, int> &W,  // DONT use const and LHS = W[]
int* csrM,
int* csrD,
int* csrW){
  DEBUG1 std::cout<< "Computing CSR.." << '\n';
  DEBUG2 std::cout<< "csr[0]=0" << '\n';
  for(int ii=0; ii < nSize; ++ii) {
    csrM[ii+1]= csrM[ii]+(int) adjList[ii].size() ; // NOTE +1
    DEBUG2 std::cout<< "csr"<<"["<<ii+1<< "]= "<< csrM[ii+1] << '\n';
  }

  int u   = 0;
  int idx = 0;
  DEBUG2 std::cout<< "CSRDATA" << '\n';
  for(auto nodes : adjList){
    // u begins with 0
    DEBUG2 std::cout<< u << ":" << ' ';
    for(int v : nodes){
      csrD[idx] = v ;
      csrW[idx] = W.at({u,v});
      DEBUG2 printf("(%d,%d),",csrD[idx],csrW[idx]);
      ++idx;
    }
    DEBUG2 std::cout<<  '\n';
    ++u; //NOTE
  }
  if( idx != mSize ) //* (UNDIRECTED?2:1)
    std::cout<< "WARN: possibly different #edges" << '\n';

}

void printAdjList(const std::vector< std::vector<int> > &graph){
  int i = 0;
  for (auto vec : graph){
    std::cout << i << ": ";
    for(auto v : vec){
      std::cout<< v << " ";
    }
    i++;
    std::cout << std::endl;
  }
}

unsigned long long int BellmanFordMoore(
int nSize,
const std::vector< std::vector<int> > &adjList,
const std::map< std::pair<int,int>, int> &W,
int source = 0
) {
  DEBUG1 std::cout<< "SEQ CPU BFord..start" << '\n';
  std::vector<int> minDist(nSize, INT_MAX/2);
	//~ vector<int> parent(N, -1);
	// all parent are -1
	// all minDist are INT_MAX

	minDist[source] =0;
	bool updated = true;
	for (int k =1; k < (nSize-1) && updated; ++k){
		updated = false;
		DEBUG2 std::cout << k <<" of " << nSize-1 << std::endl;
		for(int u=0, endU = nSize; u < endU; u++){
			for(int j=0, endJ = adjList[u].size(); j < endJ; ++j){
				int v = adjList[u][j];
        int w = W.at({u,v}); // edge weight of (u,v)
				int newDist =  minDist[u]+w;

				if(	newDist < minDist[v] ){ // the to perform relax!
					minDist[v] = newDist;
					//~ parent[v] = u;
					updated=true;
				}
			}
		}
	}
  DEBUG1 std::cout<< "SEQ CPU BFord..end" << '\n';
  unsigned long long int sum = 0;
  for (auto &v : minDist)
    sum += v;

  return sum;
}
unsigned long long int BellmanFordMooreDSL(
int nSize,
const std::vector< std::vector<int> > &adjList,
const std::map< std::pair<int,int>, int> &W,
int source = 0
) {
  DEBUG1 std::cout<< "SEQ CPU BFord..start" << '\n';
  std::vector<int> minDist(nSize, INT_MAX/2);
  std::vector<bool> modifiedPrev(nSize, false);
  std::vector<bool> modifiedNext(nSize, false);
	//~ vector<int> parent(N, -1);
	// all parent are -1
	// all minDist are INT_MAX
  modifiedPrev[source]=true;
	minDist[source] =0;
	bool updated = true;
	for (int k =1; k < (nSize-1) && updated; ++k){ //
		updated = false;
		DEBUG2 std::cout << k <<" of " << nSize-1 << std::endl;
		for(int u=0, endU = nSize; u < endU; u++){
      if(modifiedPrev[u]){
        std::cout<< "k:"<< k << " u:"<< u << '\n';
        for(int j=0, endJ = adjList[u].size(); j < endJ; ++j){
          int v = adjList[u][j];
          int w = W.at({u,v}); // edge weight of (u,v)
          int newDist =  minDist[u]+w;

          if(	newDist < minDist[v] ){ // the to perform relax!
            minDist[v] = newDist;
            //~ parent[v] = u;
            modifiedNext[v]=true;
            updated=true;
          }
        }
      }
    }
    modifiedNext.swap(modifiedPrev);
    fill(modifiedNext.begin(),modifiedNext.end(),false);
	}
  DEBUG1 std::cout<< "SEQ CPU BFord..end" << '\n';
  unsigned long long int sum = 0;
  for (auto &v : minDist)
    sum += v;

  return sum;
}


unsigned long long int Dijkstra( int nSize,
const std::vector< std::vector<int> > &adjList,
const std::map< std::pair<int,int>, int> &W,
std::vector<int> &minDist,
int source=0
) {

  DEBUG1 std::cout<< "SEQ CPU Dijkstra..start" << '\n';

  //~ std::vector<int> minDist(nSize, INT_MAX/2);
	//~ vector<int> parent(graph.size() , -1);
  minDist[ source ] = 0;
  std::set< std::pair<int,int> > active_vertices;
  active_vertices.insert( {0,source} );

  while (!active_vertices.empty()) {
    int u = active_vertices.begin()->second;

    active_vertices.erase( active_vertices.begin() );
    for (auto v : adjList[u]) {
      auto newdist = minDist[u] + W.at({u,v});
      if (newdist < minDist[v]) {
        active_vertices.erase( { minDist[v], v } );
        minDist[v] = newdist;
        //~ parent[v] = u;
        active_vertices.insert( { newdist, v } );
      }
    }
  }

  DEBUG1 std::cout<< "SEQ CPU Dijkstra..end" << '\n';
  unsigned long long int sum = 0;
  for (auto &v : minDist)
    sum += v;

  return sum;
}
int check(int *gDist, std::vector<int> cDist, int nSize){
  DEBUG1 std::cout<< "CHECK..start" << '\n';
  for(int ii=0; ii < nSize; ++ii){
    if(gDist[ii] != cDist[ii]){
      DEBUG1 std::cout<< "Diff at "<< ii << '\n';
      DEBUG2 printf("(G,C):(%d,%d)\n",gDist[ii], cDist[ii]);
      return ii + 1; // NOTE +1
    }
  }
  std::cout<< "Perfect!" << '\n';
  return 0;
}

int main(int argc, char** argv){

  if(argc > 0){
    DEBUGCODE = argc-1;
    DEBUG1 std::cout<< "DEBUG LEVEL:" << DEBUGCODE << '\n';
  }

  unsigned int nSize = DEFAULT;
  unsigned int mSize = DEFAULT;

  std::map< std::pair<int,int>, int> W;
  std::vector < std::vector <int>> adjList;

  readInputGr  (nSize, mSize, adjList, W);
  DEBUG2 printAdjList(adjList);

  int* csrM = (int*) malloc(sizeof(int)*(nSize+1)); //NOTE +1
  int* csrD = (int*) malloc(sizeof(int)*( mSize ));
  int* csrW = (int*) malloc(sizeof(int)*( mSize ));

  int*  distance = (int*)  malloc(sizeof(int) * nSize);
  bool* modified = (bool*) malloc(sizeof(bool)* nSize);
  //~ int*  parent= (int*) malloc(sizeof(int)*nSize);
  std::vector<int> cpuDistance(nSize, INT_MAX/2);

  computeCSR (nSize, mSize, adjList, W, csrM, csrD, csrW);
  //~ initialize ()
  auto gpuHash = processSSSP(nSize, mSize, csrM, csrD, csrW, distance, modified);
  auto cpuHash = Dijkstra(nSize, adjList, W,cpuDistance);
  //~ auto cpuHash = BellmanFordMoore(nSize, adjList, W);
  //~ auto cpuHash = BellmanFordMooreDSL(nSize, adjList, W);
  DEBUG1 printf("(G,C): (%lld,%lld)", gpuHash, cpuHash);
  printf(":%s\n", (gpuHash!=cpuHash?"False": "True"));
  //deallocate(csrM, csrD, csrW, distance, modified););

  check(distance,cpuDistance,nSize);

  free(csrM);
  free(csrD);
  free(csrW);
  free(distance);
  free(modified);

  return 0;
}
