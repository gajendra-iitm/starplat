// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "barenya.h"

void do_max_flow(graph& g,int source,int sink,int kernel_parameter
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_label;
  hipMalloc(&d_label, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_label,(int)0);

  initIndex<int><<<1,1>>>(V,d_label,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity);
  hipDeviceSynchronize();



  int excesstotal = 0; // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(::excesstotal), &excesstotal, sizeof(int), 0, hipMemcpyHostToDevice);
  do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(&excesstotal, HIP_SYMBOL(::excesstotal), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  bool flag = true; // asst in .cu

  do{
    int x = kernel_parameter; // asst in .cu

    do{
      x = x + -1;
      hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
      do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_residual_capacity,d_label);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu


    }while(x > 0);
    if (excess[source] + excess[sink] == excesstotal){ // if filter begin 
      flag = false;

    } // if filter end

  }while(flag);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_excess);
  hipFree(d_label);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
