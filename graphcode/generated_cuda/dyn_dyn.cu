// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "dyn_dyn.h"
#define THREADS_PER_BLOCK 1024
void recalculate_max_flow(graph& g, std::vector<update> updateBatch, int batchSize, int source, 
  int sink)
{
  int E = g.num_edges();
  int *d_meta, *d_data,*d_weight;
  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int _batchSize = batchSize;
  update *d_updateBatch;
  hipMalloc(&d_updateBatch,sizeof(update)*_batchSize);
  int batchElements = 0;
  for( int updateIndex = 0 ; updateIndex < updateBatch.size() ; updateIndex += _batchSize){
    if((updateIndex + _batchSize) > updateBatch.size())
    {
      batchElements = updateBatch.size() - updateIndex ;
    }
    else
    batchElements = _batchSize ;
    hipMemcpy(d_updateBatch,&updateBatch[updateIndex],batchElements*sizeof(update),hipMemcpyHostToDevice);
    unsigned updateThreads = (batchElements < THREADS_PER_BLOCK)? batchElements: THREADS_PER_BLOCK;
    unsigned updateBlocks = (batchElements+updateThreads-1)/updateThreads;

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    OnAdd_kernel<<<updateBlocks,updateThreads>>>(d_updateBatch,batchElements, d_meta, d_data, d_weight, d_residual_capacity, d_rev_residual_capacity);


  }


}
