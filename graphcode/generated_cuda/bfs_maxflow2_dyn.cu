// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "bfs_maxflow2.h"

void computeBC(graph& g,int n,int source,int* d_residual_capacity,
  int* d_rev_residual_capacity,int* d_height,int* d_reverse_edge)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)n);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)0); //InitIndexDevice
  bool* d_scanned;
  hipMalloc(&d_scanned, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_scanned,(bool)false);

  initIndex<bool><<<1,1>>>(V,d_scanned,source,(bool)true); //InitIndexDevice

  //EXTRA vars for ITBFS AND REVBFS
  bool finished;
  int hops_from_source=0;
  bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
  int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));  hipMemset(d_hops_from_source,0,sizeof(int));
  int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

  //EXTRA vars INITIALIZATION
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_level,-1);
  initIndex<int><<<1,1>>>(V,d_level,source, 0);

  // long k =0 ;// For DEBUG
  do {
    finished = true;
    hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

    //Kernel LAUNCH
    fwd_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished,d_residual_capacity,d_rev_residual_capacity,d_height,d_reverse_edge); ///DONE from varList

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    incrementDeviceVar<<<1,1>>>(d_hops_from_source);
    hipDeviceSynchronize(); //MUST - rupesh
    ++hops_from_source; // updating the level to process in the next iteration
    // k++; //DEBUG

    hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
  }while(!finished);

  hops_from_source--;
  hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

  //BACKWARD PASS
  while(hops_from_source > 1) {

    //KERNEL Launch
    back_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished
      ,d_residual_capacity,d_rev_residual_capacity,d_height,d_reverse_edge); ///DONE from varList

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
  }
  //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_scanned);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
