// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "bc_dslV2.h"

#define CUDA_CHECK(err) { \
  if (err != hipSuccess) { \
      fprintf(stderr, "CUDA error at %s: line:%d: Error number: %d. Error name: %s. Error description: %s.\n", __FILE__, __LINE__, (int)err, hipGetErrorName(err), hipGetErrorString(err)); \
      exit(err); \
  } \
}

// #define CUDA_CHECK(err) { \
//   if (err != hipSuccess) { \
//       fprintf(stderr, "CUDA error at %d: %s. Error number: %d. Error name: %s. Error description: %s.\n", __FILE__, __LINE__, hipGetErrorString(err), (int)err, hipGetErrorName(err)); \
//       exit(err); \
//   } \
// }

//hipError_t err = hipGetLastError();

void Compute_BC(graph& g,float* BC,std::set<int>& sourceSet)

{
  hipError_t err = hipGetLastError();
  CUDA_CHECK(err);
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_BC;
  hipMalloc(&d_BC, sizeof(float)*(V));
  err = hipGetLastError();
  CUDA_CHECK(err);

  //BEGIN DSL PARSING 
  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_BC,(float)0);
  err = hipGetLastError();
  CUDA_CHECK(err);
  //MILESTONE 1: reached, initKernel device function throwing CUDA error "named symbol not found"
  //print numblocks here
  float* d_sigma;
  hipMalloc(&d_sigma, sizeof(float)*(V));

  float* d_delta;
  hipMalloc(&d_delta, sizeof(float)*(V));

  //FOR SIGNATURE of SET - Assumes set for on .cu only
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++) 
  {
    hipError_t err = hipGetLastError();
    CUDA_CHECK(err);
    int src = *itr;
    //print numblocks here

    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_delta,(float)0);
    //print numblocks here
    err = hipGetLastError();
    CUDA_CHECK(err);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_sigma,(float)0);
    err = hipGetLastError();
    CUDA_CHECK(err);

    initIndex<float><<<1,1>>>(V,d_sigma,src,(float)1); //InitIndexDevice
    err = hipGetLastError();
    CUDA_CHECK(err);

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;
    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));  hipMemset(d_hops_from_source,0,sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    //EXTRA vars INITIALIZATION
    printf("numBlocks: %d, threadsPerBlock: %d\n");
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_level,-1);
    // Check for errors during kernel launch
    err = hipGetLastError();
    CUDA_CHECK(err);

    initIndex<int><<<1,1>>>(V,d_level,src, 0);
    // Check for errors during kernel launch
    //err = hipGetLastError();
    CUDA_CHECK(err);

    // long k =0 ;// For DEBUG
    do {
      finished = true;
      hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished,d_BC); ///DONE from varList
      // Check for errors during kernel launch
      //hipError_t err = hipGetLastError();
      CUDA_CHECK(err);

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      // Check for errors during kernel launch
      CUDA_CHECK(err);
  
            // Synchronize the device to catch errors that occur during kernel execution
      err = hipDeviceSynchronize();
      CUDA_CHECK(err); //MUST - rupesh

      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    }while(!finished);

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished
        ,d_BC); ///DONE from varList
      // Check for errors during kernel launch
      //hipError_t err = hipGetLastError();
      CUDA_CHECK(err);

      //err = hipDeviceSynchronize();
      CUDA_CHECK(err);

      hops_from_source--;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
    }
    //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);

  }

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_delta);
  hipFree(d_sigma);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(      BC,     d_BC, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
