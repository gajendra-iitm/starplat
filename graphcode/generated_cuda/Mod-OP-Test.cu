// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "Mod-OP-Test.h"

void tst(int a,int b)

{

  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int c = a % b; // asst in .cu


  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
