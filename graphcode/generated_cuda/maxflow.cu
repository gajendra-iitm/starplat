// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "maxflow.h"

void maxflow(graph& g,int s,int t)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_cf;
  hipMalloc(&d_cf, sizeof(int)*(E));

  int* d_h;
  hipMalloc(&d_h, sizeof(int)*(V));

  int* d_e;
  hipMalloc(&d_e, sizeof(int)*(V));

  int ExcessTotal = 0; // asst in .cu

  int N = g.num_nodes(); // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::s), &s, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(::N), &N, sizeof(int), 0, hipMemcpyHostToDevice);
  maxflow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_cf,d_e,d_h);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&s, HIP_SYMBOL(::s), sizeof(int), 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(&N, HIP_SYMBOL(::N), sizeof(int), 0, hipMemcpyDeviceToHost);



  int v = 0; // asst in .cu

  do{
    if (v == s){ // if filter begin 
      d_h[v] = N;
      d_e[v] = 0;

    } // if filter end
    else
    d_h[v] = 0;
    d_e[v] = 0;

    int e1 = edge;
    d_cf[e1] = weight[e1];
    if (cf[e1] == -1){ // if filter begin 
      d_cf_backward[e1] = 0;

    } // if filter end

  } //  end FOR NBR ITR. TMP FIX!
  v = v + 1;

}while(v < N);
bool finished = false; // asst in .cu

do{
  hipMemcpyToSymbol(HIP_SYMBOL(::v), &v, sizeof(int), 0, hipMemcpyHostToDevice);
  maxflow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_h,d_e);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&v, HIP_SYMBOL(::v), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu


}while(finished);

//hipFree up!! all propVars in this BLOCK!
hipFree(d_e);
hipFree(d_h);

//TIMER STOP
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
