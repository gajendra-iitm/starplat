// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "dynamic.h"

void do_max_flow(graph& g,int source,int sink,int kernel_parameter
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_label;
  hipMalloc(&d_label, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_label,(int)0);

  initIndex<int><<<1,1>>>(V,d_label,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_excess);
  hipFree(d_label);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
