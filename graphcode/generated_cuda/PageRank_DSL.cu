#include "hip/hip_runtime.h"
#include"PageRank_DSL.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  int* gpu_OA;
  int* gpu_edgeList;
  int* gpu_edgeList;

  hipMalloc(&gpu_OA, sizeof(int)*(1+V));
  hipMalloc(&gpu_edgeList, sizeof(int)*(E));
  hipMalloc(&gpu_edgeList, sizeof(int)*(E));

  if( V <= 1024)
  {
    block_size = V;
    num_blocks = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  hipMemcpy(&d_gpu_OA,OA, sizeof(int)*(1+V), hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  Compute_PR_kernel<<<num_blocks, block_size>>>(gpu_OA, gpu_edgeList, V, E ;
    hipDeviceSynchronize();


    float num_nodes = (float)g.num_nodes( );
    for (int t = 0; t < V; g ++) 
    {
      pageRank[t] = 1 / num_nodes;
    }
    int iterCount = 0;
    float diff = 0.0 ;
    do
    diff = 0.000000;
    iterCount++;
    while((diff > beta) && (iterCount < maxIter));
}