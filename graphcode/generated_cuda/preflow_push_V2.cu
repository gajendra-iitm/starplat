// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "preflow_push_V2.h"

void do_max_flow(graph& g,int source,int sink,int kernel_parameter
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_label;
  hipMalloc(&d_label, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_label,(int)0);

  initIndex<int><<<1,1>>>(V,d_label,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity);
  hipDeviceSynchronize();



  int excesstotal = 0; // asst in .cu

  int* d_visit;
  hipMalloc(&d_visit, sizeof(int)*(V));

  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(::excesstotal), &excesstotal, sizeof(int), 0, hipMemcpyHostToDevice);
  do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(&excesstotal, HIP_SYMBOL(::excesstotal), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag = true; // asst in .cu

  do{
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::kernel_parameter), &kernel_parameter, sizeof(int), 0, hipMemcpyHostToDevice);
    do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_label,d_residual_capacity);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&kernel_parameter, HIP_SYMBOL(::kernel_parameter), sizeof(int), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess,d_label);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visit,(int)0);

    d_label[sink] = 0;
    d_visit[sink] = 1;
    int finished = 0; // asst in .cu

    do{
      finished = 0;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(int), 0, hipMemcpyHostToDevice);
      do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_src,d_rev_meta,d_residual_capacity,d_label,d_visit);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(int), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


    }while(finished > 0);
    hipMemcpyToSymbol(HIP_SYMBOL(::excesstotal), &excesstotal, sizeof(int), 0, hipMemcpyHostToDevice);
    do_max_flow_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_visit,d_excess,d_label);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&excesstotal, HIP_SYMBOL(::excesstotal), sizeof(int), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    if (excess[source] + excess[sink] == excesstotal){ // if filter begin 
      flag = false;

    } // if filter end

  }while(flag);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_visit);
  hipFree(d_excess);
  hipFree(d_label);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
