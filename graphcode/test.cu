#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "graph.cpp"
using namespace std;

env environment("cuda", "static", "test.cu");

class Layer
{
public:
	int32_t num_features;
	double *weights;
	double *bias;
	double epsilon = 0.01;
	double grad_epsilon;
	double *aggregatedFeatures;
	double *preActivatedFeatures;
	double *postActivatedFeatures;
	double *grad_pre_act_output;
	double *grad_weights;
	double *grad_bias;
	double *m_weights;
	double *m_biases;
	double m_epsilon = 0.0f;
	double v_epsilon = 0.0f;
	double *v_weights;
	double *v_biases;

	Layer() : num_features(0), weights(nullptr), bias(nullptr), epsilon(0.01), grad_epsilon(0.0), aggregatedFeatures(nullptr), preActivatedFeatures(nullptr), postActivatedFeatures(nullptr), grad_pre_act_output(nullptr), grad_weights(nullptr), grad_bias(nullptr), m_weights(nullptr), m_biases(nullptr), m_epsilon(0.0f), v_epsilon(0.0f), v_weights(nullptr), v_biases(nullptr) {}

	__device__ void xaviers(double *weights, int num_feat_current, int num_feat_prev)
	{
		double x = sqrt(6.0f / (num_feat_current + num_feat_prev));
		hiprandState state;
		// printf("num_feat_current: %d\n", num_feat_current);
		hiprand_init(42, 5, 0, &state);
		for (int i = 0; i < num_feat_current * num_feat_prev; i++)
		{
			hiprandState state;
			hiprand_init(42, i, 0, &state);
			float random_value = hiprand_uniform(&state) * x;
			weights[i] = random_value;
			// printf("%f\n",weights[i]);
		}
	}

	__device__ void he(double *weights, int num_feat_current, int num_feat_prev)
	{
		double x = sqrt(6.0f / (num_feat_prev));
		hiprandState state;
		// printf("num_feat_current: %d\n", num_feat_current);
		hiprand_init(3566, 5, 0, &state);
		for (int i = 0; i < num_feat_current * num_feat_prev; i++)
		{
			hiprandState state;
			hiprand_init(3566, i, 0, &state);
			float random_value = hiprand_uniform(&state) * x;
			weights[i] = random_value;
			// printf("%f\n",weights[i]);
		}
	}
};

class cudaVars
{
public:
	Layer *layers;
	int *d_nodesPtr;
	int *d_edgesList;
	float *d_edgeWeights;
	int *d_y_true;
};

cudaVars globalCudaVar;

__global__ void initializeLayerFields(Layer *layers, int num_layers, int *num_features, double *d_features, int num_nodes, int initialization_type)
{
	int i = threadIdx.x;
	if (i == 0)
	{
		layers[i].num_features = num_features[i];
		// printf("num_features: %d and i = %d\n", layers[i].num_features, i);
		for (int j = 0; j < num_features[0] * num_nodes; j++)
		{
			layers[i].preActivatedFeatures[j] = 0;
			layers[i].postActivatedFeatures[j] = d_features[j];
			// printf("postActivatedFeatures: %f\n", layers[i].postActivatedFeatures[j]);
			// printf("preActivatedFeatures: %f\n", layers[i].preActivatedFeatures[j]);
		}
	}

	else if (i < num_layers)
	{
		layers[i].num_features = num_features[i];
		if (initialization_type == 1)
		{
			layers[i].xaviers(layers[i].weights, num_features[i - 1], num_features[i]);
		}
		else if (initialization_type == 2)
		{
			layers[i].he(layers[i].weights, num_features[i - 1], num_features[i]);
		}

		for (int j = 0; j < num_features[i]; j++)
		{
			layers[i].bias[j] = 0.5;
			layers[i].grad_bias[j] = 0;
			layers[i].m_biases[j] = 0;
			layers[i].v_biases[j] = 0;
		}

		for (int j = 0; j < num_features[i - 1] * num_features[i]; j++)
		{
			layers[i].grad_weights[j] = 0;
			layers[i].m_weights[j] = 0;
			layers[i].v_weights[j] = 0;
		}

		for (int j = 0; j < num_features[i - 1] * num_nodes; j++)
		{
			layers[i].aggregatedFeatures[j] = 0;
		}

		for (int j = 0; j < num_features[i] * num_nodes; j++)
		{
			layers[i].preActivatedFeatures[j] = 0;
			layers[i].postActivatedFeatures[j] = 0;
			layers[i].grad_pre_act_output[j] = 0;
		}
	}
}

void initializeLayers_cuda(GNN &gnn, Layer *d_layers, vector<int> numFeaturesPerLayer, double *d_features, int num_nodes, const char *transformation)
{
	int *d_numFeaturesPerLayer;
	hipMalloc(&d_numFeaturesPerLayer, numFeaturesPerLayer.size() * sizeof(int));
	hipMemcpy(d_numFeaturesPerLayer, numFeaturesPerLayer.data(), numFeaturesPerLayer.size() * sizeof(int), hipMemcpyHostToDevice);

	int initialization_type;
	if (transformation == "xaviers")
	{
		initialization_type = 1;
	}
	else if (transformation == "he")
	{
		initialization_type = 2;
	}

	initializeLayerFields<<<1, numFeaturesPerLayer.size()>>>(d_layers, numFeaturesPerLayer.size(), d_numFeaturesPerLayer, d_features, num_nodes, initialization_type);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	}

	globalCudaVar.layers = d_layers;

	hipMalloc(&globalCudaVar.d_nodesPtr, gnn.getGraph().num_nodes() * sizeof(int));
	hipMemcpy(globalCudaVar.d_nodesPtr, gnn.getGraph().getNodesptr(), gnn.getGraph().num_nodes() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_edgesList, gnn.getGraph().num_edges() * sizeof(int));
	hipMemcpy(globalCudaVar.d_edgesList, gnn.getGraph().getEdgesList(), gnn.getGraph().num_edges() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_edgeWeights, gnn.getGraph().num_edges() * sizeof(float));
	hipMemcpy(globalCudaVar.d_edgeWeights, gnn.getGraph().getEdgeLen(), gnn.getGraph().num_edges() * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_y_true, gnn.getGraph().num_nodes() * sizeof(int));
	hipMemcpy(globalCudaVar.d_y_true, gnn.getLabels().data(), gnn.getGraph().num_nodes() * sizeof(int), hipMemcpyHostToDevice);

	// copy the features to a double  and print the features of node 4

	// copy the weights of layer 1 to a double array and print the first 10 weights
	//  double *weights = new double[numFeaturesPerLayer[0] * numFeaturesPerLayer[1]];
	//  hipMemcpy(weights, h_layers[2].weights, numFeaturesPerLayer[0] * numFeaturesPerLayer[1] * sizeof(double), hipMemcpyDeviceToHost);
	//  for (int i = 0; i < 10; i++)
	//  {
	//  	cout << weights[i] << " ";
	//  }
}

__global__ void GCN_aggregate_cuda(Layer *layers, int batchNumber, int batchSize, int *V, int *E, float *edgeWeights, int layerNumber)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		int prevNumFeatures = layers[layerNumber - 1].num_features;
		int idx = batchNumber * batchSize + tid;
		// if (idx == 0)
		// printf("NUM FEATURES: %d and layernumber %d\n", layers[layerNumber - 1].num_features, layerNumber);
		for (int i = 0; i < prevNumFeatures; i++)
		{
			layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + i] = 0;
		}

		for (int j = V[idx]; j < V[idx + 1]; j++)
		{
			int neighbor = E[j];
			double weight = edgeWeights[j];
			for (int i = 0; i < prevNumFeatures; i++)
			{
				layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + i] += layers[layerNumber - 1].postActivatedFeatures[neighbor * prevNumFeatures + i] * weight;
			}
		}
		// if(epoch>1)
		// printf("Node %d: aggregatedFeatures = %f\n", idx, layers[layerNumber].aggregatedFeatures[idx * layers[layerNumber].num_features]);
	}
}

__global__ void printprint(Layer *layers, int batchNumber, int batchSize, int layerNumber)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid == 0)
	{
		printf("printprint tid: %d\n", tid);
		// nested for for print of aggregated features
		for (int node = 0; node < batchSize; node++)
		{
			for (int featIdx = 0; featIdx < layers[layerNumber - 1].num_features; featIdx++)
			{
				printf("Node %d: aggregatedFeatures %d = %f\n", node, featIdx, layers[layerNumber].aggregatedFeatures[node * layers[layerNumber].num_features + featIdx]);
			}
		}
	}
}

__global__ void NeuralNetworkOperation(Layer *layers, int batchNumber, int batchSize, int totalLayers, int layerNumber, int activationType)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("NN tid: %d\n", tid);

		int idx = batchNumber * batchSize + tid;
		int currNumFeatures = layers[layerNumber].num_features;
		int prevNumFeatures = layers[layerNumber - 1].num_features;
		if (layerNumber < totalLayers)
			for (int i = 0; i < currNumFeatures; i++)
			{
				float sum = 0.0f;

				for (int j = 0; j < prevNumFeatures; j++)
				{
					sum += layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + j] * layers[layerNumber].weights[j * currNumFeatures + i];
				}

				layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i] = sum + layers[layerNumber].bias[i];
				if (layerNumber == 1 && idx == 0)
				{
					//// printf("preActivatedFeatures: %f\n", layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i]);
				}
				if (layerNumber == totalLayers - 1)
				{
					// softmax
					// softmax(layers[layerNumber].postActivatedFeatures[node], layers[layerNumber].num_features, layers[layerNumber].postActivatedFeatures[node]);
				}

				else if (activationType == 1)
				{
					layers[layerNumber].postActivatedFeatures[idx * currNumFeatures + i] =
						fmaxf(0.0f, layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i]);
				}
				else if (activationType == 2)
				{
					layers[layerNumber].postActivatedFeatures[idx * currNumFeatures + i] =
						tanh(layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i]);
				}
				// printf("Node %d: preActivatedFeatures = %f\n", idx, layers[layerNumber].preActivatedFeatures[idx * layers[layerNumber + 1].num_features + i]);

				// printf("Node %d: postActivatedFeatures = %f\n", idx, layers[layerNumber].postActivatedFeatures[idx * layers[layerNumber + 1].num_features + i]);
			}
	}
}

__global__ void computeGradientsOutputLayer(Layer *layers, int *y_true, int batchNumber, int batchSize, int num_classes, int num_layers)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("tid: %d\n", tid);

		int idx = batchNumber * batchSize + tid;
		int label = y_true[idx];

		double *y_pred = layers[num_layers - 1].postActivatedFeatures;
		// double *grad_output = globalCudaVar.layers[num_layers - 1].grad_pre_act_output;

		for (int j = 0; j < num_classes; ++j)
		{
			// grad_output[idx * num_classes + j] = y_pred[idx * num_classes + j] - (label == j ? 1.0f : 0.0f);
			layers[num_layers - 1].grad_pre_act_output[idx * num_classes + j] = y_pred[idx * num_classes + j] - (label == j ? 1.0f : 0.0f);
			//// printf("grad_pre_act_output of layer %d entry %d: %f\n", num_layers - 1, idx * num_classes + j, layers[num_layers - 1].grad_pre_act_output[idx * num_classes + j]);
		}
	}
}

__global__ void computeGradientsIntermediateLayer(Layer *layers, int batchNumber, int batchSize, int layer_num, int activation_type)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("tid: %d\n", tid);

		int current_num_features = layers[layer_num].num_features;
		int next_num_features = layers[layer_num + 1].num_features;

		int idx = batchNumber * batchSize + tid;
		for (int i = 0; i < current_num_features; ++i)
		{
			layers[layer_num].grad_pre_act_output[idx * current_num_features + i] = 0.0f;
			for (int j = 0; j < next_num_features; ++j)
			{
				layers[layer_num].grad_pre_act_output[idx * current_num_features + i] += layers[layer_num + 1].grad_pre_act_output[idx * next_num_features + j] * layers[layer_num + 1].weights[i * next_num_features + j];
			}
			// printf("grad_pre_act_output of layer %d entry %d: %f\n", layer_num, idx * current_num_features + i, layers[layer_num].grad_pre_act_output[idx * current_num_features + i]);
			if (activation_type == 1)
			{ // ReLU
				layers[layer_num].grad_pre_act_output[idx * current_num_features + i] *= (layers[layer_num].preActivatedFeatures[idx * current_num_features + i] > 0) ? 1 : 0;
			}
			else if (activation_type == 2)
			{ // Tanh
				layers[layer_num].grad_pre_act_output[idx * current_num_features + i] *= (1 - layers[layer_num].postActivatedFeatures[idx * current_num_features + i] * layers[layer_num].postActivatedFeatures[idx * current_num_features + i]);
			}
		}
	}
}

__global__ void computeWeightBiasGradients(Layer *layers, int layer_num, int num_nodes, double *weightNorm, double *biasNorm)
{
	int featIdx = blockIdx.x * blockDim.x + threadIdx.x;

	int prev_features = layers[layer_num - 1].num_features;
	if (featIdx < prev_features)
	{
		int curr_num_features = layers[layer_num].num_features;
		for (int j = 0; j < curr_num_features; ++j)
		{
			double weight_grad_sum = 0.0f;
			double bias_grad_sum = 0.0f;

			for (int idx = 0; idx < num_nodes; ++idx)
			{
				weight_grad_sum += layers[layer_num].aggregatedFeatures[idx * prev_features + featIdx] * layers[layer_num].grad_pre_act_output[idx * curr_num_features + j];
				bias_grad_sum += layers[layer_num].grad_pre_act_output[idx * curr_num_features + j];
			}

			layers[layer_num].grad_weights[featIdx * curr_num_features + j] = weight_grad_sum;
			*weightNorm += weight_grad_sum * weight_grad_sum;

			// printf("weightNorm: %f\n", weightNorm);
			if (featIdx == 0)
			{
				layers[layer_num].grad_bias[j] = bias_grad_sum;
				*biasNorm += bias_grad_sum * bias_grad_sum;
			}
		}
		// printf("grad_bias of layer %d entry %d: %f\n", layer_num, featIdx, layers[layer_num].grad_bias[featIdx]);
	}
}

__global__ void clipGradients(Layer *layers, int layer_num, double clip_value, double *weightNorm, double *biasNorm)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	double weightNorm1 = sqrtf(*weightNorm);
	double biasNorm1 = sqrtf(*biasNorm);
	// printf("weightNorm: %f\n", weightNorm1);
	int num_input_features = layers[layer_num - 1].num_features;
	int num_output_features = layers[layer_num].num_features;
	if (tid < num_output_features)
	{
		for(int i = 0; i < num_input_features; i++)
		{
			int weight_idx = i * num_output_features + tid;
			if (weightNorm1 > clip_value)
			{
				layers[layer_num].grad_weights[weight_idx] *= clip_value / weightNorm1;
				// printf("grad_weights of layer %d entry %d: %f\n", layer_num, weightIdx, layers[layer_num].grad_weights[weightIdx]);
			}
		}
		if (biasNorm1 > clip_value)
		{
			layers[layer_num].grad_bias[tid] *= clip_value / biasNorm1;
		}
	}
}

__global__ void adam(Layer *layers, int num_layers, int batchNumber, int batchSize, double learning_rate, double beta1, double beta2, double epsilon, int t)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = 1; i < num_layers; i++)
	{
		int numfeats = layers[i].num_features;

		if (tid < numfeats)
		{
			int j = tid;

			double beta1_pow_t = pow(beta1, t); // Precompute beta1^t
			double beta2_pow_t = pow(beta2, t); // Precompute beta2^t
												// int num_features = layers[i].num_features;
			// Update biased first moment estimate for biases j =
			layers[i].m_biases[j] = beta1 * layers[i].m_biases[j] + (1 - beta1) * layers[i].grad_bias[j];
			// Update biased second raw moment estimate for biases
			layers[i].v_biases[j] = beta2 * layers[i].v_biases[j] + (1 - beta2) * layers[i].grad_bias[j] * layers[i].grad_bias[j];

			// Correct bias for the first and second moments (bias correction)
			double m_bias_hat = layers[i].m_biases[j] / (1 - beta1_pow_t);
			double v_bias_hat = layers[i].v_biases[j] / (1 - beta2_pow_t);

			// Update biases
			layers[i].bias[j] -= learning_rate * m_bias_hat / (sqrt(v_bias_hat) + epsilon);

			for (int k = 0; k < layers[i - 1].num_features; k++)
			{
				int weight_idx = k * layers[i].num_features + j;

				// Update biased first moment estimate for weights
				layers[i].m_weights[weight_idx] = beta1 * layers[i].m_weights[weight_idx] + (1 - beta1) * layers[i].grad_weights[weight_idx];
				// Update biased second raw moment estimate for weights
				layers[i].v_weights[weight_idx] = beta2 * layers[i].v_weights[weight_idx] + (1 - beta2) * layers[i].grad_weights[weight_idx] * layers[i].grad_weights[weight_idx];

				// Correct bias for the first and second moments (bias correction)
				double m_weight_hat = layers[i].m_weights[weight_idx] / (1 - beta1_pow_t);
				double v_weight_hat = layers[i].v_weights[weight_idx] / (1 - beta2_pow_t);

				// Update weights
				layers[i].weights[weight_idx] -= learning_rate * m_weight_hat / (sqrt(v_weight_hat) + epsilon);
				//// printf("weights of layer %d entry %d: %f\n", i, weight_idx, layers[i].weights[weight_idx]);
			}
		}
	}
}

// __global__ void calculateAccuracy(Layer *layers, int *ground_truth, int *correct_predictions, int num_classes, int batchNumber, int batchSize)
__global__ void calculateAccuracy(Layer *layers, int *ground_truth, int *correct_predictions, int num_classes, int num_nodes)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// printf("JEIIJ\n");
	if (idx < num_nodes)
	{
		double *predictions = layers[1].preActivatedFeatures;
		// Find the index of the maximum predicted class (argmax)
		int predicted_class = 0;
		double max_value = predictions[idx * num_classes];

		for (int i = 0; i < num_classes; ++i)
		{
			if (predictions[idx * num_classes + i] > max_value)
			{
				max_value = predictions[idx * num_classes + i];
				// printf("max_value: %f\n", max_value);
				predicted_class = i;
			}
		}
		// printf("max_value: %f\n", max_value);

		// Compare with the ground truth
		if (predicted_class == ground_truth[idx])
		{
			atomicAdd(correct_predictions, 1); // Increment the counter for correct predictions
		}
	}
}

void forward(GNN &gnn, int layerNumber, int batchNumber, int batchSize, int activationType)
{
	if (layerNumber == 0)
	{
		return;
	}
	int numNodesInBatch = batchSize;
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numNodesInBatch + threadsPerBlock - 1) / threadsPerBlock;

	Layer *layers = globalCudaVar.layers;

	// printf("aa blocksPerGrid: %d\n", blocksPerGrid);
	// printf("aa threadsPerBlock: %d\n", threadsPerBlock);
	// printf("aa Total threads: %d\n", blocksPerGrid * threadsPerBlock);

	// GCN_aggregate_cuda<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(layers, numNodes, gnn.getGraph().getNodesptr(), gnn.getGraph().getEdgesList(), gnn.getGraph().getEdgeLen(), layerNumber, node, activationType);
	GCN_aggregate_cuda<<<blocksPerGrid, threadsPerBlock>>>(layers, batchNumber, batchSize, globalCudaVar.d_nodesPtr, globalCudaVar.d_edgesList, globalCudaVar.d_edgeWeights, layerNumber);
	hipDeviceSynchronize();
	// if (layerNumber == 2){
	// 	printprint<<<1, 1>>>(layers, batchNumber, batchSize, layerNumber);
	// 	hipDeviceSynchronize();
	// }

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("CUDA aaError1: %s\n", hipGetErrorString(err1));
	}

	NeuralNetworkOperation<<<blocksPerGrid, threadsPerBlock>>>(layers, batchNumber, batchSize, 3, layerNumber, activationType);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA ffError1: %s\n", hipGetErrorString(err));
	}
}

void backPropagation_cuda(GNN &gnn, int layerNumber, int totalLayers, int batchNumber, int batchSize, int num_features)
{
	// printf("Backpropagation Started\n");
	if (layerNumber == 0)
	{
		return;
	}
	// double *d_y_pred, *d_grad_output, *d_weights, *d_preActivatedFeatures, *d_aggregatedFeatures, *d_bias;
	if (layerNumber == totalLayers - 1)
	{
		int threads = 1024;
		int blocks = (batchSize + threads - 1) / threads;
		computeGradientsOutputLayer<<<blocks, threads>>>(globalCudaVar.layers, globalCudaVar.d_y_true, batchNumber, batchSize, gnn.numClasses(), totalLayers);
		hipDeviceSynchronize();
	}
	else
	{
		int threads = 1024;
		int blocks = (batchSize + threads - 1) / threads;
		computeGradientsIntermediateLayer<<<blocks, threads>>>(globalCudaVar.layers, batchNumber, batchSize, layerNumber, 1);
		hipDeviceSynchronize();
	}
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("CUDA bp1Error: %s\n", hipGetErrorString(err1));
	}

	int threads = 1024;
	int blocks = (num_features + threads - 1) / threads;
	double weightNorm = 0.0;
	double *d_weightNorm;
	double biasNorm = 0.0;
	double *d_biasNorm;
	hipMalloc((void **)&d_weightNorm, sizeof(float));
	hipMalloc((void **)&d_biasNorm, sizeof(float));
	hipMemcpy(d_weightNorm, &weightNorm, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_biasNorm, &biasNorm, sizeof(float), hipMemcpyHostToDevice);

	computeWeightBiasGradients<<<blocks, threads>>>(globalCudaVar.layers, layerNumber, gnn.getGraph
	().num_nodes(), d_weightNorm, d_biasNorm);
	hipDeviceSynchronize();
	clipGradients<<<blocks, threads>>>(globalCudaVar.layers, layerNumber, GRADIENT_NORM_CLIP_VALUE, d_weightNorm, d_biasNorm);
	hipDeviceSynchronize();
	// hipError_t err = hipGetLastError();
	// if (err != hipSuccess)
	// {
	// 	printf("CUDA bpError: %s\n", hipGetErrorString(err));
	// }
}

void optimizer(vector<int> numFeaturesPerLayer, int num_layers, int batchNumber, int batchSize, double learning_rate, double beta1, double beta2, double epsilon, int epochNumber)
{
	int t = epochNumber * (batchSize + 1);
	int threads = 1024;
	// int blocks = (batchSize + threads - 1) / threads;
	// Total number of threads is equal to the maximum of number of features among every layer
	int max_features = 0;
	int cpu_layers = num_layers;
	for (int i = 0; i < cpu_layers; i++)
	{
		if (numFeaturesPerLayer[i] > max_features)
		{
			max_features = numFeaturesPerLayer[i];
		}
	}

	int blocks = (max_features + threads - 1) / threads;

	adam<<<blocks, threads>>>(globalCudaVar.layers, num_layers, batchNumber, batchSize, learning_rate, beta1, beta2, epsilon, t);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA opError: %s\n", hipGetErrorString(err));
	}
}

void testAccuracy(Layer *d_layers, int num_classes, int num_nodes, int totalLayers)
{
	int *d_correct_predictions, correct_predictions = 0;
	hipMalloc(&d_correct_predictions, sizeof(int));
	hipMemcpy(d_correct_predictions, &correct_predictions, sizeof(int), hipMemcpyHostToDevice);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA t1Error: %s\n", hipGetErrorString(err));
	}
	int threadsPerBlock = 1024;
	int blocksPerGrid = (num_nodes + threadsPerBlock - 1) / threadsPerBlock;
	// printf("Test Accuracy\n");
	// Launch the accuracy calculation kernel
	// printf("hjfjhv\n");
	calculateAccuracy<<<blocksPerGrid, threadsPerBlock>>>(globalCudaVar.layers, globalCudaVar.d_y_true, d_correct_predictions, num_classes, num_nodes);
	hipDeviceSynchronize();

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("CUDA ttError: %s\n", hipGetErrorString(err1));
	}
	hipMemcpy(&correct_predictions, d_correct_predictions, sizeof(int), hipMemcpyDeviceToHost);

	// Calculate and print the accuracy
	double accuracy = (double)correct_predictions / (double)num_nodes;
	printf("Test Accuracy: %.2f%%\n", accuracy * 100);

	hipFree(d_correct_predictions);
}

int main()
{
	vector<int> numFeaturesPerLayer = {500, 16, 3};
	// vector<int> numFeaturesPerLayer = {16, 8, 2};
	const char *transformation = "xaviers";
	graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Pubmed/pubmed_edgelist.txt");
	// graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph.txt");
	G.parseGraph();

	GNN gnn(G, "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Pubmed/pubmed_features.txt", "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Pubmed/pubmed_labels.txt");
	// GNN gnn(G, "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph_feat.txt", "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph_labels.txt");
	gnn.gcnPreprocessing();
	cout << "preprosessing done" << endl;
	printf("num_layers_cpu: %d\n", gnn.getLayers().size());

	int num_layers = numFeaturesPerLayer.size();

	Layer *d_layers;
	hipMalloc(&d_layers, num_layers * sizeof(Layer));

	Layer *h_layers = new Layer[num_layers];
	for (int i = 0; i < num_layers; i++)
	{
		h_layers[i] = Layer();
		hipMalloc(&h_layers[i].weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].bias, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_bias, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].m_weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].m_biases, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].v_weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].v_biases, numFeaturesPerLayer[i] * sizeof(double));

		hipMalloc(&h_layers[i].aggregatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i > 0 ? i - 1 : i] * sizeof(double));
		hipMalloc(&h_layers[i].preActivatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].postActivatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_pre_act_output, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));

		hipMemcpy(&d_layers[i], &h_layers[i], sizeof(Layer), hipMemcpyHostToDevice);
	}
	vector<vector<double>> feats = gnn.getFeatures();
	// convert it to a 1D array
	vector<double> flat_feats;
	for (int i = 0; i < feats.size(); i++)
	{
		for (int j = 0; j < feats[i].size(); j++)
		{
			flat_feats.push_back(feats[i][j]);
		}
	}

	double *d_features;
	hipMalloc(&d_features, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double));
	// .data of vector of vector is a 1D array
	hipMemcpy(d_features, flat_feats.data(), gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyHostToDevice);

	initializeLayers_cuda(gnn, d_layers, numFeaturesPerLayer, d_features, gnn.getGraph().num_nodes(), transformation);
	hipDeviceSynchronize();
	double *features = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	hipMemcpy(features, d_features, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);

	// printf("\n\ndone\n\n");
	// traverse all the layers for forward porpagation

	int num_nodes = gnn.getGraph().num_nodes();
	// int num_nodes = 8000;
	for (int epoch = 1; epoch < 100; epoch++)
	{
		for (int i = 0; i < 3; i++)
		{
			forward(gnn, i, 0, num_nodes, 1);
		}
		for (int i = 2; i >= 0; i--)
		{
			backPropagation_cuda(gnn, i, numFeaturesPerLayer.size(), 0, num_nodes, numFeaturesPerLayer[i]);
		}
		// printf("Bp over\n");
		optimizer(numFeaturesPerLayer, numFeaturesPerLayer.size(), 0, gnn.getGraph().num_nodes(), 0.01, 0.9, 0.999, 1e-8, epoch);
		// printf("Optimization done\n");
		testAccuracy(d_layers, gnn.numClasses(), num_nodes, numFeaturesPerLayer.size());
		// printf("Accuracy calculated\n");
	}
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA ffError: %s\n", hipGetErrorString(err));
	}
	//  double *features = new double[num_nodes * feats[0].size()];
	//  hipMemcpy(features, d_features, num_nodes * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);
	//  for (int i = 0; i < feats[0].size(); i++)
	//  {
	//  	cout << features[4 * feats[0].size() + i] << " ";
	//  }

	// print features of node 4
	// double *featuress = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	// hipMemcpy(featuress, h_layers[1].preActivatedFeatures, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);

	return 0;
}
