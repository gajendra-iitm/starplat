#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "graph.cpp"
#define GRADIENT_NORM_CLIP_VALUE 3.0f
using namespace std;

env environment("cuda", "static", "test.cu");

class Layer
{
public:
	int32_t num_features;
	double *weights;
	double *bias;
	double epsilon = 0.01;
	double grad_epsilon;
	double *aggregatedFeatures;
	double *preActivatedFeatures;
	double *postActivatedFeatures;
	double *grad_pre_act_output;
	double *grad_weights;
	double *grad_bias;
	double *m_weights;
	double *m_biases;
	double m_epsilon = 0.0f;
	double v_epsilon = 0.0f;
	double *v_weights;
	double *v_biases;

	Layer() : num_features(0), weights(nullptr), bias(nullptr), epsilon(0.01), grad_epsilon(0.0), aggregatedFeatures(nullptr), preActivatedFeatures(nullptr), postActivatedFeatures(nullptr), grad_pre_act_output(nullptr), grad_weights(nullptr), grad_bias(nullptr), m_weights(nullptr), m_biases(nullptr), m_epsilon(0.0f), v_epsilon(0.0f), v_weights(nullptr), v_biases(nullptr) {}

	__device__ void xaviers(double *weights, int num_feat_current, int num_feat_prev)
	{
		double x = sqrt(6.0f / (num_feat_current + num_feat_prev));
		hiprandState state;
		// printf("num_feat_current: %d\n", num_feat_current);
		for (int i = 0; i < num_feat_current * num_feat_prev; i++)
		{
			hiprandState state;
			hiprand_init(3244, i, 0, &state);
			weights[i] = hiprand_uniform(&state) * x;
			// printf("%f\n",weights[i]);
		}
	}

	__device__ void he(double *weights, int num_feat_current, int num_feat_prev)
	{
		double x = sqrt(2.0f / (num_feat_prev));
		hiprandState state;
		// printf("num_feat_current: %d\n", num_feat_current);
		hiprand_init(42, 5, 0, &state);
		for (int i = 0; i < num_feat_current * num_feat_prev; i++)
		{
			hiprandState state;
			hiprand_init(42, i, 0, &state);
			double random_value = hiprand_uniform(&state) * x;
			weights[i] = random_value;
			// printf("%f\n",weights[i]);
		}
	}
};

class cudaVars
{
public:
	Layer *layers;
	int *d_nodesPtr;
	int *d_edgesList;
	float *d_edgeWeights;
	int *d_y_true;
};

cudaVars globalCudaVar;

__global__ void initializeLayerFields(Layer *layers, int num_layers, int *num_features, double *d_features, int num_nodes, int initialization_type)
{
	int i = threadIdx.x;
	if (i == 0)
	{
		layers[i].num_features = num_features[i];
		// printf("num_features: %d and i = %d\n", layers[i].num_features, i);
		for (int j = 0; j < num_nodes; j++)
		{
			for (int k = 0; k < layers[i].num_features; k++)
			{
				layers[i].preActivatedFeatures[j * layers[i].num_features + k] = 0;
				layers[i].postActivatedFeatures[j * layers[i].num_features + k] = d_features[j * layers[i].num_features + k];
			}
		}
	}

	else if (i < num_layers)
	{
		layers[i].num_features = num_features[i];
		// if (initialization_type == 1)
		// {
		// 	layers[i].xaviers(layers[i].weights, num_features[i], num_features[i - 1]);
		// }
		// else if (initialization_type == 2)
		// {
		// 	layers[i].he(layers[i].weights, num_features[i - 1], num_features[i]);
		// }

		for (int j = 0; j < num_features[i]; j++)
		{
			// layers[i].bias[j] = 0.5;
			layers[i].bias[j] = 0;
			layers[i].grad_bias[j] = 0;
			layers[i].m_biases[j] = 0;
			layers[i].v_biases[j] = 0;
		}

		for (int j = 0; j < num_features[i - 1] * num_features[i]; j++)
		{
			layers[i].grad_weights[j] = 0;
			layers[i].m_weights[j] = 0;
			layers[i].v_weights[j] = 0;
		}

		for (int j = 0; j < num_features[i - 1] * num_nodes; j++)
		{
			layers[i].aggregatedFeatures[j] = 0;
		}

		for (int j = 0; j < num_features[i] * num_nodes; j++)
		{
			layers[i].preActivatedFeatures[j] = 0;
			layers[i].postActivatedFeatures[j] = 0;
			layers[i].grad_pre_act_output[j] = 0;
		}
	}
}

__global__ void weights_copy_per_layer(Layer *layers, int layerNumber, double *weights, int num_features_current, int num_features_prev)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < num_features_current * num_features_prev)
	{
		layers[layerNumber].weights[i] = weights[i];
	}
}

void initializeLayers_cuda(GNN &gnn, Layer *d_layers, vector<int> numFeaturesPerLayer, double *d_features, int num_nodes, const char *transformation)
{
	int *d_numFeaturesPerLayer;
	hipMalloc(&d_numFeaturesPerLayer, numFeaturesPerLayer.size() * sizeof(int));
	hipMemcpy(d_numFeaturesPerLayer, numFeaturesPerLayer.data(), numFeaturesPerLayer.size() * sizeof(int), hipMemcpyHostToDevice);

	int initialization_type;
	if (transformation == "xaviers")
	{
		initialization_type = 1;
		for (int i = 1; i < numFeaturesPerLayer.size(); i++)
		{
			double xavier = sqrt(6.0 / (numFeaturesPerLayer[i] + numFeaturesPerLayer[i - 1]));
			std::random_device rd; // Obtain a random number from hardware
			std::mt19937 gen(42);  // Seed the generator
			std::uniform_real_distribution<> dist(0, xavier);

			double *weights = new double[numFeaturesPerLayer[i] * numFeaturesPerLayer[i - 1]];
			double *d_weights;
			hipMalloc(&d_weights, numFeaturesPerLayer[i] * numFeaturesPerLayer[i - 1] * sizeof(double));
			for (int j = 0; j < numFeaturesPerLayer[i] * numFeaturesPerLayer[i - 1]; j++)
			{
				weights[j] = dist(gen);
			}
			hipMemcpy(d_weights, weights, numFeaturesPerLayer[i] * numFeaturesPerLayer[i - 1] * sizeof(double), hipMemcpyHostToDevice);

			int threadsPerBlock = 1024;
			int blocksPerGrid = (numFeaturesPerLayer[i] * numFeaturesPerLayer[i - 1] + threadsPerBlock - 1) / threadsPerBlock;
			weights_copy_per_layer<<<blocksPerGrid, threadsPerBlock>>>(d_layers, i, d_weights, numFeaturesPerLayer[i], numFeaturesPerLayer[i - 1]);
			hipDeviceSynchronize();
			hipError_t err = hipGetLastError();
			if (err != hipSuccess)
			{
				printf("CUDA Errorxaviers: %s\n", hipGetErrorString(err));
			}
			delete weights;
		}
	}
	else if (transformation == "he")
	{
		initialization_type = 2;
	}

	initializeLayerFields<<<1, numFeaturesPerLayer.size()>>>(d_layers, numFeaturesPerLayer.size(), d_numFeaturesPerLayer, d_features, num_nodes, initialization_type);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	}

	globalCudaVar.layers = d_layers;

	hipMalloc(&globalCudaVar.d_nodesPtr, gnn.getGraph().num_nodes() * sizeof(int));
	hipMemcpy(globalCudaVar.d_nodesPtr, gnn.getGraph().getNodesptr(), gnn.getGraph().num_nodes() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_edgesList, gnn.getGraph().num_edges() * sizeof(int));
	hipMemcpy(globalCudaVar.d_edgesList, gnn.getGraph().getEdgesList(), gnn.getGraph().num_edges() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_edgeWeights, gnn.getGraph().num_edges() * sizeof(float));
	hipMemcpy(globalCudaVar.d_edgeWeights, gnn.getGraph().getEdgeLen(), gnn.getGraph().num_edges() * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_y_true, gnn.getGraph().num_nodes() * sizeof(int));
	hipMemcpy(globalCudaVar.d_y_true, gnn.getLabels().data(), gnn.getGraph().num_nodes() * sizeof(int), hipMemcpyHostToDevice);

	// copy the features to a double  and print the features of node 4

	// copy the weights of layer 1 to a double array and print the first 10 weights
	//  double *weights = new double[numFeaturesPerLayer[0] * numFeaturesPerLayer[1]];
	//  hipMemcpy(weights, h_layers[2].weights, numFeaturesPerLayer[0] * numFeaturesPerLayer[1] * sizeof(double), hipMemcpyDeviceToHost);
	//  for (int i = 0; i < 10; i++)
	//  {
	//  	cout << weights[i] << " ";
	//  }
}

__global__ void GCN_aggregate_cuda(Layer *layers, int batchNumber, int batchSize, int *V, int *E, float *edgeWeights, int layerNumber)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		int prevNumFeatures = layers[layerNumber - 1].num_features;
		int idx = batchNumber * batchSize + tid;
		// if (idx == 0)
		// printf("NUM FEATURES: %d and layernumber %d\n", layers[layerNumber - 1].num_features, layerNumber);
		for (int i = 0; i < prevNumFeatures; i++)
		{
			layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + i] = 0;
		}

		for (int j = V[idx]; j < V[idx + 1]; j++)
		{
			int neighbor = E[j];
			double weight = (double)edgeWeights[j];
			for (int i = 0; i < prevNumFeatures; i++)
			{
				layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + i] += layers[layerNumber - 1].postActivatedFeatures[neighbor * prevNumFeatures + i] * weight;
			}
			// if (idx == 102)
			// {
			// 	// Print the neighbour and the postact value of it for 0th feature
			// 	printf("Node %d: Neighbor = %d, PostAct = %f\n", idx, neighbor, layers[layerNumber - 1].postActivatedFeatures[neighbor * prevNumFeatures]);
			// 	// edge weight also
			// 	printf("Node %d: Neighbor = %d, EdgeWeight = %f\n", idx, neighbor, weight);
			// }			
		}
		// // if(epoch>1)
		// if(idx == 102)
		// printf("Node %d: Layer %d: aggregatedFeaturesnumber = %f\n", idx, layerNumber, layers[layerNumber].aggregatedFeatures[idx * layers[layerNumber - 1].num_features]);
	}
}
/*
	double *weights;
	double *bias;
	double epsilon = 0.01;
	double grad_epsilon;
	double *aggregatedFeatures;
	double *preActivatedFeatures;
	double *postActivatedFeatures;
	double *grad_pre_act_output;
	double *grad_weights;
	double *grad_bias;
*/

__global__ void NeuralNetworkOperation(Layer *layers, int batchNumber, int batchSize, int totalLayers, int layerNumber, int activationType)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("NN tid: %d\n", tid);

		int idx = batchNumber * batchSize + tid;
		int currNumFeatures = layers[layerNumber].num_features;
		int prevNumFeatures = layers[layerNumber - 1].num_features;
		if (layerNumber < totalLayers)
			for (int i = 0; i < currNumFeatures; i++)
			{
				double sum = 0.0f;

				for (int j = 0; j < prevNumFeatures; j++)
				{
					sum += layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + j] * layers[layerNumber].weights[j * currNumFeatures + i];
				}

				// if (idx == 102 && i == 0) {
				// 	for (int j = 0; j < prevNumFeatures; j++) {
				// 		printf("Node %d: index %d: Aggregated Features: %f\n", idx, j, layers[layerNumber].aggregatedFeatures[idx * prevNumFeatures + j]);
				// 		printf("Weight from feature %d to feature %d: %f\n", j, i, layers[layerNumber].weights[j * currNumFeatures + i]);
				// 	}
				// 	printf("Sum: %f\n", sum);
				// }

				layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i] = sum + layers[layerNumber].bias[i];
				if (layerNumber == 1 && idx == 0)
				{
					//// printf("preActivatedFeatures: %f\n", layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i]);
				}
				if (layerNumber == totalLayers - 1)
				{
					// softmax

					// softmax(layers[layerNumber].postActivatedFeatures[node], layers[layerNumber].num_features, layers[layerNumber].postActivatedFeatures[node]);
				}

				else if (activationType == 1)
				{
					layers[layerNumber].postActivatedFeatures[idx * currNumFeatures + i] = (layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i] > 0 ? layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i] : 0.0f);
				}
				else if (activationType == 2)
				{
					layers[layerNumber].postActivatedFeatures[idx * currNumFeatures + i] =
						tanh(layers[layerNumber].preActivatedFeatures[idx * currNumFeatures + i]);
				}
				// printf("Node %d: preActivatedFeatures = %f\n", idx, layers[layerNumber].preActivatedFeatures[idx * layers[layerNumber + 1].num_features + i]);

				// printf("Node %d: postActivatedFeatures = %f\n", idx, layers[layerNumber].postActivatedFeatures[idx * layers[layerNumber + 1].num_features + i]);
			}
	}
}

__global__ void computeGradientsOutputLayer(Layer *layers, int *y_true, int batchNumber, int batchSize, int num_classes, int num_layers)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("tid: %d\n", tid);

		int idx = batchNumber * batchSize + tid;
		int label = y_true[idx];

		double *y_pred = layers[num_layers - 1].preActivatedFeatures;
		// double *grad_output = globalCudaVar.layers[num_layers - 1].grad_pre_act_output;

		for (int j = 0; j < num_classes; ++j)
		{
			// grad_output[idx * num_classes + j] = y_pred[idx * num_classes + j] - (label == j ? 1.0f : 0.0f);
			layers[num_layers - 1].grad_pre_act_output[idx * num_classes + j] = y_pred[idx * num_classes + j] - (label == j ? 1.0f : 0.0f);
			// printf("grad_pre_act_output of layer %d entry %d: %f\n", num_layers - 1, idx * num_classes + j, layers[num_layers - 1].grad_pre_act_output[idx * num_classes + j]);
			// print y_pred
			// printf("y_pred for node %d for class %d: %f\n", idx, j, y_pred[idx * num_classes + j]);
		}
	}
}

__global__ void computeGradientsIntermediateLayer(Layer *layers, int batchNumber, int batchSize, int layer_num, int activation_type)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("tid: %d\n", tid);

		int current_num_features = layers[layer_num].num_features;
		int next_num_features = layers[layer_num + 1].num_features;

		int idx = batchNumber * batchSize + tid;
		for (int i = 0; i < current_num_features; ++i)
		{
			layers[layer_num].grad_pre_act_output[idx * current_num_features + i] = 0.0f;
			for (int j = 0; j < next_num_features; ++j)
			{
				layers[layer_num].grad_pre_act_output[idx * current_num_features + i] += layers[layer_num + 1].grad_pre_act_output[idx * next_num_features + j] * layers[layer_num + 1].weights[i * next_num_features + j];
			}
			// printf("grad_pre_act_output of layer %d entry %d: %f\n", layer_num, idx * current_num_features + i, layers[layer_num].grad_pre_act_output[idx * current_num_features + i]);
			if (activation_type == 1)
			{ // ReLU
				layers[layer_num].grad_pre_act_output[idx * current_num_features + i] *= (layers[layer_num].preActivatedFeatures[idx * current_num_features + i] > 0) ? 1 : 0;
			}
			else if (activation_type == 2)
			{ // Tanh
				layers[layer_num].grad_pre_act_output[idx * current_num_features + i] *= (1 - layers[layer_num].postActivatedFeatures[idx * current_num_features + i] * layers[layer_num].postActivatedFeatures[idx * current_num_features + i]);
			}
		}
	}
}

__global__ void computeWeightBiasGradients(Layer *layers, int layer_num, int num_nodes, float *weightNorm, float *biasNorm, int epoch)
{
	int featIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int prev_features = layers[layer_num - 1].num_features;

	if (featIdx < prev_features)
	{
		int curr_num_features = layers[layer_num].num_features;
		for (int j = 0; j < curr_num_features; ++j)
		{
			double weight_grad_sum = 0.0;
			double bias_grad_sum = 0.0;

			// Accumulate gradient sums
			for (int nod = 0; nod < num_nodes; ++nod)
			{
				weight_grad_sum += layers[layer_num].aggregatedFeatures[nod * prev_features + featIdx] *
								   layers[layer_num].grad_pre_act_output[nod * curr_num_features + j];
				bias_grad_sum += layers[layer_num].grad_pre_act_output[nod * curr_num_features + j];
			}

			// Update gradient weights
			layers[layer_num].grad_weights[featIdx * curr_num_features + j] = weight_grad_sum;
// if(epoch ==3)
// printf("Aggregated Features of layer %d in epoch %d: %f\n", layer_num, epoch, layers[layer_num].grad_pre_act_output[featIdx * prev_features + j]);
			atomicAdd(weightNorm, (float)(weight_grad_sum * weight_grad_sum));
			// printf("Weight Norm: %f\n", *weightNorm);
			// Update bias gradient if featIdx == 0 to avoid overwrites
			if (featIdx == 0)
			{
				layers[layer_num].grad_bias[j] = bias_grad_sum;

				// Use atomicAdd to safely update biasNorm
				atomicAdd(biasNorm, (float)(bias_grad_sum * bias_grad_sum));
			}
		}
	}
}

__global__ void clipGradients(Layer *layers, int layer_num, float clip_value, float *weightNorm, float *biasNorm, int epochNumber)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	double weightNorm1 = sqrtf(*weightNorm);
	double biasNorm1 = sqrtf(*biasNorm);
	// if(tid==0)
		// printf("WeightNorm of layer %d in epoch %d: %f\n", layer_num, epochNumber, weightNorm1);
	int num_input_features = layers[layer_num - 1].num_features;
	int num_output_features = layers[layer_num].num_features;
	if (tid < num_output_features)
	{
		for (int i = 0; i < num_input_features; i++)
		{
			int weight_idx = i * num_output_features + tid;
			if (weightNorm1 > clip_value)
			{
				// printf("Weight Norm: %f\n", weightNorm1);
				// printf("Pre update weight %d: %f\n", weight_idx, layers[layer_num].grad_weights[weight_idx]);
				layers[layer_num].grad_weights[weight_idx] *= clip_value / weightNorm1;
				// printf("Post update weight %d: %f\n", weight_idx, layers[layer_num].grad_weights[weight_idx]);
				
			}
		}
		if (biasNorm1 > clip_value)
		{
			layers[layer_num].grad_bias[tid] *= clip_value / biasNorm1;
		}
	}
}

__global__ void printprint(Layer *layers, int total_layers, int num_nodes, int epochNumber)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid == 0)
	{
		// print all the values of all the layers
		for (int i = 1; i < total_layers; i++)
		{
			printf("\nLayer %d\n", i);
			for (int j = 0; j < layers[i].num_features * layers[i - 1].num_features; j++)
			{
				printf("Weight %d of layer %d in epoch %d: %f\n", j, i, epochNumber, layers[i].weights[j]);
			}
			// 	for(int j=0;j<layers[i].num_features;j++){
			// 		printf("Bias %d: %f\n", j, layers[i].bias[j]);
			// 	}
			// for (int j = 0; j < layers[i].num_features * num_nodes; j++)
			// {
			// 	printf("Node %d: preActivatedFeatures = %f\n", j, layers[i].preActivatedFeatures[j]);
			// 	printf("Node %d: postActivatedFeatures = %f\n", j, layers[i].postActivatedFeatures[j]);
			// }
			// Prints the grad weights
			// for (int j = 0; j < layers[i].num_features * layers[i - 1].num_features; j++)
			// {
			// 	printf("Grad Weight %d of layer %d in epoch %d: %f\n", j, i, epochNumber, layers[i].grad_weights[j]);
			// }
			// for (int j = 0; j < layers[i].num_features; j++)
			// {
			// 	printf("Grad Bias %d: %f\n", j, layers[i].grad_bias[j]);
			// }
		}
	}
}

__global__ void adam(Layer *layers, int num_layers, int batchNumber, int batchSize, double learning_rate, double beta1, double beta2, double epsilon, int t)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = 1; i < num_layers; i++)
	{
		int numfeats = layers[i].num_features;

		if (tid < numfeats)
		{
			int k = tid;

			double beta1_pow_t = pow(beta1, t); // Precompute beta1^t
			double beta2_pow_t = pow(beta2, t); // Precompute beta2^t
												// int num_features = layers[i].num_features;
			// Update biased first moment estimate for biases j =
			layers[i].m_biases[k] = beta1 * layers[i].m_biases[k] + (1 - beta1) * layers[i].grad_bias[k];
			// Update biased second raw moment estimate for biases
			layers[i].v_biases[k] = beta2 * layers[i].v_biases[k] + (1 - beta2) * layers[i].grad_bias[k] * layers[i].grad_bias[k];

			// Correct bias for the first and second moments (bias correction)
			double m_bias_hat = layers[i].m_biases[k] / (1 - beta1_pow_t);
			double v_bias_hat = layers[i].v_biases[k] / (1 - beta2_pow_t);

			// Update biases
			// printf("Pre updated bias %d: %f\n", k, layers[i].bias[k]);
			layers[i].bias[k] -= learning_rate * m_bias_hat / (sqrt(v_bias_hat) + epsilon);
			// printf("Post updated bias %d: %f\n", k, layers[i].bias[k]);
			for (int j = 0; j < layers[i - 1].num_features; j++)
			{
				int weight_idx = j * layers[i].num_features + k;

				// Update biased first moment estimate for weights
				layers[i].m_weights[weight_idx] = beta1 * layers[i].m_weights[weight_idx] + (1 - beta1) * layers[i].grad_weights[weight_idx];
				// Update biased second raw moment estimate for weights
				layers[i].v_weights[weight_idx] = beta2 * layers[i].v_weights[weight_idx] + (1 - beta2) * layers[i].grad_weights[weight_idx] * layers[i].grad_weights[weight_idx];

				// Correct bias for the first and second moments (bias correction)
				double m_weight_hat = layers[i].m_weights[weight_idx] / (1 - beta1_pow_t);
				double v_weight_hat = layers[i].v_weights[weight_idx] / (1 - beta2_pow_t);

				// Update weights
				// print pre update weights
				// printf(" Pre updated Weight %d: %f\n", weight_idx, layers[i].weights[weight_idx]);
				layers[i].weights[weight_idx] -= learning_rate * (m_weight_hat / (sqrt(v_weight_hat) + epsilon) );

				// print the weights
				//  printf("Post updated Weight %d: %f\n", weight_idx, layers[i].weights[weight_idx]);
			}
		}
	}
}

// __global__ void calculateAccuracy(Layer *layers, int *ground_truth, int *correct_predictions, int num_classes, int batchNumber, int batchSize)
__global__ void calculateAccuracy(Layer *layers, int *ground_truth, int *correct_predictions, int num_classes, int num_nodes, int num_layers)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// printf("JEIIJ\n");
	if (idx < num_nodes)
	{
		double *predictions = layers[num_layers - 1].preActivatedFeatures;
		// Find the index of the maximum predicted class (argmax)
		int predicted_class = 0;
		double max_value = predictions[idx * num_classes];

		for (int i = 0; i < num_classes; ++i)
		{
			if (predictions[idx * num_classes + i] > max_value)
			{
				max_value = predictions[idx * num_classes + i];
				// printf("max_value: %f\n", max_value);
				predicted_class = i;
			}
		}
		// printf("max_value: %f\n", max_value);

		// Compare with the ground truth
		if (predicted_class == ground_truth[idx])
		{
			atomicAdd(correct_predictions, 1); // Increment the counter for correct predictions
		}
	}
}

void forwardPass_cuda(GNN &gnn, int layerNumber, int batchNumber, int batchSize, int activationType)
{
	if (layerNumber == 0)
	{
		return;
	}
	int numNodesInBatch = batchSize;
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numNodesInBatch + threadsPerBlock - 1) / threadsPerBlock;

	Layer *layers = globalCudaVar.layers;

	// printf("aa blocksPerGrid: %d\n", blocksPerGrid);
	// printf("aa threadsPerBlock: %d\n", threadsPerBlock);
	// printf("aa Total threads: %d\n", blocksPerGrid * threadsPerBlock);

	// GCN_aggregate_cuda<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(layers, numNodes, gnn.getGraph().getNodesptr(), gnn.getGraph().getEdgesList(), gnn.getGraph().getEdgeLen(), layerNumber, node, activationType);
	GCN_aggregate_cuda<<<blocksPerGrid, threadsPerBlock>>>(layers, batchNumber, batchSize, globalCudaVar.d_nodesPtr, globalCudaVar.d_edgesList, globalCudaVar.d_edgeWeights, layerNumber);
	hipDeviceSynchronize();

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("CUDA aaError1: %s\n", hipGetErrorString(err1));
	}

	NeuralNetworkOperation<<<blocksPerGrid, threadsPerBlock>>>(layers, batchNumber, batchSize, 3, layerNumber, activationType);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA ffError1: %s\n", hipGetErrorString(err));
	}
}

void backPropagation_cuda(GNN &gnn, int layerNumber, int totalLayers, int batchNumber, int batchSize, int num_features, int epochNumber, int activationType)
{
	// printf("Backpropagation Started\n");
	if (layerNumber == 0)
	{
		return;
	}
	// double *d_y_pred, *d_grad_output, *d_weights, *d_preActivatedFeatures, *d_aggregatedFeatures, *d_bias;
	if (layerNumber == totalLayers - 1)
	{
		int threads = 1024;
		int blocks = (batchSize + threads - 1) / threads;
		computeGradientsOutputLayer<<<blocks, threads>>>(globalCudaVar.layers, globalCudaVar.d_y_true, batchNumber, batchSize, gnn.numClasses(), totalLayers);
		hipDeviceSynchronize();
	}
	else
	{
		int threads = 1024;
		int blocks = (batchSize + threads - 1) / threads;
		computeGradientsIntermediateLayer<<<blocks, threads>>>(globalCudaVar.layers, batchNumber, batchSize, layerNumber, activationType);
		hipDeviceSynchronize();
	}
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("CUDA bp1Error: %s\n", hipGetErrorString(err1));
	}

	int threads = 1024;
	int blocks = (num_features + threads - 1) / threads;

	float *d_weightNorm, *d_biasNorm;
	hipMalloc(&d_weightNorm, sizeof(float));
	hipMalloc(&d_biasNorm, sizeof(float));
	hipMemset(d_weightNorm, 0, sizeof(float));
	hipMemset(d_biasNorm, 0, sizeof(float));

	computeWeightBiasGradients<<<blocks, threads>>>(globalCudaVar.layers, layerNumber, gnn.getGraph().num_nodes(), d_weightNorm, d_biasNorm, epochNumber);
	hipDeviceSynchronize();

	clipGradients<<<blocks, threads>>>(globalCudaVar.layers, layerNumber, GRADIENT_NORM_CLIP_VALUE, d_weightNorm, d_biasNorm, epochNumber);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA bpError: %s\n", hipGetErrorString(err));
	}
}

void adamOptimizer_cuda(vector<int> numFeaturesPerLayer, int num_layers, int batchNumber, int batchSize, double learning_rate, double beta1, double beta2, double epsilon, int epochNumber)
{
	int t = epochNumber * (batchSize + 1);
	int threads = 1024;
	// int blocks = (batchSize + threads - 1) / threads;
	// Total number of threads is equal to the maximum of number of features among every layer
	int max_features = 0;
	int cpu_layers = num_layers;
	for (int i = 0; i < cpu_layers; i++)
	{
		if (numFeaturesPerLayer[i] > max_features)
		{
			max_features = numFeaturesPerLayer[i];
		}
	}

	int blocks = (max_features + threads - 1) / threads;

	adam<<<blocks, threads>>>(globalCudaVar.layers, num_layers, batchNumber, batchSize, learning_rate, beta1, beta2, epsilon, t);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA opError: %s\n", hipGetErrorString(err));
	}
}

void predict_cuda(Layer *d_layers, int num_classes, int num_nodes, int totalLayers)
{
	int *d_correct_predictions, correct_predictions = 0;
	hipMalloc(&d_correct_predictions, sizeof(int));
	hipMemcpy(d_correct_predictions, &correct_predictions, sizeof(int), hipMemcpyHostToDevice);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA t1Error: %s\n", hipGetErrorString(err));
	}
	int threadsPerBlock = 1024;
	int blocksPerGrid = (num_nodes + threadsPerBlock - 1) / threadsPerBlock;
	// printf("Test Accuracy\n");
	// Launch the accuracy calculation kernel
	// printf("hjfjhv\n");
	calculateAccuracy<<<blocksPerGrid, threadsPerBlock>>>(globalCudaVar.layers, globalCudaVar.d_y_true, d_correct_predictions, num_classes, num_nodes, totalLayers);
	hipDeviceSynchronize();

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("CUDA ttError: %s\n", hipGetErrorString(err1));
	}
	hipMemcpy(&correct_predictions, d_correct_predictions, sizeof(int), hipMemcpyDeviceToHost);

	// Calculate and print the accuracy
	double accuracy = (double)correct_predictions / (double)num_nodes;
	printf("Test Accuracy: %.2f%%\n", accuracy * 100);
	fflush(stdout);

	hipFree(d_correct_predictions);
}

int main()
{
	// vector<int> numFeaturesPerLayer = {500, 16, 3}; //pubmed
	// vector<int> numFeaturesPerLayer = {4973, 40, 17}; //wiki
	vector<int> numFeaturesPerLayer = {500, 16, 7}; // flickr
	// vector<int> numFeaturesPerLayer = {16, 8, 2};//sample graph
	// vector<int> numFeaturesPerLayer = {745, 16, 8}; // amazon
	const char *transformation = "xaviers";
	// graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Wiki/wiki_edgelist.txt");
	// graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Amazon/amazon_edgelist.txt");
	graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/flickr/flickr_edgelist.txt");

	// graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Pubmed/pubmed_edgelist.txt");
	// graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph.txt");
	G.parseGraph();
	// GNN gnn(G, "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Amazon/amazon_features.txt", "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Amazon/amazon_labels.txt");
	GNN gnn(G,"/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/flickr/flickr_features.txt","/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/flickr/flickr_labels.txt" );

	// GNN gnn(G,"/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Wiki/wiki_features.txt","/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Wiki/wiki_labels.txt" );
	// GNN gnn(G, "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Pubmed/pubmed_features.txt", "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/Pubmed/pubmed_labels.txt");
	// GNN gnn(G, "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph_feat.txt", "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph_labels.txt");
	gnn.gcnPreprocessing();
	cout << "preprosessing done" << endl;
	printf("num_layers_cpu: %d\n", gnn.getLayers().size());

	int num_layers = numFeaturesPerLayer.size();

	Layer *d_layers;
	hipMalloc(&d_layers, num_layers * sizeof(Layer));

	Layer *h_layers = new Layer[num_layers];
	for (int i = 0; i < num_layers; i++)
	{
		h_layers[i] = Layer();
		hipMalloc(&h_layers[i].weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].bias, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_bias, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].m_weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].m_biases, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].v_weights, numFeaturesPerLayer[(i > 0 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].v_biases, numFeaturesPerLayer[i] * sizeof(double));

		hipMalloc(&h_layers[i].aggregatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i > 0 ? i - 1 : i] * sizeof(double));
		hipMalloc(&h_layers[i].preActivatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].postActivatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_pre_act_output, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));

		hipMemcpy(&d_layers[i], &h_layers[i], sizeof(Layer), hipMemcpyHostToDevice);
	}
	vector<vector<double>> feats = gnn.getFeatures();
	// convert it to a 1D array
	// vector<double> flat_feats;
	// for (int i = 0; i < feats.size(); i++)
	// {
	// 	for (int j = 0; j < feats[i].size(); j++)
	// 	{
	// 		flat_feats.push_back(feats[i][j]);
	// 	}
	// }

	// In host
	double *flat_feats = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	for (int i = 0; i < gnn.getGraph().num_nodes(); i++)
	{
		for (int j = 0; j < feats[0].size(); j++)
		{
			flat_feats[i * feats[0].size() + j] = feats[i][j];
		}
	}

	double *d_features;
	hipMalloc(&d_features, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double));
	// .data of vector of vector is a 1D array
	hipMemcpy(d_features, flat_feats, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyHostToDevice);

	initializeLayers_cuda(gnn, d_layers, numFeaturesPerLayer, d_features, gnn.getGraph().num_nodes(), transformation);
	hipDeviceSynchronize();

	// printprint<<<1, 1>>>(d_layers, numFeaturesPerLayer.size(), gnn.getGraph().num_nodes(),1);
	// hipDeviceSynchronize();

	double *features = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	hipMemcpy(features, d_features, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);

	// printf("\n\ndone\n\n");
	// traverse all the layers for forward porpagation

	int num_nodes = gnn.getGraph().num_nodes();
	// int num_nodes = 8000;

		// printprint<<<1, 1>>>(globalCudaVar.layers, numFeaturesPerLayer.size(), gnn.getGraph().num_nodes(), 1);
		// hipDeviceSynchronize();
    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);
	for (int epoch = 1; epoch < 50; epoch++)
	{
		for (int i = 0; i < 3; i++)
		{
			forwardPass_cuda(gnn, i, 0, num_nodes, 1);
		}
		// printf("\n\n");
		// if(epoch == 3)
		// exit(0);

		for (int i = 2; i >= 0; i--)
		{
			backPropagation_cuda(gnn, i, numFeaturesPerLayer.size(), 0, num_nodes, numFeaturesPerLayer[i], epoch, 1);
		}
		// printf("Bp over\n");

		adamOptimizer_cuda(numFeaturesPerLayer, numFeaturesPerLayer.size(), 0, gnn.getGraph().num_nodes(), 0.001, 0.9, 0.999, 1e-8, epoch);
		// exit(0);
		// printf("Optimization done\n");
		predict_cuda(d_layers, gnn.numClasses(), num_nodes, numFeaturesPerLayer.size());
		// printf("Accuracy calculated\n");
	}
	   hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA ffError: %s\n", hipGetErrorString(err));
	}
	//  double *features = new double[num_nodes * feats[0].size()];
	//  hipMemcpy(features, d_features, num_nodes * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);
	//  for (int i = 0; i < feats[0].size(); i++)
	//  {
	//  	cout << features[4 * feats[0].size() + i] << " ";
	//  }

	// print features of node 4
	// double *featuress = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	// hipMemcpy(featuress, h_layers[1].preActivatedFeatures, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);

	return 0;
}
