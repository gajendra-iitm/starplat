// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "scc.h"

#include<bits/stdc++.h>
using namespace std;


void vHong(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_modified;
  hipMalloc(&d_modified, sizeof(int)*(V));

  int* d_outDeg;
  hipMalloc(&d_outDeg, sizeof(int)*(V));

  int* d_inDeg;
  hipMalloc(&d_inDeg, sizeof(int)*(V));

  bool* d_visitFw;
  hipMalloc(&d_visitFw, sizeof(bool)*(V));

  bool* d_visitBw;
  hipMalloc(&d_visitBw, sizeof(bool)*(V));

  bool* d_propFw;
  hipMalloc(&d_propFw, sizeof(bool)*(V));

  bool* d_propBw;
  hipMalloc(&d_propBw, sizeof(bool)*(V));

  bool* d_isPivot;
  hipMalloc(&d_isPivot, sizeof(bool)*(V));

  int* d_scc;
  hipMalloc(&d_scc, sizeof(int)*(V));

  int* d_range;
  hipMalloc(&d_range, sizeof(int)*(V));

  int* d_pivotField;
  hipMalloc(&d_pivotField, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(int)false);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_outDeg,(int)0);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_inDeg,(int)0);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw,(bool)false);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw,(bool)false);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propFw,(bool)false);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propBw,(bool)false);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_isPivot,(bool)false);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range,(int)0);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_pivotField,(int)-1);

  vHong_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_outDeg,d_inDeg);
  hipDeviceSynchronize();



  ; // asst in .cu

  bool fpoint1 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint1) {

    fpoint1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_range,d_isPivot);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT

    bool* h_scc1;
    h_scc1 = (bool *)malloc( (V)*sizeof(bool));
    hipMemcpy(  h_scc1,   d_isPivot, sizeof(bool)*(V), hipMemcpyDeviceToHost);
    int scc_cnt1 = 0;
    for(int i = 0; i < V; i++) {
      cout<<h_scc1[i]<<" ";
    }cout << endl;
    cout << "No.of strongly connected components in graph : " << scc_cnt1 << endl;


  vHong_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_inDeg,d_scc,d_outDeg,d_pivotField,d_range);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  vHong_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_pivotField,d_range,d_visitFw,d_visitBw,d_isPivot);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  bool fpoint2 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint2) {

    fpoint2 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_visitBw,d_propFw,d_scc,d_propBw,d_range,d_visitFw);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT

  vHong_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_range,d_visitBw,d_visitFw,d_propBw,d_propFw);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  fpoint1 = false;
  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint1) {

    fpoint1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_range,d_isPivot);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range,(int)0);

  vHong_kernel_8<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_range);
  hipDeviceSynchronize();



  bool fpoint4 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint4) {

    fpoint4 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint4), &fpoint4, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint4), &fpoint4, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_9<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_range);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint4, HIP_SYMBOL(::fpoint4), sizeof(bool), 0, hipMemcpyDeviceToHost);



    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint4), &fpoint4, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_10<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_range);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint4, HIP_SYMBOL(::fpoint4), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint4, HIP_SYMBOL(::fpoint4), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT

  bool fpoint5 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint5) {

    fpoint5 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint5), &fpoint5, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_11<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_inDeg,d_scc,d_outDeg,d_pivotField,d_range);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    vHong_kernel_12<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_pivotField,d_range,d_visitFw,d_visitBw,d_isPivot);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    fpoint2 = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    while(!fpoint2) {

      fpoint2 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
      vHong_kernel_13<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_visitBw,d_propFw,d_scc,d_propBw,d_range,d_visitFw);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT

    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint5), &fpoint5, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel_14<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_range,d_visitBw,d_visitFw,d_propBw,d_propFw);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint5, HIP_SYMBOL(::fpoint5), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint5, HIP_SYMBOL(::fpoint5), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT
bool* h_scc;
    h_scc = (bool *)malloc( (V)*sizeof(bool));
    hipMemcpy(  h_scc,   d_isPivot, sizeof(bool)*(V), hipMemcpyDeviceToHost);
    int scc_cnt = 0;
    for(int i = 0; i < V; i++) {
      cout<<h_scc[i]<<" ";
    }cout << endl;
    cout << "No.of strongly connected components in graph : " << scc_cnt << endl;


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_range);
  hipFree(d_propBw);
  hipFree(d_visitFw);
  hipFree(d_propFw);
  hipFree(d_inDeg);
  hipFree(d_pivotField);
  hipFree(d_isPivot);
  hipFree(d_outDeg);
  hipFree(d_visitBw);
  hipFree(d_scc);
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN





int main(int argc,char* argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  vHong(g);
  return 0;
}
