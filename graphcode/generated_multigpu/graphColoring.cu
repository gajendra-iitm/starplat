// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "graphColoring.h"

void colorGraph(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int numNodes = g.num_nodes( ); // asst in .cu 
  int** h_numNodes;
  h_numNodes = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_numNodes[i] = (int*)malloc(sizeof(int));
  }

  int** d_numNodes;
  d_numNodes = (int**)malloc(sizeof(int*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_numNodes[i],sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_numNodes[i],g.num_nodes( ));
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  long* h_color;
  h_color=(long*)malloc(sizeof(long)*(V+1));
  long** d_color;
  d_color = (long**)malloc(sizeof(long*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_color[i], sizeof(long)*(V+1));
  }

  int* h_color1;
  h_color1=(int*)malloc(sizeof(int)*(V+1));
  int** d_color1;
  d_color1 = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_color1[i], sizeof(int)*(V+1));
  }

  int* h_color2;
  h_color2=(int*)malloc(sizeof(int)*(V+1));
  int** d_color2;
  d_color2 = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_color2[i], sizeof(int)*(V+1));
  }

  bool* h_modified;
  h_modified=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  bool* h_modified_next;
  h_modified_next=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<long> <<<numBlocks,threadsPerBlock>>>(V,d_color[i],(long)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_color+h_vertex_partition[i],d_color[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(long),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified+h_vertex_partition[i],d_modified[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified_next[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified_next+h_vertex_partition[i],d_modified_next[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  int fpoint1 = 0; // asst in .cu 
  int** h_fpoint1;
  h_fpoint1 = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint1[i] = (int*)malloc(sizeof(int));
  }

  int** d_fpoint1;
  d_fpoint1 = (int**)malloc(sizeof(int*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_fpoint1[i],0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  int iter = 0; // asst in .cu 
  int** h_iter;
  h_iter = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_iter[i] = (int*)malloc(sizeof(int));
  }

  int** d_iter;
  d_iter = (int**)malloc(sizeof(int*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_iter[i],sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_iter[i],0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  do{
    int x11=rand();
    int x12=rand();
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
      hiprandSetPseudoRandomGeneratorSeed(gen,x11);
      hiprandGenerate(gen,(unsigned int*)d_color1[i],(V+1));
      hiprandSetPseudoRandomGeneratorSeed(gen,x12);
      hiprandGenerate(gen,(unsigned int*)d_color2[i],(V+1));
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      combineRandom<<<numBlocks,numThreads>>>(V,d_color[i],d_color1[i],d_color2[i]);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      colorGraph_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_modified[i],d_fpoint1[i],d_color[i],d_modified_next[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    if(devicecount==1){
    }
    if(devicecount>1){
      //v v
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified_next+h_vertex_partition[i],d_modified_next[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified_next[i],h_modified_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i],d_fpoint1[i],sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    int fpoint1_=0;
    for(int i=0;i<devicecount;i++){
      fpoint1_ += h_fpoint1[i][0];
    } //end of for
    fpoint1=fpoint1_;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    iter = iter + 1;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      //printed here

      initKernel<int> <<<1,1>>>(1,d_iter[i],(int)iter);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }while(fpoint1 < numNodes);
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
