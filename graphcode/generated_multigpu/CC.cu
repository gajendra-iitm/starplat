// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "CC.h"

void Compute_CC(graph& g,float* CC,std::set<int>& sourceSet)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  float* h_CC;
  h_CC= (float*)malloc(sizeof(float)*(V+1));
  float** d_CC;
  d_CC = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_CC[i], sizeof(float)*(V+1));
  }


  //BEGIN DSL PARSING 
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_CC[i],(float)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_CC+h_vertex_partition[i],d_CC[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  int numNodes = g.num_nodes( ); // asst in .cu 
  //fixed_pt_var

  int* h_dist;
  h_dist=(int*)malloc(sizeof(int)*(V+1));
  int** d_dist;
  d_dist = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_dist[i], sizeof(int)*(V+1));
  }

  bool* h_modified;
  h_modified=(bool*)malloc(sizeof(bool)*(V+1));
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  //FOR SIGNATURE of SET - Assumes set for on .cu only
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++) 
  {
    int src = *itr;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist[i],(int)INT_MAX);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_dist+h_vertex_partition[i],d_dist[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified+h_vertex_partition[i],d_modified[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //hi2
    h_modified[src]=true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initIndex<bool><<<1,1>>>(V,d_modified[i],src,(bool)true); //InitIndexDevice
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //hi2
    h_dist[src]=0;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initIndex<int><<<1,1>>>(V,d_dist[i],src,(int)0); //InitIndexDevice
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    bool finished = false; // asst in .cu 
    //fixed_pt_var
    bool** h_finished;
    h_finished = (bool**)malloc(sizeof(bool*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_finished[i] = (bool*)malloc(sizeof(bool));
    }

    bool** d_finished;
    d_finished = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_finished[i],sizeof(bool));
      initKernel<bool> <<<1,1>>>(1,d_finished[i],false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    bool** d_modified_next;
    d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
    for (int i = 0; i < devicecount; i++) {
      hipSetDevice(i);
      hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
    }


    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    while(!finished) {

      finished = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_finished[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        Compute_CC_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_dist[i],d_modified[i],d_modified_next[i],d_finished[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      //global loop var v iden v
      if(devicecount>1){
        //pull based

        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_dist+h_vertex_partition[i],d_dist[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_dist[i],h_dist,sizeof(int)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      bool* h_modified1;
      if(devicecount==1){
        hipMemcpy(d_modified[0],d_modified_next[0],sizeof(bool)*(V+1),hipMemcpyDeviceToDevice);
      }
      if(devicecount>1){
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_modified+h_vertex_partition[i],d_modified_next[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_modified[i],h_modified,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        }
        for(int i=0;i<devicecount;i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_finished[i], d_finished[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        finished&=h_finished[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_finished[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    int temp = 0; // asst in .cu 
    //fixed_pt_var
    int** h_temp;
    h_temp = (int**)malloc(sizeof(int*)*(devicecount+1));
    for(int i=0;i<=devicecount;i+=1){
      h_temp[i] = (int*)malloc(sizeof(int));
    }

    int** d_temp;
    d_temp = (int**)malloc(sizeof(int*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_temp[i],sizeof(int));
      initKernel<int> <<<1,1>>>(1,d_temp[i],0);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_CC_kernel2<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_temp[i],d_dist[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_temp[i],d_temp[i],sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    int temp_=0;
    for(int i=0;i<devicecount;i++){
      temp_ += h_temp[i][0];
    } //end of for
    temp=temp_;
    //hi2
    h_CC[src]=1.000000 / temp;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initIndex<float><<<1,1>>>(V,d_CC[i],src,(float)1.000000 / temp); //InitIndexDevice
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  }
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    int s = h_vertex_partition[i], e = h_vertex_partition[i+1] ; 
    hipMemcpyAsync(      CC + s,     d_CC[i] + s, sizeof(float)*(e-s), hipMemcpyDeviceToHost);
  }
  for (int i = 0 ; i < devicecount; i++){
    hipSetDevice(i) ; 
    hipDeviceSynchronize();
  }
} //end FUN
