// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "bc_dslV2.h"

void Compute_BC(graph& g,float* BC,std::set<int>& sourceSet)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  float* h_BC;
  h_BC= (float*)malloc(sizeof(float)*(V+1));
  float** d_BC;
  d_BC = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_BC[i], sizeof(float)*(V+1));
  }


  //BEGIN DSL PARSING 
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_BC[i],(float)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_BC+h_vertex_partition[i],d_BC[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  float* h_sigma;
  h_sigma=(float*)malloc(sizeof(float)*(V+1));
  float** d_sigma;
  d_sigma = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_sigma[i], sizeof(float)*(V+1));
  }

  float* h_delta;
  h_delta=(float*)malloc(sizeof(float)*(V+1));
  float** d_delta;
  d_delta = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_delta[i], sizeof(float)*(V+1));
  }

  //FOR SIGNATURE of SET - Assumes set for on .cu only
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++) 
  {
    int src = *itr;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_delta[i],(float)0);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_delta+h_vertex_partition[i],d_delta[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_sigma[i],(float)0);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_sigma+h_vertex_partition[i],d_sigma[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //hi2
    h_sigma[src]=1;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initIndex<float><<<1,1>>>(V,d_sigma[i],src,(float)1); //InitIndexDevice
    }
    for(int i=0;i<devicecount;i+=1){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    bool* h_finished = (bool*)malloc(devicecount * sizeof(bool));
    int hops_from_source = 0 ; 

    bool **d_finished;
    d_finished = (bool**)malloc(sizeof(bool*)* devicecount) ; 
     for (int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_finished[i],sizeof(bool) *(1));
    }

    int **d_hops_from_source;
    d_hops_from_source = (int**)malloc(sizeof(int*)*devicecount); 
    for (int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_hops_from_source[i],sizeof(int) *(1));
    }

    int** d_level ; 
    d_level = (int**)malloc(sizeof(int*)*devicecount) ; 
     for (int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_level[i],sizeof(int) *(V+1));
    }

    int** h_level_temp = (int**)malloc(sizeof(int*) * devicecount);
    for (int i = 0 ; i < devicecount ; i++){
      h_level_temp[i] = (int*)malloc(sizeof(int) * (V+1));
    }

    int* h_level = (int*)malloc(sizeof(int) * (V+1));

    //EXTRA vars INITIALIZATION
    for (int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<int><<<numBlocks,threadsPerBlock>>>(V,d_level[i],-1);
      initIndex<int><<<1,1>>>(V,d_level[i],src, 0);
      hipDeviceSynchronize();
      hipMemcpy(d_hops_from_source[i], &hops_from_source, sizeof(int), hipMemcpyHostToDevice);
    }

    // long k =0 ;// For DEBUG
    h_sigma[src] = 1;

    for (int i =0  ; i < V ; i++){
      h_level[i] = -1;
    }
    do {
      finished = true;
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_finished[i], &finished, sizeof(bool), hipMemcpyHostToDevice);
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }

      //Kernel LAUNCH
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        fwd_pass<<<numBlocksKernel,threadsPerBlock>>>(V, i, h_vertex_partition[i], h_vertex_partition[i+1], d_offset[i], d_edges[i], d_weight[i], d_delta[i], d_sigma[i], d_level[i], d_hops_from_source[i], d_finished[i],d_BC[i]); ///DONE from varList
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for (int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        incrementDeviceVar<<<1,1>>>(d_hops_from_source[i]);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      if (devicecount > 1){
        for (int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          int s = h_vertex_partition[i], e = h_vertex_partition[i+1] ;
          hipMemcpyAsync(h_sigma+s, d_sigma[i]+s, sizeof(float) * (e-s), hipMemcpyDeviceToHost);
        }

        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }


      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(&h_finished[i], d_finished[i], sizeof(bool), hipMemcpyDeviceToHost);
      }

      for (int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }

      for(int i = 0 ; i < devicecount ; i++){
        finished = finished && h_finished[i];
      }

      if (devicecount > 1){
        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipMemcpyAsync(h_level_temp[i], d_level[i], sizeof(int) * (V), hipMemcpyDeviceToHost);
        }

        for (int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }

        for(int i = 0 ; i < V ; i++){
          for(int j = 0 ; j < devicecount ; j++){
            if(h_level_temp[j][i] > h_level[i]){
              h_level[i] = h_level_temp[j][i];
              break ; 
            }
          }
        }

        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_level[i], h_level, sizeof(int) * (V), hipMemcpyHostToDevice);
          hipMemcpyAsync(d_sigma[i], h_sigma, sizeof(float) * (V), hipMemcpyHostToDevice);
        }

        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }

      }
    }while(!finished);

    hops_from_source--;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_hops_from_source[i], &hops_from_source, sizeof(int), hipMemcpyHostToDevice) ; 
    }

    for (int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        back_pass<<<numBlocksKernel,threadsPerBlock>>>(V, i, h_vertex_partition[i], h_vertex_partition[i+1], d_offset[i], d_edges[i], d_weight[i], d_delta[i], d_sigma[i], d_level[i], d_hops_from_source[i], d_finished[i]
          ,d_BC[i]);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hops_from_source--;
      for (int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        decrementDeviceVar<<<1,1>>>(d_hops_from_source[i]);
      }

      for (int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i) ; 
        hipDeviceSynchronize() ; 
      }
      if (devicecount > 1) {
        for (int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          int s = h_vertex_partition[i], e = h_vertex_partition[i+1];
          hipMemcpyAsync(h_delta+s, d_delta[i]+s, sizeof(float) * (e-s), hipMemcpyDeviceToHost);
        }
        for (int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }

        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipMemcpyAsync(d_delta[i], h_delta, sizeof(float) * (V), hipMemcpyHostToDevice);
        }

        for(int i = 0 ; i < devicecount ; i++){
          hipSetDevice(i);
          hipDeviceSynchronize();
        }
      }

    }
    //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);
  }
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    int s = h_vertex_partition[i], e = h_vertex_partition[i+1] ; 
    hipMemcpyAsync(      BC + s,     d_BC[i] + s, sizeof(float)*(e-s), hipMemcpyDeviceToHost);
  }
  for (int i = 0 ; i < devicecount; i++){
    hipSetDevice(i) ; 
    hipDeviceSynchronize();
  }
} //end FUN
