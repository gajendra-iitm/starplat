// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRankDSLV2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  float** d_beta;
  d_beta = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_beta[i], sizeof(float));
    initKernel<float> <<<1,1>>>(1,d_beta[i],beta);
  }

  float** h_beta = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_beta[i] = (float*)malloc(sizeof(float));
  }
  float** d_delta;
  d_delta = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_delta[i], sizeof(float));
    initKernel<float> <<<1,1>>>(1,d_delta[i],delta);
  }

  float** h_delta = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_delta[i] = (float*)malloc(sizeof(float));
  }
  int** d_maxIter;
  d_maxIter = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_maxIter[i], sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_maxIter[i],maxIter);
  }

  int** h_maxIter = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_maxIter[i] = (int*)malloc(sizeof(int));
  }
  float* h_pageRank;
  h_pageRank= (float*)malloc(sizeof(float)*(V+1));
  float** d_pageRank;
  d_pageRank = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pageRank[i], sizeof(float)*(V+1));
  }


  //BEGIN DSL PARSING 
  float num_nodes = (float)g.num_nodes( ); // asst in .cu 
  //fixed_pt_var
  float** h_num_nodes;
  h_num_nodes = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_num_nodes[i] = (float*)malloc(sizeof(float));
  }

  float** d_num_nodes;
  d_num_nodes = (float**)malloc(sizeof(float*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_num_nodes[i],sizeof(float));
    initKernel<float> <<<1,1>>>(1,d_num_nodes[i],g.num_nodes( ));
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  float* h_pageRank_nxt;
  h_pageRank_nxt=(float*)malloc(sizeof(float)*(V+1));
  float** d_pageRank_nxt;
  d_pageRank_nxt = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pageRank_nxt[i], sizeof(float)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank[i],(float)1 / num_nodes);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_pageRank+h_vertex_partition[i],d_pageRank[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  int iterCount = 0; // asst in .cu 
  //fixed_pt_var

  float diff; // asst in .cu 
  //fixed_pt_var

  do{
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_PR_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_delta[i],d_num_nodes[i],d_pageRank[i],d_pageRank_nxt[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //fixed_pt_var

    //fixed_pt_var

    if(devicecount>1){
      //v v
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_pageRank_nxt+h_vertex_partition[i],d_pageRank_nxt[i]+h_vertex_partition[i],sizeof(float)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_pageRank_nxt[i],h_pageRank_nxt,sizeof(float)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpy(d_pageRank[i],d_pageRank_nxt[i],sizeof(float)*(V+1),hipMemcpyDeviceToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    iterCount++;
  }while((diff > beta) && (iterCount < maxIter));
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    int s = h_vertex_partition[i], e = h_vertex_partition[i+1] ; 
    hipMemcpyAsync(pageRank + s, d_pageRank[i] + s, sizeof(float)*(e-s), hipMemcpyDeviceToHost);
  }
  for (int i = 0 ; i < devicecount; i++){
    hipSetDevice(i) ; 
    hipDeviceSynchronize();
  }
} //end FUN
