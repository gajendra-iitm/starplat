// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_dsl.h"

void Boruvka(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int** h_nodeId;
  h_nodeId = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_nodeId[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_nodeId;
  d_nodeId = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_nodeId[i], sizeof(int)*(V+1));
  }

  int** h_color;
  h_color = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_color[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_color;
  d_color = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_color[i], sizeof(int)*(V+1));
  }

  bool** h_isMSTEdge;
  h_isMSTEdge = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_isMSTEdge[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_isMSTEdge;
  d_isMSTEdge = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_isMSTEdge[i], sizeof(bool)*E);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_nodeId[i],(int)-1);
  }for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_nodeId[i],d_nodeId[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color[i],(int)-1);
  }for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_color[i],d_color[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    Boruvka_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_color[i],d_nodeId[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  //u u
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_nodeId[devicecount]+h_vertex_partition[i],d_nodeId[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_nodeId[i],h_nodeId[devicecount],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_nodeId[i],d_nodeId[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  //u u
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_color[devicecount]+h_vertex_partition[i],d_color[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_color[i],h_color[devicecount],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_color[i],d_color[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
