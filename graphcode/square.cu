#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

// CUDA Kernel function to square a single number
__global__ void square(float *d_out, float *d_in) {
    *d_out = (*d_in) * (*d_in);
}

// CUDA Kernel function to calculate the square root
__global__ void squareRoot(float *d_out, float *d_in) {
    *d_out = sqrtf(*d_in);
}

int main() {
    // Number to be squared
    float h_in = 3.0f; // Change this value to square a different number
    float h_squared, h_sqrt;

    // Device pointers
    float *d_in, *d_squared, *d_sqrt;

    // Allocate memory on the device
    hipMalloc((void**)&d_in, sizeof(float));
    hipMalloc((void**)&d_squared, sizeof(float));
    hipMalloc((void**)&d_sqrt, sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_in, &h_in, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to square the number
    square<<<1, 1>>>(d_squared, d_in);
    hipDeviceSynchronize(); // Ensure the square operation is complete

    // Launch kernel to calculate the square root of the squared number
    squareRoot<<<1, 1>>>(d_sqrt, d_squared);
    hipDeviceSynchronize(); // Ensure the square root operation is complete

    // Copy results back from device to host
    hipMemcpy(&h_squared, d_squared, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sqrt, d_sqrt, sizeof(float), hipMemcpyDeviceToHost);

    // Print the results
    std::cout << "The square of " << h_in << " is " << h_squared << std::endl;
    std::cout << "The square root of " << h_squared << " is " << h_sqrt << std::endl;

    // Free device memory
    hipFree(d_in);
    hipFree(d_squared);
    hipFree(d_sqrt);

    return 0;
}

