#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "graph.cpp"
using namespace std;

env environment("cuda", "static", "test.cu");

class Layer
{
public:
	int32_t num_features;
	double *weights;
	double *bias;
	double epsilon = 0.01;
	double grad_epsilon;
	double *aggregatedFeatures;
	double *preActivatedFeatures;
	double *postActivatedFeatures;
	double *grad_pre_act_output;
	double *grad_weights;
	double *grad_bias;
	double *m_weights;
	double *m_biases;
	double m_epsilon = 0.0f;
	double v_epsilon = 0.0f;
	double *v_weights;
	double *v_biases;

	Layer() : num_features(0), weights(nullptr), bias(nullptr), epsilon(0.01), grad_epsilon(0.0), aggregatedFeatures(nullptr), preActivatedFeatures(nullptr), postActivatedFeatures(nullptr), grad_pre_act_output(nullptr), grad_weights(nullptr), grad_bias(nullptr), m_weights(nullptr), m_biases(nullptr), m_epsilon(0.0f), v_epsilon(0.0f), v_weights(nullptr), v_biases(nullptr) {}

	__device__ void xaviers(double *weights, int num_feat_current, int num_feat_prev)
	{
		double x = sqrt(6.0f / (num_feat_current + num_feat_prev));
		hiprandState state;
		printf("num_feat_current: %d\n", num_feat_current);
		hiprand_init(3566, 5, 0, &state);
		for (int i = 0; i < num_feat_current * num_feat_prev; i++)
		{
			hiprandState state;
			hiprand_init(3566, i, 0, &state);
			float random_value = hiprand_uniform(&state) * x;
			weights[i] = random_value;
			// printf("%f\n",weights[i]);
		}
	}

	__device__ void he(double *weights, int num_feat_current, int num_feat_prev)
	{
		double x = sqrt(6.0f / (num_feat_prev));
		hiprandState state;
		printf("num_feat_current: %d\n", num_feat_current);
		hiprand_init(3566, 5, 0, &state);
		for (int i = 0; i < num_feat_current * num_feat_prev; i++)
		{
			hiprandState state;
			hiprand_init(3566, i, 0, &state);
			float random_value = hiprand_uniform(&state) * x;
			weights[i] = random_value;
			// printf("%f\n",weights[i]);
		}
	}
};

class cudaVars
{
public:
	Layer *layers;
	int *d_nodesPtr;
	int *d_edgesList;
	float *d_edgeWeights;
};

cudaVars globalCudaVar;

__global__ void initializeLayerFields(Layer *layers, int num_layers, int *num_features, double *d_features, int num_nodes, int initialization_type)
{
	int i = threadIdx.x;
	if (i == 0)
	{
		layers[i].num_features = num_features[i];
		for (int j = 0; j < num_features[0] * num_nodes; j++)
		{
			layers[i].preActivatedFeatures[j] = 0;
			layers[i].postActivatedFeatures[j] = d_features[j];
		}
	}

	else if (i < num_layers)
	{
		layers[i].num_features = num_features[i];
		if (initialization_type == 1)
		{
			layers[i].xaviers(layers[i].weights, num_features[i - 1], num_features[i]);
		}
		else if (initialization_type == 2)
		{
			layers[i].he(layers[i].weights, num_features[i - 1], num_features[i]);
		}
		for (int j = 0; j < num_features[i]; j++)
		{
			layers[i].bias[j] = 0.5;
			layers[i].grad_bias[j] = 0;
			layers[i].m_biases[j] = 0;
			layers[i].v_biases[j] = 0;
		}

		for (int j = 0; j < num_features[i - 1] * num_features[i]; j++)
		{
			layers[i].grad_weights[j] = 0;
			layers[i].m_weights[j] = 0;
			layers[i].v_weights[j] = 0;
		}

		for (int j = 0; j < num_features[i] * num_nodes; j++)
		{
			layers[i].aggregatedFeatures[j] = 0;
			layers[i].preActivatedFeatures[j] = 0;
			layers[i].postActivatedFeatures[j] = 0;
			layers[i].grad_pre_act_output[j] = 0;
		}
	}
}

void initializeLayers_cuda(GNN &gnn, Layer *d_layers, vector<int> numFeaturesPerLayer, double *d_features, int num_nodes, const char *transformation)
{
	int *d_numFeaturesPerLayer;
	hipMalloc(&d_numFeaturesPerLayer, numFeaturesPerLayer.size() * sizeof(int));
	hipMemcpy(d_numFeaturesPerLayer, numFeaturesPerLayer.data(), numFeaturesPerLayer.size() * sizeof(int), hipMemcpyHostToDevice);

	int initialization_type;
	if (transformation == "xaviers")
	{
		initialization_type = 1;
	}
	else if (transformation == "he")
	{
		initialization_type = 2;
	}

	initializeLayerFields<<<1, numFeaturesPerLayer.size()>>>(d_layers, numFeaturesPerLayer.size(), d_numFeaturesPerLayer, d_features, num_nodes, initialization_type);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	}
	else
	{
		printf("Success\n");
	}

	globalCudaVar.layers = d_layers;

	hipMalloc(&globalCudaVar.d_nodesPtr, gnn.getGraph().num_nodes() * sizeof(int));
	hipMemcpy(globalCudaVar.d_nodesPtr, gnn.getGraph().getNodesptr(), gnn.getGraph().num_nodes() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_edgesList, gnn.getGraph().num_edges() * sizeof(int));
	hipMemcpy(globalCudaVar.d_edgesList, gnn.getGraph().getEdgesList(), gnn.getGraph().num_edges() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&globalCudaVar.d_edgeWeights, gnn.getGraph().num_edges() * sizeof(float));
	hipMemcpy(globalCudaVar.d_edgeWeights, gnn.getGraph().getEdgeLen(), gnn.getGraph().num_edges() * sizeof(float), hipMemcpyHostToDevice);

	// copy the features to a double  and print the features of node 4

	// copy the weights of layer 1 to a double array and print the first 10 weights
	//  double *weights = new double[numFeaturesPerLayer[0] * numFeaturesPerLayer[1]];
	//  hipMemcpy(weights, h_layers[2].weights, numFeaturesPerLayer[0] * numFeaturesPerLayer[1] * sizeof(double), hipMemcpyDeviceToHost);
	//  for (int i = 0; i < 10; i++)
	//  {
	//  	cout << weights[i] << " ";
	//  }
}

__global__ void GCN_cuda(Layer* layers, int batchNumber, int batchSize, int *V, int *E, float *edgeWeights, int layerNumber)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		int idx = batchNumber * batchSize + tid;
		for (int i = 0; i < layers[layerNumber].num_features; i++)
		{
			layers[layerNumber].aggregatedFeatures[idx * layers[layerNumber].num_features + i] = 0;
		}

		for (int j = V[idx]; j < V[idx + 1]; j++)
		{
			int neighbor = E[j];
			double weight = edgeWeights[j];
			for (int i = 0; i < layers[layerNumber].num_features; i++)
			{
				layers[layerNumber].aggregatedFeatures[idx * layers[layerNumber].num_features + i] += layers[layerNumber - 1].postActivatedFeatures[neighbor * layers[layerNumber].num_features + i] * weight;
			}
		}

		printf("Node %d: aggregatedFeatures = %f\n", idx, layers[layerNumber].aggregatedFeatures[idx * layers[layerNumber].num_features]);
	}
}

__global__ void NeuralNetworkOperation(Layer* layers, int batchNumber, int batchSize, int totalLayers, int layerNumber, int activationType)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < batchSize)
	{
		// printf("tid: %d\n", tid);

		int idx = batchNumber * batchSize + tid;
		if (layerNumber < totalLayers - 1)
			for (int i = 0; i < layers[layerNumber].num_features; i++)
			{
				float sum = 0.0f;

				for (int j = 0; j < layers[layerNumber - 1].num_features; j++)
				{
					sum += layers[layerNumber].weights[j * layers[layerNumber].num_features + i] *
						   layers[layerNumber].aggregatedFeatures[idx * layers[layerNumber - 1].num_features + j];
				}

				layers[layerNumber].preActivatedFeatures[idx * layers[layerNumber + 1].num_features + i] = sum;
				if (activationType == 1)
				{
					layers[layerNumber].postActivatedFeatures[idx * layers[layerNumber + 1].num_features + i] =
						fmaxf(0.0f, sum);
				}
				else if (activationType == 2)
				{
					layers[layerNumber].postActivatedFeatures[idx * layers[layerNumber + 1].num_features + i] =
						tanh(sum);
				}
				printf("Node %d: preActivatedFeatures = %f\n", idx, layers[layerNumber].preActivatedFeatures[idx * layers[layerNumber + 1].num_features + i]);

				printf("Node %d: postActivatedFeatures = %f\n", idx, layers[layerNumber].postActivatedFeatures[idx * layers[layerNumber + 1].num_features + i]);
			}
	}
}

void forwardPass_cuda(GNN &gnn, int layerNumber, int batchNumber, int batchSize, int activationType)
{
	int numNodesInBatch = batchSize;
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numNodesInBatch + threadsPerBlock - 1) / threadsPerBlock;
	hipStream_t stream1, stream2;

	Layer* layers = globalCudaVar.layers;

	// GCN_cuda<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(layers, numNodes, gnn.getGraph().getNodesptr(), gnn.getGraph().getEdgesList(), gnn.getGraph().getEdgeLen(), layerNumber, node, activationType);
	GCN_cuda<<<blocksPerGrid, threadsPerBlock>>>(layers, batchNumber, batchSize, globalCudaVar.d_nodesPtr, globalCudaVar.d_edgesList, globalCudaVar.d_edgeWeights, layerNumber);
	hipDeviceSynchronize();

	// Launch kernels  to overlap execution
	NeuralNetworkOperation<<<blocksPerGrid, threadsPerBlock>>>(layers,batchNumber, batchSize, 4, layerNumber, activationType);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA ffError: %s\n", hipGetErrorString(err));
	}
	else
	{
		printf("Suffccess\n");
	}
}

// __global__ void computeGradientsOutputLayer(double *y_pred, int *y_true, double *grad_output, int num_nodes, int num_classes)
// {

// 	int nodeIdx = blockIdx.x * blockDim.x + threadIdx.x;

// 	if (nodeIdx < num_nodes)
// 	{
// 		int label = y_true[nodeIdx];
// 		for (int j = 0; j < num_classes; ++j)
// 		{
// 			grad_output[nodeIdx * num_classes + j] = y_pred[nodeIdx * num_classes + j] - (label == j ? 1.0f : 0.0f);
// 		}
// 	}
// }

// __global__ void computeGradientsIntermediateLayer(double *grad_next_layer, double *weights_next_layer, double *grad_current_layer, double *pre_activated_features, int num_nodes, int num_features, int next_layer_features, int activation_type)
// {
// 	int nodeIdx = blockIdx.x * blockDim.x + threadIdx.x;

// 	if (nodeIdx < num_nodes)
// 	{
// 		for (int i = 0; i < num_features; ++i)
// 		{
// 			double grad_sum = 0.0f;
// 			for (int j = 0; j < next_layer_features; ++j)
// 			{
// 				grad_sum += grad_next_layer[nodeIdx * next_layer_features + j] * weights_next_layer[i * next_layer_features + j];
// 			}

// 			if (activation_type == 1)
// 			{ // Tanh
// 				grad_current_layer[nodeIdx * num_features + i] = grad_sum * (1 - pre_activated_features[nodeIdx * num_features + i] * pre_activated_features[nodeIdx * num_features + i]);
// 			}
// 			else if (activation_type == 2)
// 			{ // ReLU
// 				grad_current_layer[nodeIdx * num_features + i] = (pre_activated_features[nodeIdx * num_features + i] > 0) ? grad_sum : 0;
// 			}
// 		}
// 	}
// }
// __global__ void computeWeightBiasGradients(double *aggregated_features, double *grad_pre_act_output, double *grad_weights, double *grad_bias, int num_nodes, int num_input_features, int num_output_features)
// {
// 	int inputIdx = blockIdx.x * blockDim.x + threadIdx.x;

// 	if (inputIdx < num_input_features)
// 	{
// 		for (int j = 0; j < num_output_features; ++j)
// 		{
// 			double weight_grad_sum = 0.0f;
// 			double bias_grad_sum = 0.0f;

// 			for (int nodeIdx = 0; nodeIdx < num_nodes; ++nodeIdx)
// 			{
// 				weight_grad_sum += aggregated_features[nodeIdx * num_input_features + inputIdx] * grad_pre_act_output[nodeIdx * num_output_features + j];
// 				bias_grad_sum += grad_pre_act_output[nodeIdx * num_output_features + j];
// 			}

// 			grad_weights[inputIdx * num_output_features + j] = weight_grad_sum;
// 			if (inputIdx == 0)
// 			{
// 				grad_bias[j] = bias_grad_sum;
// 			}
// 		}
// 	}
// }
// __global__ void clipGradients(double *grad_weights, double *grad_bias, int num_input_features, int num_output_features, double clip_value)
// {
// 	int weightIdx = blockIdx.x * blockDim.x + threadIdx.x;

// 	if (weightIdx < num_input_features * num_output_features)
// 	{
// 		if (abs(grad_weights[weightIdx]) > clip_value)
// 		{
// 			grad_weights[weightIdx] = clip_value * (grad_weights[weightIdx] > 0 ? 1.0f : -1.0f);
// 		}
// 	}

// 	int biasIdx = threadIdx.x;
// 	if (biasIdx < num_output_features)
// 	{
// 		if (abs(grad_bias[biasIdx]) > clip_value)
// 		{
// 			grad_bias[biasIdx] = clip_value * (grad_bias[biasIdx] > 0 ? 1.0f : -1.0f);
// 		}
// 	}
// }

// void backPropagation_cuda(GNN &gnn, int layerNumber, int numNodes, int numInputFeatures, int numOutputFeatures)
// {
// 	double *d_y_pred, *d_grad_output, *d_weights, *d_preActivatedFeatures, *d_aggregatedFeatures, *d_bias;
// 	int *d_y_true;
// 	int totalLayers;
// 	if (layerNumber == totalLayers - 1)
// 	{
// 		int threads = 1024;
// 		int blocks = (numNodes + threads - 1) / threads;
// 		computeGradientsOutputLayer<<<blocks, threads>>>(d_y_pred, d_y_true, d_grad_output, numNodes, gnn.numClasses());
// 	}
// 	else
// 	{
// 		int threads = 1024;
// 		int blocks = (numNodes + threads - 1) / threads;
// 		computeGradientsIntermediateLayer<<<blocks, threads>>>(d_grad_output, d_weights, d_grad_output, d_preActivatedFeatures, numNodes, numInputFeatures, numOutputFeatures, gnn.initType());
// 	}

// 	int threads = 1024;
// 	int blocks = (numInputFeatures + threads - 1) / threads;
// 	computeWeightBiasGradients<<<blocks, threads>>>(d_aggregatedFeatures, d_grad_output, d_weights, d_bias, numNodes, numInputFeatures, numOutputFeatures);
// 	clipGradients<<<blocks, threads>>>(d_weights, d_bias, numInputFeatures, numOutputFeatures, 3.0);
// }

int main()
{
	vector<int> numFeaturesPerLayer = {16, 12, 8};
	const char *transformation = "xaviers";
	graph G("/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph.txt");
	G.parseGraph();

	GNN gnn(G, "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph_feat.txt", "/home/anubhav/new_q/newww_tha/Starplat-GNN/graphcode/sample_graphs/sample_graph/sample_graph_labels.txt");
	gnn.gcnPreprocessing();
	cout << "preprosessing done" << endl;

	int num_layers = numFeaturesPerLayer.size();

	Layer *d_layers;
	hipMalloc(&d_layers, num_layers * sizeof(Layer));

	Layer *h_layers = new Layer[num_layers];
	for (int i = 0; i < num_layers; i++)
	{
		h_layers[i] = Layer();
		hipMalloc(&h_layers[i].weights, numFeaturesPerLayer[(i > 1 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].bias, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_weights, numFeaturesPerLayer[(i > 1 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_bias, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].m_weights, numFeaturesPerLayer[(i > 1 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].m_biases, numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].v_weights, numFeaturesPerLayer[(i > 1 ? i - 1 : i)] * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].v_biases, numFeaturesPerLayer[i] * sizeof(double));

		hipMalloc(&h_layers[i].aggregatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].preActivatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].postActivatedFeatures, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));
		hipMalloc(&h_layers[i].grad_pre_act_output, gnn.getGraph().num_nodes() * numFeaturesPerLayer[i] * sizeof(double));

		hipMemcpy(&d_layers[i], &h_layers[i], sizeof(Layer), hipMemcpyHostToDevice);
	}
	vector<vector<double>> feats = gnn.getFeatures();
	// convert it to a 1D array
	vector<double> flat_feats;
	for (int i = 0; i < feats.size(); i++)
	{
		for (int j = 0; j < feats[i].size(); j++)
		{
			flat_feats.push_back(feats[i][j]);
		}
	}

	double *d_features;
	hipMalloc(&d_features, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double));
	// .data of vector of vector is a 1D array
	hipMemcpy(d_features, flat_feats.data(), gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyHostToDevice);

	initializeLayers_cuda(gnn, d_layers, numFeaturesPerLayer, d_features, gnn.getGraph().num_nodes(), transformation);
	hipDeviceSynchronize();
	double *features = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	hipMemcpy(features, d_features, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);

	printf("\n\ndone\n\n");
	forwardPass_cuda(gnn, 1, 0, 3, 1);
	//  double *features = new double[num_nodes * feats[0].size()];
	//  hipMemcpy(features, d_features, num_nodes * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);
	//  for (int i = 0; i < feats[0].size(); i++)
	//  {
	//  	cout << features[4 * feats[0].size() + i] << " ";
	//  }

	// print features of node 4
	double *featuress = new double[gnn.getGraph().num_nodes() * feats[0].size()];
	hipMemcpy(featuress, h_layers[1].preActivatedFeatures, gnn.getGraph().num_nodes() * feats[0].size() * sizeof(double), hipMemcpyDeviceToHost);

	return 0;
}
