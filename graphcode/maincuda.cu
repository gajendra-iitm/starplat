//make sure to first generate the cuda backend code:
//by cd into 'src'
//command to run on your terminal to generate "betweeness centrality (for static graphs) cuda backend code" using bc_dslV2 as input dsl:
// ./StarPlat -s -f ../graphcode/staticDSLCodes/bc_dslV2 -b cuda
//for executing on Google collab: remember to add ! before the above command.

//generating the bc_dslV2 cuda backend code by using the above command will generate two files 'bc_dslV2.cu' and 'bc_dslV2.h' 
//in the directory ../graphcode/generated_cuda which is required for this code to run
//as we have the main caller function for that code written here in this file.
//nvcc bc_dslV2mainCuda.cu -o bc_dslV2mainCuda -arch=sm_70 -std=c++14 -rdc=true
//./bc_dslV2mainCuda ../graphcode/generated_cuda/sample_graph.txt ../graphcode/generated_cuda/src_nodes.txt
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "/lfs/usrhome/mtech/cs23m006/new_repo/starplat/graphcode/generated_cuda/APFB_Matching.cu"

// main fn: reads Input and Calls `Compute_BC`
int main(int argc, char* argv[]) {
    char* filePath;
    char* updateFile;
    if (argc == 2) {
        filePath = argv[1];
    } else if (argc==3){
        filePath = argv[1];
        updateFile=argv[2];
        
    }else{
        return 1;
    }
    graph g(filePath);
    g.parseGraph();
    std::cout << "Number of nodes: " << g.num_nodes() << std::endl;
    std::cout << "Number of edges: " << g.num_edges() << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    APFB(g, g.num_nodes());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "\nGPU Time: " << milliseconds / 1000.0 << " seconds" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}