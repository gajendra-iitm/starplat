#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "graph.hpp"
#include<bits/stdc++.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include<bits/stdc++.h>
#include "mst_dsl_boruvkas-modify.h"
using namespace std;




void mst_boruvkas(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));

  int* d_minedge;
  hipMalloc(&d_minedge, sizeof(int)*(V));

  int* d_minweight;
  hipMalloc(&d_minweight, sizeof(int)*(V));

  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_parent,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);


  // Initializing arrays
  mst_boruvkas_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_isCovered,d_destination,d_parent);
  hipDeviceSynchronize();



  bool hasComponent = true; // asst in .cu

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  

  do{
    hasComponent = false;
    bool finished = false; // asst in .cu  // need to shift in dsl code
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);

    // finding min weight for each node
    mst_boruvkas_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered);
    hipDeviceSynchronize();



    ; // asst in .cu
    // finding min edge for each node   // can club 2 and 3 to one
    mst_boruvkas_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered,d_minedge);
    hipDeviceSynchronize();



    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)true);

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_minedge,d_minweight,d_modified,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    mst_boruvkas_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::hasComponent), &hasComponent, sizeof(bool), 0, hipMemcpyHostToDevice);
    mst_boruvkas_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge,d_isCovered);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&hasComponent, HIP_SYMBOL(::hasComponent), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    finished = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


  }while(hasComponent);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_minweight);
  hipFree(d_minedge);
  hipFree(d_modified);
  hipFree(d_parent);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  bool *h_isCovered;
  h_isCovered = (bool*)malloc(sizeof(bool)*E);
  hipMemcpy(h_isCovered,d_isCovered,hipMemcpyDeviceToHost);
  //PANIC Need to create edge weight array 
  long long int mstw = 0; 
  for(int i=0;i<E;i++)
  {
    if(h_isCovered[i]){
      mstw+=edgeLen[i];
    }
  }
  return mstw;
} //end FUN

int  main( int  argc, char** argv) {
  char* totalgraph=argv[1];
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();
  long long int eval = Boruvka(G1);
  printf("MST weight:%lld\n",eval);
  return 0;
}