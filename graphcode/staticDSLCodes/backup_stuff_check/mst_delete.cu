// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_delete.h"

void recolor(bool* isCovered,graph& g,int* parent,int* destination
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));

  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));


  //BEGIN DSL PARSING 
  recolor_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
  hipDeviceSynchronize();



  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)true);

  bool rfinish1 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!rfinish1) {

    rfinish1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::rfinish1), &rfinish1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::rfinish1), &rfinish1, sizeof(bool), 0, hipMemcpyHostToDevice);
    recolor_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_parent);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&rfinish1, HIP_SYMBOL(::rfinish1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    bool rfinish2 = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!rfinish2) {

      rfinish2 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::rfinish2), &rfinish2, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::rfinish2), &rfinish2, sizeof(bool), 0, hipMemcpyHostToDevice);
      recolor_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&rfinish2, HIP_SYMBOL(::rfinish2), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&rfinish2, HIP_SYMBOL(::rfinish2), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


    hipMemcpyFromSymbol(&rfinish1, HIP_SYMBOL(::rfinish1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(isCovered, d_isCovered, sizeof(bool)*(E), hipMemcpyDeviceToHost);
  hipMemcpy(  parent, d_parent, sizeof(int)*(V), hipMemcpyDeviceToHost);
  hipMemcpy(destination, d_destination, sizeof(int)*(E), hipMemcpyDeviceToHost);
} //end FUN
void mst_boruvkas_add_edges(graph& g,bool* isCovered,int* parent)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));

  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));


  //BEGIN DSL PARSING 
  int* d_minedge;
  hipMalloc(&d_minedge, sizeof(int)*(V));

  int* d_minweight;
  hipMalloc(&d_minweight, sizeof(int)*(V));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_parent,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);

  mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_destination);
  hipDeviceSynchronize();



  bool hasComponent = true; // asst in .cu

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  bool finished = false; // asst in .cu

  do{
    hasComponent = false;
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);

    mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered);
    hipDeviceSynchronize();



    ; // asst in .cu

    mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered,d_minedge);
    hipDeviceSynchronize();



    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)true);

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_modified,d_minedge,d_minweight,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::hasComponent), &hasComponent, sizeof(bool), 0, hipMemcpyHostToDevice);
    mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge,d_isCovered);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&hasComponent, HIP_SYMBOL(::hasComponent), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    finished = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_add_edges_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


  }while(hasComponent);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);
  hipFree(d_minweight);
  hipFree(d_minedge);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(isCovered, d_isCovered, sizeof(bool)*(E), hipMemcpyDeviceToHost);
  hipMemcpy(  parent, d_parent, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN
void total_code(graph& g,bool* isCovered)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));


  //BEGIN DSL PARSING 
  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_parent,(int)-1);

  total_code_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_destination,d_parent);
  hipDeviceSynchronize();



  (isCovered,g,parent,destination);

  (g,isCovered,parent);


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_parent);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(isCovered, d_isCovered, sizeof(bool)*(E), hipMemcpyDeviceToHost);
} //end FUN
