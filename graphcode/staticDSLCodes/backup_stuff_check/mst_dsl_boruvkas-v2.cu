// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_dsl_boruvkas-v2.h"

void checkCudaError( int  i)
{       
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)  
  {   
    printf("%d , CUDA error: %s\n", i, hipGetErrorString(error));
  } 
} 
long long int mst_boruvkas(graph& g)
{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();
  checkCudaError(1);
  int *h_meta;
  int *h_data;
  // int *h_weight;// #### MY ADDITION ###

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  // h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    // addition of edgeLen
    // temp = edgeLen[i]; // #### MY ADDITION ###
    // h_weight[i] = temp;// #### MY ADDITION ###
  }


  int* d_meta;
  int* d_data;
  int* d_weight; // #### MY ADDITION ###

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));// #### MY ADDITION ###
  checkCudaError(2);
  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(  d_weight,   edgeLen, sizeof(int)*(E), hipMemcpyHostToDevice);// #### MY ADDITION ###
checkCudaError(3);
  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));

  int* d_minedge;
  hipMalloc(&d_minedge, sizeof(int)*(V));

  int* d_minweight;
  hipMalloc(&d_minweight, sizeof(int)*(V));

  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));
  checkCudaError(4);
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_parent,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);
  printf("BK1");
  checkCudaError(5);
  mst_boruvkas_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_parent,d_isCovered,d_destination);
  checkCudaError(6);
  hipDeviceSynchronize();
checkCudaError(7);


  bool hasComponent = true; // asst in .cu

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  bool finished = false; // asst in .cu
  int dowhile =0;
  do{
    hasComponent = false;
    printf("dowhile:%d\n",dowhile);
    dowhile++;
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);

    mst_boruvkas_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered,d_minedge,d_weight);
    hipDeviceSynchronize();



    ; // asst in .cu

    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)true);

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_minweight,d_minedge,d_modified,d_modified_next,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
      printf("k1:%d\n",k);
    } // END FIXED POINT

    mst_boruvkas_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::hasComponent), &hasComponent, sizeof(bool), 0, hipMemcpyHostToDevice);
    mst_boruvkas_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge,d_isCovered);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&hasComponent, HIP_SYMBOL(::hasComponent), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    finished = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
      printf("k2:%d\n",k);
    } // END FIXED POINT


  }while(hasComponent);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_minweight);
  hipFree(d_minedge);
  hipFree(d_modified);
  hipFree(d_parent);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  bool *h_isCovered;
  h_isCovered = (bool*)malloc(sizeof(bool)*E);
  hipMemcpy(h_isCovered,d_isCovered,sizeof(bool)*E,hipMemcpyDeviceToHost);
  //PANIC Need to create edge weight array 
  long long int mstw = 0; 
  for(int i=0;i<E;i++)
  {
    if(h_isCovered[i]){
      printf("i:%d, w:%d\n",i,edgeLen[i]);
      mstw+=edgeLen[i];
    }
  }
  return mstw;

} //end FUN


int  main( int  argc, char** argv) {
  char* totalgraph=argv[1];
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();
  long long int eval = mst_boruvkas(G1);
  printf("MST weight:%lld\n",eval);
  return 0;
}

