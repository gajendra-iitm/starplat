// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_dsl_boruvkas-modify.h"

void mst_boruvkas(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }


  int* d_meta;
  int* d_data;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_rank;
  hipMalloc(&d_rank, sizeof(int)*(V));

  int* d_parent;
  hipMalloc(&d_parent, sizeof(int)*(V));

  int* d_minedge;
  hipMalloc(&d_minedge, sizeof(int)*(V));

  int* d_minweight;
  hipMalloc(&d_minweight, sizeof(int)*(V));

  bool* d_isCovered;
  hipMalloc(&d_isCovered, sizeof(bool)*(E));

  int* d_destination;
  hipMalloc(&d_destination, sizeof(int)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_rank,(int)0);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_parent,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);

  mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_isCovered,d_destination,d_parent,d_rank);
  hipDeviceSynchronize();



  bool hasComponent = true; // asst in .cu

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  bool finished = false; // asst in .cu

  do{
    hasComponent = false;
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minedge,(int)-1);

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minweight,(int)-1);

    mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered);
    hipDeviceSynchronize();



    ; // asst in .cu

    mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_minweight,d_isCovered,d_minedge);
    hipDeviceSynchronize();



    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)true);

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_modified,d_minedge,d_minweight,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::hasComponent), &hasComponent, sizeof(bool), 0, hipMemcpyHostToDevice);
    mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent,d_destination,d_minedge,d_isCovered);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&hasComponent, HIP_SYMBOL(::hasComponent), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    finished = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      mst_boruvkas_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_parent);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


  }while(hasComponent);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);
  hipFree(d_minweight);
  hipFree(d_minedge);
  hipFree(d_parent);
  hipFree(d_rank);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
