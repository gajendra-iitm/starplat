// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "lock-free-maxflow-v2.h"
// #include "graph.hpp"
#include<bits/stdc++.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include<bits/stdc++.h>

int do_max_flow(graph& g,int source,int sink)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight; //  ADDITION

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  // Need to sort out weight
  h_weight = (int *)malloc( (E)*sizeof(int));//  ADDITION

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    h_weight[i] = edgeLen[i]; // ADDED
  }


  int* d_meta;
  int* d_data;
  int* d_weight; // ADDED

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));// ADDED

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(  d_weight,   h_weight, sizeof(int)*(E), hipMemcpyHostToDevice); // ADDED

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

  do_max_flow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_reverse_edge,d_weight);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  do_max_flow_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_reverse_edge,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  bool flag = true; // asst in .cu

  do{
    flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag), &flag, sizeof(bool), 0, hipMemcpyHostToDevice);
    do_max_flow_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_reverse_edge,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag, HIP_SYMBOL(::flag), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag);

  //hipFree up!! all propVars in this BLOCK!


  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);
  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost);
  int val = h_excess[sink];
  hipFree(d_excess);
  hipFree(d_height);
  free(h_excess);
  return val;

} //end FUN
int  main( int  argc, char** argv) {
  char* totalgraph=argv[1];
  int source = atoi(argv[2]);
  int sink = atoi(argv[3]);
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();
  int maxflow = do_max_flow(G1,source,sink);
  hipDeviceSynchronize();
  printf("Maxflow:%d\n",maxflow);
  return 0;
}