#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
// #include "graph.hpp"
#include "lock-free-maxflow-v3.h"
#include<bits/stdc++.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include<bits/stdc++.h>



__global__ void update_kernel(int src, int dest,int new_capacity, int* d_residual_capacity,int *d_reverse_edge,int *d_weight, int* d_meta,int* d_data){
  int backward_edge = -1;
  int forward_edge = -1;
  for( int  i=d_meta[src];i<d_meta[src+1];i++)
  {
    if(d_data[i]==dest)
    {
      forward_edge = i;
      break;
    }
  }
  backward_edge = d_reverse_edge[forward_edge];
  printf("src:%d dest:%d new_capacity:%d old_capacity:%d residual[src,dst]:%d residual[dst,src]:%d\n",src,dest,new_capacity,d_weight[forward_edge],d_residual_capacity[forward_edge],d_residual_capacity[backward_edge]);

  if(new_capacity<d_weight[forward_edge]){
    if(d_residual_capacity[backward_edge]>new_capacity){
      d_residual_capacity[backward_edge] = new_capacity;
    }
    d_residual_capacity[forward_edge] = new_capacity - d_residual_capacity[backward_edge];
    
  } else{
     d_residual_capacity[forward_edge] =  d_residual_capacity[forward_edge] + new_capacity - d_weight[forward_edge];
  }
  d_weight[forward_edge] = new_capacity;
}

int do_max_flow(graph& g,int source,int sink)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight; //  ADDITION

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  // Need to sort out weight
  h_weight = (int *)malloc( (E)*sizeof(int));//  ADDITION

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    h_weight[i] = edgeLen[i]; // ADDED
  }


  int* d_meta;
  int* d_data;
  int* d_weight; // ADDED

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));// ADDED

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(  d_weight,   h_weight, sizeof(int)*(E), hipMemcpyHostToDevice); // ADDED

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

  do_max_flow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_reverse_edge,d_weight);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  do_max_flow_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_reverse_edge,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  bool flag = true; // asst in .cu

  do{
    flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag), &flag, sizeof(bool), 0, hipMemcpyHostToDevice);
    do_max_flow_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_reverse_edge,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag, HIP_SYMBOL(::flag), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);
  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost);
  int val = h_excess[sink];
  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_excess);
  hipFree(d_height);
  free(h_excess);
  return val;
} //end FUN



int recalculate_max_flow(graph& g, char *updatesinp,int source2,int sink2)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight; //  ADDITION


  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  // Need to sort out weight
  h_weight = (int *)malloc( (E)*sizeof(int));//  ADDITION

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    h_weight[i] = edgeLen[i]; // ADDED  
  }


  int* d_meta;
  int* d_data;
  int* d_weight; // ADDED

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));// ADDED

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(  d_weight,   h_weight, sizeof(int)*(E), hipMemcpyHostToDevice); // ADDED

    int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

  recalculate_max_flow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_reverse_edge,d_weight);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
  recalculate_max_flow_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_reverse_edge,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  bool flag2 = true; // asst in .cu

  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    recalculate_max_flow_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_reverse_edge,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag2);

  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost);
  printf("Static Maxflow:%d\n",h_excess[sink2]);
  printf("After static calculation\n");
  print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
  print_residual<<<numBlocks, threadsPerBlock>>>(V, d_meta,d_residual_capacity,d_data);
  // recalculate_max_flow_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_residual_capacity,d_excess,d_meta,d_weight);
  // hipDeviceSynchronize();

  std::vector<update> updateEdges=g.parseUpdates(updatesinp);

  for(auto &u:updateEdges){
    update_kernel<<<1,1>>>(u.source,u.destination,u.weight,d_residual_capacity,d_reverse_edge,d_weight,d_meta,d_data);
  }
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);
  recalculate_max_flow_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_residual_capacity,d_excess,d_meta,d_weight);
  hipDeviceSynchronize();
  
  printf("After updates \n");
  print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
  print_residual<<<numBlocks, threadsPerBlock>>>(V, d_meta,d_residual_capacity,d_data);
  int* d_height2;
  hipMalloc(&d_height2, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height2,(int)0);

  initIndex<int><<<1,1>>>(V,d_height2,sink2,(int)g.num_nodes()); //InitIndexDevice
  hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
  recalculate_max_flow_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_reverse_edge,d_residual_capacity,d_excess);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  ; // asst in .cu

  flag2 = true;
  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    recalculate_max_flow_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_reverse_edge,d_residual_capacity,d_height2);
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag2);
  printf("After negative max flow calculation\n");
  print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);


  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  flag2 = true;
  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    recalculate_max_flow_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_excess,d_reverse_edge,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag2);
  printf("After +ve maxflow calculation\n");
  print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost);
  int val = h_excess[sink2];
  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_excess);
  hipFree(d_height2);
  hipFree(d_height);
  free(h_excess);
  return val;

} //end FUN

int  main( int  argc, char** argv) {

  char* totalgraph=argv[1];
  char* updatedgraph = argv[2];
  char* updatesinp = argv[3];
  int source = atoi(argv[4]);
  int sink = atoi(argv[5]);
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();

   int reval = recalculate_max_flow(G1,updatesinp,source,sink);
  hipDeviceSynchronize();
  graph G2(updatedgraph,"cuda",true);
  G2.parseGraph();
  int eval = do_max_flow(G2,source,sink);
  if(reval!=eval){
    printf("reval:%d eval%d\n",reval,eval);
    printf("=====================PANIC INCORRECT========================\n");
  } else {
    printf("=====================ANSWER MATCHES========================\n");
  }
  return 0;
}