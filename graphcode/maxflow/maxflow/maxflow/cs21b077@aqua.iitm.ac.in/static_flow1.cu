#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "static_flow.h"
#define THREADS_PER_BLOCK 1024
void checkCudaError( int  i)
{       
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)  
  {   
    printf("%d , CUDA error: %s\n", i, hipGetErrorString(error));
  } 
} 

__global__ void print_excess(int V,int*d_excess){
   unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
  if(v >= V) return;
  printf("excess[%d]=%d\n",v,d_excess[v]);

}
__global__ void print_residual(int V, int *d_meta,int *d_residual_capacity,int *d_rev_residual_capacity, int *d_data){
     unsigned v = blockIdx.x * blockDim.x + threadIdx.x;
  if(v >= V) return;
  for(int edge = d_meta[v];edge<d_meta[v+1];edge++){
    int dst = d_data[edge];
    printf("residual[%d to %d ]=%d \n residual[%d to %d ]=%d \n ",v,dst,d_residual_capacity[edge],dst,v,d_rev_residual_capacity[edge]);
  }
}

int staticMaxFlow(graph& g,int source2,int sink2,int kernel)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = THREADS_PER_BLOCK;
  unsigned numThreads   = (V < threadsPerBlock)? V: THREADS_PER_BLOCK;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;





  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

    // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  staticMaxFlow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(1);


  hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
  staticMaxFlow_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(5);
  hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
  print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
  print_residual<<<numBlocks, threadsPerBlock>>>(V, d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
  hipDeviceSynchronize();

  ; // asst in .cu

  bool flag1 = true; // asst in .cu

  do{
    flag1 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::kernel), &kernel, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    staticMaxFlow_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_reverse_edge,d_excess,d_height,d_rev_residual_capacity,d_residual_capacity);
    hipDeviceSynchronize();
    checkCudaError(7);
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&kernel, HIP_SYMBOL(::kernel), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
    print_residual<<<numBlocks, threadsPerBlock>>>(V, d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
    hipDeviceSynchronize();

    break;
  }while(flag1);



  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
    //hipFree up!! all propVars in this BLOCK!
  int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);
  hipMemcpy(h_excess,d_excess,sizeof(int)*V,hipMemcpyDeviceToHost);
  int val = h_excess[sink2];
  free(h_excess);
  hipFree(d_height);
  hipFree(d_excess);
  return val;
} //end FUN
int  main( int  argc, char** argv) {

  char* totalgraph=argv[1];
  int source = atoi(argv[2]);
  int sink = atoi(argv[3]);
  int kernel = atoi(argv[4]);
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();
  int eval = staticMaxFlow(G1,source,sink,kernel);
  hipDeviceSynchronize();
  printf("STATIC EVAL VALUE:%d\n",eval);
  return 0;
}