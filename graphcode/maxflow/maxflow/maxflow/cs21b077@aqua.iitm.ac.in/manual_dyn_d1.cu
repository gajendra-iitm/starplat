#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "manual_dyn.h"
#define THREADS_PER_BLOCK 512
void checkCudaError( int  i)
{       
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)  
  {   
    printf("%d , CUDA error: %s\n", i, hipGetErrorString(error));
    exit(0);
  } 
} 
__global__ void update_edges_kernel(int batchelements,update* d_batch_updates, int* d_residual_capacity,int *d_rev_residual_capacity,int *d_weight, int* d_meta,int* d_data){
  
  int id  = blockIdx.x * blockDim.x + threadIdx.x;
  if(id>=batchelements) return;
  update u = d_batch_updates[id];
  int src = u.source;
  int dest = u.destination;
  int new_capacity = u.weight;
  
  char type = u.type;
  int edge = -1;
  for( int  i=d_meta[src];i<d_meta[src+1];i++)
  {
    if(d_data[i]==dest)
    {
      edge = i;
      break;
    }
  }
  printf("update src:%d dst%d newcap:%d old weight:%d\n",src,dest,new_capacity,d_weight[edge]);
  if(new_capacity<d_weight[edge]){
    if(d_rev_residual_capacity[edge]>new_capacity){
      d_rev_residual_capacity[edge] = new_capacity;
    }
    d_residual_capacity[edge] = new_capacity - d_rev_residual_capacity[edge];
    
  } else{
     d_residual_capacity[edge] =  d_residual_capacity[edge] + new_capacity - d_weight[edge];
  }
  d_weight[edge] = new_capacity;
}
__global__ void update_excess(int V, int *d_residual_capacity, int *d_rev_residual_capacity, int *d_meta, int *d_excess ,int *d_reverse_edge, int*d_rev_meta){
  int id  = blockIdx.x * blockDim.x + threadIdx.x;
  if(id>=V) return;
  int e = 0;
  for(int fedge = d_meta[id];fedge<d_meta[id+1];fedge++){
    e-=d_rev_residual_capacity[fedge];
    if(id==4){
      printf("sub d_rev_residual_cap:%d\n",d_rev_residual_capacity[fedge]);
    }
  }
  for(int bedge = d_rev_meta[id];bedge<d_rev_meta[id+1];bedge++){
      int edge = d_reverse_edge[bedge];
      e+=d_rev_residual_capacity[edge];
      if(id==4){
      printf("add d_rev_residual_cap:%d\n",d_rev_residual_capacity[edge]);
    }
  }
  d_excess[id] = e;
}
void staticMaxFlow(graph& g,int source,int sink,int* residual_capacity,int* rev_residual_capacity,int* reverse_edge,int* excess)
{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  
  unsigned numThreads   = (V <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: V;
  unsigned numBlocks    = (V+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;



  int *d_excess ; // ADDED
  hipMalloc(&d_excess,sizeof(int)*V); //ADDED
  int* d_residual_capacity; //ADDED
  hipMalloc(&d_residual_capacity,sizeof(int)*E);//ADDED
  int* d_rev_residual_capacity;//ADDED
  hipMalloc(&d_rev_residual_capacity,sizeof(int)*E);//ADDED
  int* d_reverse_edge;//ADDED
  hipMalloc(&d_reverse_edge,sizeof(int)*E);//ADDED

  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);



  initKernel<int> <<<numBlocks, numThreads>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  initKernel<int> <<<numBlocks, numThreads>>>(V,d_excess,(int)0);

  staticMaxFlow_kernel_1<<<numBlocks,  numThreads>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(1);

    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
  staticMaxFlow_kernel_3<<<numBlocks,  numThreads>>>(V,E,d_meta, d_data, d_weight,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
      checkCudaError(3);
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);
// printf("push from source:%d sink:%d\n",source,sink);
    //  print_excess<<<numBlocks,numThreads>>>(V,d_excess);
  // print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
  
  bool flag1 = true; // asst in .cu

  do{
    flag1 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag1), &flag1, sizeof(bool), 0, hipMemcpyHostToDevice);
    staticMaxFlow_kernel_5<<<numBlocks,  numThreads>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_excess,d_height);
    hipDeviceSynchronize();
    checkCudaError(5);
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag1, HIP_SYMBOL(::flag1), sizeof(bool), 0, hipMemcpyDeviceToHost);
  }while(flag1);

  //hipFree up!! all propVars in this BLOCK!
   print_excess<<<numBlocks,numThreads>>>(V,d_excess);
  print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Initial Static GPU Time: %.6f ms\n", milliseconds);


  hipMemcpy(residual_capacity, d_residual_capacity, sizeof(int)*(E), hipMemcpyDeviceToHost);
  hipMemcpy(rev_residual_capacity, d_rev_residual_capacity, sizeof(int)*(E), hipMemcpyDeviceToHost);
  hipMemcpy(reverse_edge, d_reverse_edge, sizeof(int)*(E), hipMemcpyDeviceToHost);
  hipMemcpy(  excess, d_excess, sizeof(int)*(V), hipMemcpyDeviceToHost);

  // Property declared within function is cleared on its own!
  hipFree(d_height);
  

  // Graph arrays not cleared.
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_src);
  hipFree(d_weight);
  hipFree(d_rev_meta);


  // properties passed in as parameters! not cleared as not malloced
  hipFree(d_excess); //ADDED
  hipFree(d_residual_capacity);//ADDED
  hipFree(d_rev_residual_capacity);//ADDED
  hipFree(d_reverse_edge);//ADDED

  
} //end FUN
void Incremental(int V, int E, int* d_meta,int* d_data,int* d_src,int* d_weight,int* d_rev_meta,int source, int sink, int* d_residual_capacity, int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess,int* d_height)
{
  //LAUNCH CONFIG
  unsigned numThreads   = (V <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: V;
  unsigned numBlocks    = (V+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;

  initKernel<int> <<<numBlocks, numThreads>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)V); //InitIndexDevice
  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  incremental_kernel_8<<<numBlocks,  numThreads>>>(V,E,d_meta,d_data,d_weight,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(8);
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
  printf("AFTER INCREMENTAL PUSH FROM SRC:%d SINK:%d\n",source,sink);
  print_excess<<<numBlocks,numThreads>>>(V,d_excess);
  print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);

  bool flag2 = true;// how to deal with this?

  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    incremental_kernel_10<<<numBlocks,  numThreads>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    checkCudaError(10);
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);
    break;
  }while(flag2);
  printf("AFTER INCREMENTAL LOOP SRC:%d SINK:%d\n",source,sink);
  print_excess<<<numBlocks,numThreads>>>(V,d_excess);
  print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
}
void Decremental( int V, int E, int* d_meta,int* d_data,int* d_src,int* d_weight,int* d_rev_meta, int source, int sink, int* d_residual_capacity, int* d_rev_residual_capacity, int* d_reverse_edge, int* d_excess,int* d_height2)
{
  //LAUNCH CONFIG
  unsigned numThreads   = (V <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: V;
  unsigned numBlocks    = (V+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;

  initKernel<int> <<<numBlocks, numThreads>>>(V,d_height2,(int)0);

  initIndex<int><<<1,1>>>(V,d_height2,sink,(int)V); //InitIndexDevice
  hipDeviceSynchronize();

  hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
  decremental_kernel_16<<<numBlocks,  numThreads>>>(V,E,d_src,d_weight,d_rev_meta,d_reverse_edge,d_excess,d_rev_residual_capacity,d_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(16);
  printf("AFTER DK16\n");
  hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);

  bool flag3 = true; // asst in .cu
  int cnt = 0;
  do{
    flag3 = false;
    cnt++;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag3), &flag3, sizeof(bool), 0, hipMemcpyHostToDevice);
    decremental_kernel_18<<<numBlocks,  numThreads>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_residual_capacity,d_excess,d_reverse_edge,d_rev_residual_capacity,d_height2);
    hipDeviceSynchronize();
    checkCudaError(18);
    printf("AFTER DK18\n");
    print_excess<<<numBlocks,numThreads>>>(V,d_excess);
    print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag3, HIP_SYMBOL(::flag3), sizeof(bool), 0, hipMemcpyDeviceToHost);
  }while(flag3);

}



int recalculate_max_flow(graph& g, std::vector<update> &updateBatch,  int source, int sink,int batchSize)
{
  int V = g.num_nodes();// ADDED
  int E = g.num_edges();//ADDED

 

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));
  
  hipMemset(d_excess,0,sizeof(int)*V);
  hipMemset(d_residual_capacity,0,sizeof(int)*E);
  hipMemset(d_rev_residual_capacity,0,sizeof(int)*E);

  int *residual_capacity,*rev_residual_capacity,*reverse_edge,*excess;//ADDED

  residual_capacity = (int *)malloc(sizeof(int)*E);//ADDED
  rev_residual_capacity = (int *)malloc(sizeof(int)*E);//ADDED
  reverse_edge = (int *)malloc(sizeof(int)*E);//ADDED
  excess = (int *) malloc(sizeof(int)*E);//ADDED
  hipMemcpy(residual_capacity,d_residual_capacity,sizeof(int)*E,hipMemcpyDeviceToHost);//ADDED
  hipMemcpy(rev_residual_capacity,d_rev_residual_capacity,sizeof(int)*E,hipMemcpyDeviceToHost);//ADDED
  hipMemcpy(reverse_edge,d_reverse_edge,sizeof(int)*E,hipMemcpyDeviceToHost);//ADDED
  hipMemcpy(excess,d_excess,sizeof(int)*V,hipMemcpyDeviceToHost);//ADDED

  staticMaxFlow(g,source,sink,residual_capacity,rev_residual_capacity,reverse_edge,excess);

  hipMemcpy(d_residual_capacity,residual_capacity,sizeof(int)*E,hipMemcpyHostToDevice);//ADDED
  hipMemcpy(d_rev_residual_capacity,rev_residual_capacity,sizeof(int)*E,hipMemcpyHostToDevice);//ADDED
  hipMemcpy(d_reverse_edge,reverse_edge,sizeof(int)*E,hipMemcpyHostToDevice);//ADDED
  hipMemcpy(d_excess,excess,sizeof(int)*V,hipMemcpyHostToDevice);//ADDED
  printf("OUT OF STATIC\n");

  unsigned numThreads   = (V <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: V;
  unsigned numBlocks    = (V+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;


  int batchElements = 0;


  update *d_batch_updates;//ADDED
  hipMalloc(&d_batch_updates,sizeof(update)*batchSize);  //ADDED

  // I WANT THE GRAPH ATTRIBUTES PROCESSED BEFORE LIKE THE USED BEFORE COPIED BEFORE
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  int *d_height;
  hipMalloc(&d_height,sizeof(int)*V);
  printf("AFTER ALL BATCH MALLOCS BATCH SIZE:%d\n",batchSize);
  print_excess<<<numBlocks,numThreads>>>(V,d_excess);
  print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
  // TIMER START
  hipEvent_t start2, stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  float milliseconds2 = 0;
  hipEventRecord(start2,0);
  printf("update batch size:%d\n",updateBatch.size());
  for( int updateIndex = 0 ; updateIndex < updateBatch.size() ; updateIndex += batchSize){
    printf("UPDATE INDEX:%d\n",updateIndex);
    if((updateIndex + batchSize) > updateBatch.size())
    {
      batchElements = updateBatch.size() - updateIndex ;
    }
    else batchElements = batchSize ;
    hipMemcpy(d_batch_updates,&updateBatch[updateIndex],batchElements*sizeof(update),hipMemcpyHostToDevice);
    
    unsigned numThreadsupdate   = (batchElements <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: batchElements;
    unsigned numBlocksupdate    = (batchElements+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;

    // HERE WRITING AS A SINGLE KERNEL --> CAN HAVE TWO KERNELS FOR ADD AND DELETE
    update_edges_kernel<<<numBlocksupdate,numThreadsupdate>>>(batchElements,d_batch_updates,d_residual_capacity,d_rev_residual_capacity,d_weight,d_meta,d_data);
    hipDeviceSynchronize();
    printf("AFTER UPDATE EDGES KERNEL\n");
    unsigned numThreadexcess   = (V <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: V;
    unsigned numBlocksexcess    = (V+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;
    update_excess<<<numBlocksexcess,numThreadexcess>>>(V,d_residual_capacity,d_rev_residual_capacity,d_meta,d_excess,d_reverse_edge,d_rev_meta);
    hipDeviceSynchronize();
    printf("AFTER UPDATE EXCESS KERNEL\n");
  print_excess<<<numBlocks,numThreads>>>(V,d_excess);
  print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
  // break;
    Decremental(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,source,sink,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_height);
    printf("AFTER DECREMENTAL\n");
    // break;
    Incremental(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,source,sink,d_residual_capacity,d_rev_residual_capacity,d_reverse_edge,d_excess,d_height);
    printf("AFTER INCREMENTAL\n");
  }
  hipEventRecord(stop2,0);
  hipEventSynchronize(stop2);
  hipEventElapsedTime(&milliseconds2, start2, stop2);
  printf("Batchwise processing GPU Time: %.6f ms\n", milliseconds2);

  int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);
  hipMemcpy(h_excess,d_excess,sizeof(int)*V,hipMemcpyDeviceToHost);
  int val = h_excess[V-1];
  free(h_excess);

  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_src);
  hipFree(d_weight);
  hipFree(d_rev_meta);


  hipFree(d_residual_capacity);
  hipFree(d_rev_residual_capacity);
  hipFree(d_reverse_edge);
  hipFree(d_excess);
  return val;
}

int  main( int  argc, char** argv) {
  char* totalgraph=argv[1];
  char* updatesinp = argv[2];
  int source = atoi(argv[3]);
  int sink = atoi(argv[4]);
  int batchsize = atoi(argv[5]);
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();
  std::vector<update> updateEdges=G1.parseUpdates(updatesinp);
  int reval = recalculate_max_flow(G1,updateEdges,source,sink,batchsize);
  hipDeviceSynchronize();
  printf("REVAL FLOW VALUE:%d\n",reval);
  return 0;
}
