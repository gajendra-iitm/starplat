#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "manual_static.h"
#include<bits/stdc++.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include<bits/stdc++.h>

#define THREADS_PER_BLOCK 512

void checkCudaError( int  i)
{       
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)  
  {   
    printf("%d , CUDA error: %s\n", i, hipGetErrorString(error));
    exit(0);
  } 
} 
__global__ void update_edges_kernel(int batchelements,update* d_batch_updates,int *d_weight, int* d_meta,int* d_data){
  
  int id  = blockIdx.x * blockDim.x + threadIdx.x;
  if(id>=batchelements) return;
  update u = d_batch_updates[id];
  int src = u.source;
  int dest = u.destination;
  int new_capacity = u.weight;
  char type = u.type;
  int edge = -1;
  for( int  i=d_meta[src];i<d_meta[src+1];i++)
  {
    if(d_data[i]==dest)
    {
      edge = i;
      break;
    }
  }
  d_weight[edge] = new_capacity;
}
int calculate_max_flow(graph& g, std::vector<update> updateBatch, int batchSize, int source, int sink)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  // UPDATE THE WEIGHTS
  update *d_batch_updates;//ADDED
  hipMalloc(&d_batch_updates,sizeof(update)*batchSize);  //ADDED
  int batchElements = 0;
  for( int updateIndex = 0 ; updateIndex < updateBatch.size() ; updateIndex += batchSize){
    if((updateIndex + batchSize) > updateBatch.size())
    {
      batchElements = updateBatch.size() - updateIndex ;
    }
    else batchElements = batchSize ;
    hipMemcpy(d_batch_updates,&updateBatch[updateIndex],batchElements*sizeof(update),hipMemcpyHostToDevice);
    unsigned numThreadsupdate   = (batchElements <  THREADS_PER_BLOCK)? THREADS_PER_BLOCK: batchElements;
    unsigned numBlocksupdate    = (batchElements+ THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK;

    // HERE WRITING AS A SINGLE KERNEL --> CAN HAVE TWO KERNELS FOR ADD AND DELETE
    update_edges_kernel<<<numBlocksupdate,numThreadsupdate>>>(batchElements,d_batch_updates,d_weight,d_meta,d_data);
    hipDeviceSynchronize();
    checkCudaError(0);
  }


  //LAUNCH CONFIG
  const unsigned threadsPerBlock = THREADS_PER_BLOCK;
  unsigned numThreads   = (V < threadsPerBlock)? THREADS_PER_BLOCK: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  int *h_excess; // ADDED BY SHRUTHI
  h_excess = (int *)malloc(sizeof(int)*V); // ADDED BY SHRUTHI


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));



  
    // TIMER START
  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  float milliseconds1 = 0;
  hipEventRecord(start1,0);
  calculate_max_flow_kernel_10<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(10);



  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  calculate_max_flow_kernel_14<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_weight,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  checkCudaError(14);
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);




  bool flag = true; // asst in .cu

  do{
    flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag), &flag, sizeof(bool), 0, hipMemcpyHostToDevice);
    calculate_max_flow_kernel_16<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    checkCudaError(16);
    print_excess<<<numBlocks,numThreads>>>(V,d_excess);
    print_residual<<<numBlocks,numThreads>>>(V,d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag, HIP_SYMBOL(::flag), sizeof(bool), 0, hipMemcpyDeviceToHost);
  }while(flag);

  hipEventRecord(stop1,0);
  hipEventSynchronize(stop1);
  hipEventElapsedTime(&milliseconds1, start1, stop1);
  printf("STATIC GPU Time: %.6f ms\n", milliseconds1);

  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost); 
  int val = h_excess[sink];

  hipFree(d_excess);
  hipFree(d_height);

  //TIMER STOP
    free(h_excess);
  return val;

} //end FUN
int  main( int  argc, char** argv) {
  char* totalgraph=argv[1];
  char* updatesinp = argv[2];
  int source = atoi(argv[3]);
  int sink = atoi(argv[4]);
  int batchsize = atoi(argv[5]);
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();
  std::vector<update> updateEdges=G1.parseUpdates(updatesinp);
  int reval = calculate_max_flow(G1,updateEdges,batchsize,source,sink);
  hipDeviceSynchronize();
  printf("EVAL FLOW VALUE:%d\n",reval);
  return 0;
}
