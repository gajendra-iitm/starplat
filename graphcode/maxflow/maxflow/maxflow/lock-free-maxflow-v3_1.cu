#include "hip/hip_runtime.h"
// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "lock-free-maxflow-v3_1.h"
#include<bits/stdc++.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include<bits/stdc++.h>
__global__ void update_kernel(int src, int dest,int new_capacity, int* d_residual_capacity,int *d_rev_residual_capacity,int *d_weight, int* d_meta,int* d_data){
  int edge = -1;
  for( int  i=d_meta[src];i<d_meta[src+1];i++)
  {
    if(d_data[i]==dest)
    {
      edge = i;
      break;
    }
  }
  
  printf("src:%d dest:%d new_capacity:%d old_capacity:%d residual[src,dst]:%d residual[dst,src]:%d\n",src,dest,new_capacity,d_weight[edge],d_residual_capacity[edge],d_rev_residual_capacity[edge]);

  if(new_capacity<d_weight[edge]){
    if(d_rev_residual_capacity[edge]>new_capacity){
      d_rev_residual_capacity[edge] = new_capacity;
    }
    d_residual_capacity[edge] = new_capacity - d_rev_residual_capacity[edge];
    
  } else{
     d_residual_capacity[edge] =  d_residual_capacity[edge] + new_capacity - d_weight[edge];
  }
  d_weight[edge] = new_capacity;
}

int do_max_flow(graph& g,int source,int sink)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START



  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

    hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);
  do_max_flow_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
  do_max_flow_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_weight,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag2 = true; // asst in .cu

  do{
    flag2 = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source), &source, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink), &sink, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag2), &flag2, sizeof(bool), 0, hipMemcpyHostToDevice);
    do_max_flow_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source, HIP_SYMBOL(::source), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink, HIP_SYMBOL(::sink), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag2, HIP_SYMBOL(::flag2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag2);

  //hipFree up!! all propVars in this BLOCK!


  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  int *h_excess;
  h_excess = (int *)malloc(sizeof(int)*V);
  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost);
  int val = h_excess[sink];
  hipFree(d_excess);
  hipFree(d_height);
  free(h_excess);
  return val;

} //end FUN
int recalculate_max_flow(graph& g,char *updatesinp,int source2,int sink2)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


    int *h_excess; // ADDED BY SHRUTHI
  h_excess = (int *)malloc(sizeof(int)*V); // ADDED BY SHRUTHI


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_height;
  hipMalloc(&d_height, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  int* d_excess;
  hipMalloc(&d_excess, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_excess,(int)0);

  int* d_residual_capacity;
  hipMalloc(&d_residual_capacity, sizeof(int)*(E));

  int* d_rev_residual_capacity;
  hipMalloc(&d_rev_residual_capacity, sizeof(int)*(E));

  int* d_reverse_edge;
  hipMalloc(&d_reverse_edge, sizeof(int)*(E));

    // TIMER START
  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  float milliseconds1 = 0;
  hipEventRecord(start1,0);
  recalculate_max_flow_kernel_10<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity);
  hipDeviceSynchronize();



  hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
  recalculate_max_flow_kernel_14<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_weight,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  bool flag = true; // asst in .cu

  do{
    flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag), &flag, sizeof(bool), 0, hipMemcpyHostToDevice);
    recalculate_max_flow_kernel_16<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag, HIP_SYMBOL(::flag), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag);





  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  hipEventRecord(stop1,0);
  hipEventSynchronize(stop1);
  hipEventElapsedTime(&milliseconds1, start1, stop1);
  printf("Original Graph GPU Time: %.6f ms\n", milliseconds1); // ADDED BY SHRUTHI

  hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost); // ADDED BY SHRUTHI
  printf("Static Maxflow:%d\n",h_excess[sink2]); // ADDED BY SHRUTHI
  printf("After static calculation\n");  // ADDED BY SHRUTHI
  print_residual<<<numBlocks, threadsPerBlock>>>(V, d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);


  std::vector<update> updateEdges=g.parseUpdates(updatesinp);

  for(auto &u:updateEdges){
    update_kernel<<<1,1>>>(u.source,u.destination,u.weight,d_residual_capacity,d_rev_residual_capacity,d_weight,d_meta,d_data);
  }
    
  recalculate_max_flow_kernel_19<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_rev_residual_capacity,d_excess);
  hipDeviceSynchronize();
      print_excess<<<numBlocks, threadsPerBlock>>>(V,d_excess);
  print_residual<<<numBlocks, threadsPerBlock>>>(V, d_meta,d_residual_capacity,d_rev_residual_capacity,d_data);

  int* d_height2;
  hipMalloc(&d_height2, sizeof(int)*(V));

    hipEvent_t start2, stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  float milliseconds2 = 0;
  hipEventRecord(start2,0);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height2,(int)0);

  initIndex<int><<<1,1>>>(V,d_height2,sink2,(int)g.num_nodes()); //InitIndexDevice
  hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
  recalculate_max_flow_kernel_22<<<numBlocks, threadsPerBlock>>>(V,E,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_excess,d_rev_residual_capacity);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);



  ; // asst in .cu

  flag = true;
  do{
    flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag), &flag, sizeof(bool), 0, hipMemcpyHostToDevice);
    recalculate_max_flow_kernel_24<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_reverse_edge,d_residual_capacity,d_rev_residual_capacity,d_excess,d_height2);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag, HIP_SYMBOL(::flag), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag);
  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_height,(int)0);

  initIndex<int><<<1,1>>>(V,d_height,source2,(int)g.num_nodes()); //InitIndexDevice
  flag = true;
  do{
    flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(::source2), &source2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::sink2), &sink2, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::flag), &flag, sizeof(bool), 0, hipMemcpyHostToDevice);
    recalculate_max_flow_kernel_27<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_excess,d_reverse_edge,d_rev_residual_capacity,d_residual_capacity,d_height);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&source2, HIP_SYMBOL(::source2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sink2, HIP_SYMBOL(::sink2), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&flag, HIP_SYMBOL(::flag), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu


  }while(flag);

  //hipFree up!! all propVars in this BLOCK!
    hipEventRecord(stop2,0);
  hipEventSynchronize(stop2);
  hipEventElapsedTime(&milliseconds2, start2, stop2);
  printf("Dynamic GPU Time: %.6f ms\n", milliseconds2);

    hipMemcpy(h_excess, d_excess, V * sizeof(int), hipMemcpyDeviceToHost);
  int val = h_excess[sink2];

  hipFree(d_height2);
  hipFree(d_excess);
  hipFree(d_height);

  //TIMER STOP
    free(h_excess);
  return val;

} //end FUN
int  main( int  argc, char** argv) {

  char* totalgraph=argv[1];
  char* updatedgraph = argv[2];
  char* updatesinp = argv[3];
  int source = atoi(argv[4]);
  int sink = atoi(argv[5]);
  graph G1(totalgraph,"cuda",true);
  G1.parseGraph();

   int reval = recalculate_max_flow(G1,updatesinp,source,sink);
  hipDeviceSynchronize();
  graph G2(updatedgraph,"cuda",true);
  G2.parseGraph();
  int eval = do_max_flow(G2,source,sink);
  printf("reval:%d eval%d\n",reval,eval);
  if(reval!=eval){
    
    printf("=====================PANIC INCORRECT========================\n");
  } else {
    printf("=====================ANSWER MATCHES========================\n");
  }
  return 0;
}