#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA Kernel
__global__ void kernel(float* d_data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        d_data[idx] *= idx; // Example computation
    }
}

int main() {
    int N = 10;

    // Host vector
    thrust::host_vector<float> h_vec(N, 1.0f);

    // Device vector
    thrust::device_vector<float> d_vec = h_vec;

    // Get raw pointer to device data
    float* d_ptr = thrust::raw_pointer_cast(d_vec.data());

    // Launch Kernel
    kernel<<<1, N>>>(d_ptr, N);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
if (err != hipSuccess) {
    std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
}


    thrust::host_vector<float> h(N,2.9f);
    // Copy data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h.begin());

    // Print results
    for (int i = 0; i < N; i++) {
        std::cout << h[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}

